#include "hip/hip_runtime.h"
kernal.cu
#include <cmath>
#include <limits.h>
#include <mutex>
#include <string>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#ifndef MACRO_H
#define MACRO_H



//HACK TO MAKE CPU WORK!
//HACK SLUT

#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
#else
__device__ double atomicAdd(double* a, double b) { return b; }
#endif

#define GPU __device__ 
#define CPU __host__
#define GLOBAL __global__
#define IS_GPU __HIPCC__

#define DBL_MAX 1.7976931348623158e+308 //max 64 bit double value
#define DBL_EPSILON 2.2204460492503131e-016 // smallest such that 1.0+DBL_EPSILON != 1.0


//While loop done to enfore ; after macro call. See: 
//https://stackoverflow.com/a/61363791/17430854
#define CUDA_CHECK(x)             \
do{                          \
if ((x) != hipSuccess) {    \
    throw std::runtime_error(std::string("cuda error ") + std::to_string(x) + " in file '" + __FILE__ + "' on line "+  std::to_string(__LINE__)); \
}                             \
}while(0)


__host__ __device__ __forceinline__ void cuda_syncthreads_()
{
    #ifdef __HIPCC__
    #define cuda_SYNCTHREADS() __syncthreads()
    #else
    #define cuda_SYNCTHREADS()
    #endif
} 

#ifdef __HIPCC__
#define cuda_SYNCTHREADS() __syncthreads()
#else
#define cuda_SYNCTHREADS()
#endif

#endif
#ifndef SIM_CONFIG_H
#define SIM_CONFIG_H

#define SHARED_MEMORY_PR_THREAD 32

struct io_paths;
struct output_properties;

struct sim_config
{
    //simulation setup
    unsigned int blocks = 1;
    unsigned int threads = 1;
    unsigned int cpu_threads = 1;
    unsigned int simulation_amount = 1;
    unsigned int simulation_repetitions = 1;
    unsigned long long seed = 1;
    int write_mode = 0;
    bool use_max_steps = true;
    unsigned int max_steps_pr_sim = 1;
    double max_global_progression = 1;
    bool verbose = false;
    
    enum device_opt
    {
        device,
        host,
        both
    } sim_location = device;
    
    //model parameters (setup using function)
    bool use_shared_memory = false;
    bool use_jit = false;
    unsigned max_expression_depth = 1;
    unsigned max_edge_fanout = 0;
    unsigned tracked_variable_count = 1;
    unsigned variable_count = 1;
    unsigned network_size = 1;
    unsigned node_count = 0;
    unsigned initial_urgent = 0;
    unsigned initial_committed = 0;
    
    //paths
    io_paths* paths;

    output_properties* properties;
    double alpha = 0.005;
    double epsilon = 0.005;
    
    //pointers
    void* cache = nullptr;
    hiprandState* random_state_arr = nullptr;
    
    size_t total_simulations() const
    {
        return static_cast<size_t>(blocks) * threads * simulation_amount;
    }

    bool can_use_cuda_shared_memory(const size_t model_size) const
    {
        return (static_cast<size_t>(this->threads) * SHARED_MEMORY_PR_THREAD) > (model_size);
    }
};

#endif
template<typename T>
class my_stack
{
private:
    T* store_;
    int size_;
    int count_;
public:
    CPU GPU explicit my_stack(T* store, int size)
    {
        this->store_ = store;
        this->size_ = size;
        this->count_ = 0;
    }

    CPU GPU void push(T& t)
    {
        store_[count_++] = t;
    }

    CPU GPU T pop()
    {
        // if(this->count_ <= 0) printf("stack is empty, cannot pop! PANIC!");
        return this->store_[--this->count_];
    }

    CPU GPU T peak()
    {
        // if(this->count_ <= 0) printf("stack is empty, cannot peak! PANIC!");
        return this->store_[this->count_ - 1];
    }

    CPU GPU int count() const
    {
        return this->count_;
    }

    CPU GPU void clear()
    {
        this->count_ = 0;
    }
};
#ifndef DOMAIN_H
#define DOMAIN_H

struct state;
struct edge;
struct node;


#define HAS_HIT_MAX_STEPS(x) ((x) < 0)

template<typename T>
struct arr
{
    T* store;
    int size;

    static arr<T> empty(){ return arr<T>{nullptr, 0}; }
};

#define IS_LEAF(x) ((x) < 2)
struct expr  // NOLINT(cppcoreguidelines-pro-type-member-init)
{
    enum operators
    {
        //value types
        literal_ee = 0,
        clock_variable_ee = 1,

        //random
        random_ee,

        //arithmatic types
        plus_ee,
        minus_ee,
        multiply_ee,
        division_ee,
        power_ee,
        negation_ee,
        sqrt_ee,
        modulo_ee,

        //boolean types
        less_equal_ee,
        greater_equal_ee,
        less_ee,
        greater_ee,
        equal_ee,
        not_equal_ee,
        not_ee,

        //conditional types
        conditional_ee,
        compiled_ee
        
    } operand = literal_ee;
    
    expr* left = nullptr;
    expr* right = nullptr;

    union
    {
        double value = 1.0;
        int variable_id;
        expr* conditional_else;
        int compile_id;
    };

    CPU GPU double evaluate_expression(state* state);
};


/**
 * \brief Takes in constraint::operators and returns bool whether the operand is a constraint
 * \param a constraint::operators
 */
#define IS_INVARIANT(a) ((a) < 2)
struct constraint
{
    enum operators
    {
        less_equal_c = 0,
        less_c = 1,
        greater_equal_c = 2,
        greater_c = 3,
        equal_c = 4,
        not_equal_c = 5,
        compiled_c
    } operand;

    bool uses_variable;
    union //left hand side
    {
        expr* value;
        int variable_id;
        int compile_id;
    };
    expr* expression; //right hand side
    CPU GPU bool evaluate_constraint(state* state) const;
    CPU GPU static bool evaluate_constraint_set(const arr<constraint>& con_arr, state* state);
};

struct clock_var
{
    int id;
    bool should_track;
    unsigned rate;
    double value;
    double max_value;

    CPU GPU void add_time(const double time);
    CPU GPU void set_value(const double val);
};


#define IS_URGENT(x) ((x) > 2)
struct node
{
    int id{};
    enum node_types
    {
        location = 0,
        goal = 1,
        branch = 2,
        urgent = 3,
        committed = 4,
    } type = location;
    expr* lamda{};
    arr<edge> edges = arr<edge>::empty();
    arr<constraint> invariants = arr<constraint>::empty();
    CPU GPU double max_progression(state* state, bool* is_finite) const;
};

struct update
{
    int variable_id;
    expr* expression;
    CPU GPU void apply_update(state* state) const;
};


#define TAU_CHANNEL 0
#define IS_TAU(x) ((x) == 0)
#define IS_LISTENER(x) ((x) < 0)
#define CAN_SYNC(brod, list) ((brod) == (-(list)))
#define IS_BROADCASTER(x) ((x) > 0)


struct edge
{
    int channel{};
    expr* weight{};
    node* dest{};
    arr<constraint> guards = arr<constraint>::empty();
    arr<update> updates = arr<update>::empty();
    CPU GPU void apply_updates(state* state) const;
    CPU GPU bool edge_enabled(state* state) const;
};

struct network
{
    arr<node*> automatas;
    arr<clock_var> variables;
};



struct state
{
    unsigned urgent_count = 0;
    unsigned committed_count = 0;
    unsigned simulation_id;
    unsigned steps;
    double global_time;

    arr<node*> models;
    arr<clock_var> variables;

    struct w_edge
    {
        edge* e;
        double w;
    };
    
    hiprandState* random;
    my_stack<expr*> expr_stack;
    my_stack<double> value_stack;
    my_stack<w_edge> edge_stack;

    CPU GPU void traverse_edge(int process_id, node* dest);
    CPU GPU void broadcast_channel(const int channel, const int process);
    CPU GPU static state init(void* cache, hiprandState* random, const network* model, const unsigned expr_depth, const unsigned fanout);
    CPU GPU void reset(const unsigned sim_id, const network* model);
};
#endif


//Please do not change the argument names, they are required for JIT compilation 
CPU GPU double evaluate_compiled_expression(const expr* ex, state* state)
{
    //DO NOT REMOVE FOLLOWING COMMENT! IT IS USED AS SEARCH TARGET FOR JIT COMPILATION!!!
    switch(ex->compile_id){

}

    
    return 0.0;
}

//Please do not change the argument names, they are required for JIT compilation 
CPU GPU bool evaluate_compiled_constraint(const constraint* con, state* state)
{
    //DO NOT REMOVE FOLLOWING COMMENT! IT IS USED AS SEARCH TARGET FOR JIT COMPILATION!!!
    switch(con->compile_id){
case 0: return (state->variables.store[13].value)==(0); break;
case 1: return (state->variables.store[13].value)==(1); break;
case 2: return (state->variables.store[13].value)==(1); break;
case 3: return (state->variables.store[13].value)==(0); break;
case 4: return (state->variables.store[13].value)==(5); break;
case 5: return (state->variables.store[1].value)<=(167); break;
case 6: return (state->variables.store[13].value)==(4); break;
case 7: return (state->variables.store[1].value)<=(85); break;
case 8: return (state->variables.store[13].value)==(5); break;
case 9: return (state->variables.store[1].value)<=(167); break;
case 10: return (state->variables.store[1].value)<=(159); break;
case 11: return (state->variables.store[13].value)==(5); break;
case 12: return (state->variables.store[1].value)<=(167); break;
case 13: return (state->variables.store[1].value)>=(159); break;
case 14: return (state->variables.store[13].value)==(6); break;
case 15: return (state->variables.store[13].value)==(3); break;
case 16: return (state->variables.store[1].value)<=(167); break;
case 17: return (state->variables.store[13].value)==(6); break;
case 18: return (state->variables.store[13].value)==(8); break;
case 19: return (state->variables.store[1].value)<=(167); break;
case 20: return (state->variables.store[13].value)==(0); break;
case 21: return (state->variables.store[13].value)==(8); break;
case 22: return (state->variables.store[1].value)<=(167); break;
case 23: return (state->variables.store[1].value)<(159); break;
case 24: return (state->variables.store[13].value)==(8); break;
case 25: return (state->variables.store[1].value)<=(167); break;
case 26: return (state->variables.store[1].value)>=(159); break;
case 27: return (state->variables.store[13].value)==(8); break;
case 28: return (state->variables.store[1].value)<=(167); break;
case 29: return (state->variables.store[13].value)==(3); break;
case 30: return (state->variables.store[1].value)<=(167); break;
case 31: return (state->variables.store[1].value)<(159); break;
case 32: return (state->variables.store[13].value)==(3); break;
case 33: return (state->variables.store[1].value)<=(167); break;
case 34: return (state->variables.store[13].value)==(5); break;
case 35: return (state->variables.store[1].value)<=(167); break;
case 36: return (state->variables.store[1].value)>=(159); break;
case 37: return (state->variables.store[13].value)==(7); break;
case 38: return (state->variables.store[1].value)<=(167); break;
case 39: return (state->variables.store[13].value)==(7); break;
case 40: return (state->variables.store[1].value)<=(167); break;
case 41: return (state->variables.store[1].value)<(159); break;
case 42: return (state->variables.store[13].value)==(7); break;
case 43: return (state->variables.store[1].value)<=(167); break;
case 44: return (state->variables.store[1].value)>=(159); break;
case 45: return (state->variables.store[13].value)==(7); break;
case 46: return (state->variables.store[1].value)<=(167); break;
case 47: return (state->variables.store[13].value)==(4); break;
case 48: return (state->variables.store[1].value)<=(85); break;
case 49: return (state->variables.store[1].value)>=(76); break;
case 50: return (state->variables.store[13].value)==(6); break;
case 51: return (state->variables.store[13].value)==(2); break;
case 52: return (state->variables.store[1].value)<=(85); break;
case 53: return (state->variables.store[13].value)==(2); break;
case 54: return (state->variables.store[1].value)<=(85); break;
case 55: return (state->variables.store[1].value)>=(76); break;
case 56: return (state->variables.store[13].value)==(7); break;
case 57: return (state->variables.store[1].value)<=(167); break;
case 58: return (state->variables.store[13].value)==(4); break;
case 59: return (state->variables.store[1].value)<=(85); break;
case 60: return (state->variables.store[13].value)==(3); break;
case 61: return (state->variables.store[1].value)<=(167); break;
case 62: return (state->variables.store[13].value)==(2); break;
case 63: return (state->variables.store[1].value)<=(85); break;
case 64: return (state->variables.store[14].value)==(0); break;
case 65: return (state->variables.store[14].value)==(5); break;
case 66: return (state->variables.store[5].value)<=(30); break;
case 67: return (state->variables.store[14].value)==(3); break;
case 68: return (state->variables.store[5].value)<=(30); break;
case 69: return (state->variables.store[14].value)==(1); break;
case 70: return (state->variables.store[3].value)<=(30); break;
case 71: return (state->variables.store[14].value)==(5); break;
case 72: return (state->variables.store[5].value)<=(30); break;
case 73: return (state->variables.store[14].value)==(9); break;
case 74: return (state->variables.store[3].value)<=(30); break;
case 75: return (state->variables.store[14].value)==(6); break;
case 76: return (state->variables.store[3].value)<=(30); break;
case 77: return (state->variables.store[7].value)==(state->variables.store[1].value); break;
case 78: return (state->variables.store[5].value)>=(5); break;
case 79: return (state->variables.store[14].value)==(0); break;
case 80: return (state->variables.store[14].value)==(5); break;
case 81: return (state->variables.store[5].value)<=(30); break;
case 82: return (state->variables.store[14].value)==(9); break;
case 83: return (state->variables.store[3].value)<=(30); break;
case 84: return (state->variables.store[7].value)==(state->variables.store[1].value); break;
case 85: return (state->variables.store[3].value)>=(5); break;
case 86: return (state->variables.store[14].value)==(3); break;
case 87: return (state->variables.store[5].value)<=(30); break;
case 88: return (state->variables.store[14].value)==(3); break;
case 89: return (state->variables.store[5].value)<=(30); break;
case 90: return (state->variables.store[14].value)==(7); break;
case 91: return (state->variables.store[3].value)<=(30); break;
case 92: return (state->variables.store[14].value)==(4); break;
case 93: return (state->variables.store[3].value)<=(30); break;
case 94: return (state->variables.store[11].value)==(state->variables.store[1].value); break;
case 95: return (state->variables.store[5].value)>=(5); break;
case 96: return (state->variables.store[14].value)==(0); break;
case 97: return (state->variables.store[14].value)==(3); break;
case 98: return (state->variables.store[5].value)<=(30); break;
case 99: return (state->variables.store[14].value)==(7); break;
case 100: return (state->variables.store[3].value)<=(30); break;
case 101: return (state->variables.store[11].value)==(state->variables.store[1].value); break;
case 102: return (state->variables.store[3].value)>=(5); break;
case 103: return (state->variables.store[14].value)==(1); break;
case 104: return (state->variables.store[3].value)<=(30); break;
case 105: return (state->variables.store[14].value)==(7); break;
case 106: return (state->variables.store[3].value)<=(30); break;
case 107: return (state->variables.store[14].value)==(1); break;
case 108: return (state->variables.store[3].value)<=(30); break;
case 109: return (state->variables.store[14].value)==(8); break;
case 110: return (state->variables.store[3].value)<=(30); break;
case 111: return (state->variables.store[14].value)==(2); break;
case 112: return (state->variables.store[3].value)<=(30); break;
case 113: return (state->variables.store[9].value)==(state->variables.store[1].value); break;
case 114: return (state->variables.store[3].value)>=(5); break;
case 115: return (state->variables.store[14].value)==(0); break;
case 116: return (state->variables.store[14].value)==(1); break;
case 117: return (state->variables.store[3].value)<=(30); break;
case 118: return (state->variables.store[14].value)==(8); break;
case 119: return (state->variables.store[3].value)<=(30); break;
case 120: return (state->variables.store[14].value)==(8); break;
case 121: return (state->variables.store[3].value)<=(30); break;
case 122: return (state->variables.store[9].value)==(state->variables.store[1].value); break;
case 123: return (state->variables.store[3].value)>=(5); break;
case 124: return (state->variables.store[14].value)==(5); break;
case 125: return (state->variables.store[5].value)<=(30); break;
case 126: return (state->variables.store[14].value)==(2); break;
case 127: return (state->variables.store[3].value)<=(30); break;
case 128: return (state->variables.store[9].value)==(state->variables.store[1].value); break;
case 129: return (state->variables.store[3].value)>=(5); break;
case 130: return (state->variables.store[14].value)==(3); break;
case 131: return (state->variables.store[5].value)<=(30); break;
case 132: return (state->variables.store[14].value)==(2); break;
case 133: return (state->variables.store[3].value)<=(30); break;
case 134: return (state->variables.store[14].value)==(4); break;
case 135: return (state->variables.store[3].value)<=(30); break;
case 136: return (state->variables.store[11].value)==(state->variables.store[1].value); break;
case 137: return (state->variables.store[3].value)>=(5); break;
case 138: return (state->variables.store[14].value)==(5); break;
case 139: return (state->variables.store[5].value)<=(30); break;
case 140: return (state->variables.store[14].value)==(4); break;
case 141: return (state->variables.store[3].value)<=(30); break;
case 142: return (state->variables.store[14].value)==(6); break;
case 143: return (state->variables.store[3].value)<=(30); break;
case 144: return (state->variables.store[7].value)==(state->variables.store[1].value); break;
case 145: return (state->variables.store[3].value)>=(5); break;
case 146: return (state->variables.store[14].value)==(1); break;
case 147: return (state->variables.store[3].value)<=(30); break;
case 148: return (state->variables.store[14].value)==(6); break;
case 149: return (state->variables.store[3].value)<=(30); break;
case 150: return (state->variables.store[15].value)==(0); break;
case 151: return (state->variables.store[15].value)==(1); break;
case 152: return (state->variables.store[15].value)==(1); break;
case 153: return (state->variables.store[15].value)==(0); break;
case 154: return (state->variables.store[15].value)==(5); break;
case 155: return (state->variables.store[2].value)<=(167); break;
case 156: return (state->variables.store[15].value)==(4); break;
case 157: return (state->variables.store[2].value)<=(85); break;
case 158: return (state->variables.store[15].value)==(5); break;
case 159: return (state->variables.store[2].value)<=(167); break;
case 160: return (state->variables.store[2].value)<=(159); break;
case 161: return (state->variables.store[15].value)==(5); break;
case 162: return (state->variables.store[2].value)<=(167); break;
case 163: return (state->variables.store[2].value)>=(159); break;
case 164: return (state->variables.store[15].value)==(6); break;
case 165: return (state->variables.store[15].value)==(3); break;
case 166: return (state->variables.store[2].value)<=(167); break;
case 167: return (state->variables.store[15].value)==(6); break;
case 168: return (state->variables.store[15].value)==(8); break;
case 169: return (state->variables.store[2].value)<=(167); break;
case 170: return (state->variables.store[15].value)==(0); break;
case 171: return (state->variables.store[15].value)==(8); break;
case 172: return (state->variables.store[2].value)<=(167); break;
case 173: return (state->variables.store[2].value)<(159); break;
case 174: return (state->variables.store[15].value)==(8); break;
case 175: return (state->variables.store[2].value)<=(167); break;
case 176: return (state->variables.store[2].value)>=(159); break;
case 177: return (state->variables.store[15].value)==(8); break;
case 178: return (state->variables.store[2].value)<=(167); break;
case 179: return (state->variables.store[15].value)==(3); break;
case 180: return (state->variables.store[2].value)<=(167); break;
case 181: return (state->variables.store[2].value)<(159); break;
case 182: return (state->variables.store[15].value)==(3); break;
case 183: return (state->variables.store[2].value)<=(167); break;
case 184: return (state->variables.store[15].value)==(5); break;
case 185: return (state->variables.store[2].value)<=(167); break;
case 186: return (state->variables.store[2].value)>=(159); break;
case 187: return (state->variables.store[15].value)==(7); break;
case 188: return (state->variables.store[2].value)<=(167); break;
case 189: return (state->variables.store[15].value)==(7); break;
case 190: return (state->variables.store[2].value)<=(167); break;
case 191: return (state->variables.store[2].value)<(159); break;
case 192: return (state->variables.store[15].value)==(7); break;
case 193: return (state->variables.store[2].value)<=(167); break;
case 194: return (state->variables.store[2].value)>=(159); break;
case 195: return (state->variables.store[15].value)==(7); break;
case 196: return (state->variables.store[2].value)<=(167); break;
case 197: return (state->variables.store[15].value)==(4); break;
case 198: return (state->variables.store[2].value)<=(85); break;
case 199: return (state->variables.store[2].value)>=(76); break;
case 200: return (state->variables.store[15].value)==(6); break;
case 201: return (state->variables.store[15].value)==(2); break;
case 202: return (state->variables.store[2].value)<=(85); break;
case 203: return (state->variables.store[15].value)==(2); break;
case 204: return (state->variables.store[2].value)<=(85); break;
case 205: return (state->variables.store[2].value)>=(76); break;
case 206: return (state->variables.store[15].value)==(7); break;
case 207: return (state->variables.store[2].value)<=(167); break;
case 208: return (state->variables.store[15].value)==(4); break;
case 209: return (state->variables.store[2].value)<=(85); break;
case 210: return (state->variables.store[15].value)==(3); break;
case 211: return (state->variables.store[2].value)<=(167); break;
case 212: return (state->variables.store[15].value)==(2); break;
case 213: return (state->variables.store[2].value)<=(85); break;
case 214: return (state->variables.store[16].value)==(0); break;
case 215: return (state->variables.store[16].value)==(5); break;
case 216: return (state->variables.store[6].value)<=(30); break;
case 217: return (state->variables.store[16].value)==(3); break;
case 218: return (state->variables.store[6].value)<=(30); break;
case 219: return (state->variables.store[16].value)==(1); break;
case 220: return (state->variables.store[4].value)<=(30); break;
case 221: return (state->variables.store[16].value)==(5); break;
case 222: return (state->variables.store[6].value)<=(30); break;
case 223: return (state->variables.store[16].value)==(9); break;
case 224: return (state->variables.store[4].value)<=(30); break;
case 225: return (state->variables.store[16].value)==(6); break;
case 226: return (state->variables.store[4].value)<=(30); break;
case 227: return (state->variables.store[8].value)==(state->variables.store[1].value); break;
case 228: return (state->variables.store[6].value)>=(5); break;
case 229: return (state->variables.store[16].value)==(0); break;
case 230: return (state->variables.store[16].value)==(5); break;
case 231: return (state->variables.store[6].value)<=(30); break;
case 232: return (state->variables.store[16].value)==(9); break;
case 233: return (state->variables.store[4].value)<=(30); break;
case 234: return (state->variables.store[8].value)==(state->variables.store[1].value); break;
case 235: return (state->variables.store[4].value)>=(5); break;
case 236: return (state->variables.store[16].value)==(3); break;
case 237: return (state->variables.store[6].value)<=(30); break;
case 238: return (state->variables.store[16].value)==(3); break;
case 239: return (state->variables.store[6].value)<=(30); break;
case 240: return (state->variables.store[16].value)==(7); break;
case 241: return (state->variables.store[4].value)<=(30); break;
case 242: return (state->variables.store[16].value)==(4); break;
case 243: return (state->variables.store[4].value)<=(30); break;
case 244: return (state->variables.store[12].value)==(state->variables.store[1].value); break;
case 245: return (state->variables.store[6].value)>=(5); break;
case 246: return (state->variables.store[16].value)==(0); break;
case 247: return (state->variables.store[16].value)==(3); break;
case 248: return (state->variables.store[6].value)<=(30); break;
case 249: return (state->variables.store[16].value)==(7); break;
case 250: return (state->variables.store[4].value)<=(30); break;
case 251: return (state->variables.store[12].value)==(state->variables.store[1].value); break;
case 252: return (state->variables.store[4].value)>=(5); break;
case 253: return (state->variables.store[16].value)==(1); break;
case 254: return (state->variables.store[4].value)<=(30); break;
case 255: return (state->variables.store[16].value)==(7); break;
case 256: return (state->variables.store[4].value)<=(30); break;
case 257: return (state->variables.store[16].value)==(1); break;
case 258: return (state->variables.store[4].value)<=(30); break;
case 259: return (state->variables.store[16].value)==(8); break;
case 260: return (state->variables.store[4].value)<=(30); break;
case 261: return (state->variables.store[16].value)==(2); break;
case 262: return (state->variables.store[4].value)<=(30); break;
case 263: return (state->variables.store[10].value)==(state->variables.store[1].value); break;
case 264: return (state->variables.store[4].value)>=(5); break;
case 265: return (state->variables.store[16].value)==(0); break;
case 266: return (state->variables.store[16].value)==(1); break;
case 267: return (state->variables.store[4].value)<=(30); break;
case 268: return (state->variables.store[16].value)==(8); break;
case 269: return (state->variables.store[4].value)<=(30); break;
case 270: return (state->variables.store[16].value)==(8); break;
case 271: return (state->variables.store[4].value)<=(30); break;
case 272: return (state->variables.store[10].value)==(state->variables.store[1].value); break;
case 273: return (state->variables.store[4].value)>=(5); break;
case 274: return (state->variables.store[16].value)==(5); break;
case 275: return (state->variables.store[6].value)<=(30); break;
case 276: return (state->variables.store[16].value)==(2); break;
case 277: return (state->variables.store[4].value)<=(30); break;
case 278: return (state->variables.store[10].value)==(state->variables.store[1].value); break;
case 279: return (state->variables.store[4].value)>=(5); break;
case 280: return (state->variables.store[16].value)==(3); break;
case 281: return (state->variables.store[6].value)<=(30); break;
case 282: return (state->variables.store[16].value)==(2); break;
case 283: return (state->variables.store[4].value)<=(30); break;
case 284: return (state->variables.store[16].value)==(4); break;
case 285: return (state->variables.store[4].value)<=(30); break;
case 286: return (state->variables.store[12].value)==(state->variables.store[1].value); break;
case 287: return (state->variables.store[4].value)>=(5); break;
case 288: return (state->variables.store[16].value)==(5); break;
case 289: return (state->variables.store[6].value)<=(30); break;
case 290: return (state->variables.store[16].value)==(4); break;
case 291: return (state->variables.store[4].value)<=(30); break;
case 292: return (state->variables.store[16].value)==(6); break;
case 293: return (state->variables.store[4].value)<=(30); break;
case 294: return (state->variables.store[8].value)==(state->variables.store[1].value); break;
case 295: return (state->variables.store[4].value)>=(5); break;
case 296: return (state->variables.store[16].value)==(1); break;
case 297: return (state->variables.store[4].value)<=(30); break;
case 298: return (state->variables.store[16].value)==(6); break;
case 299: return (state->variables.store[4].value)<=(30); break;

}

    
    return false;
}

//Please do not change the argument names, they are required for JIT compilation 
CPU GPU double evaluate_compiled_constraint_upper_bound(const constraint* con, state* state, bool* is_finite)
{
    //If this variable is marked const, then JIT compilation will not work.
    // ReSharper disable once CppLocalVariableMayBeConst
    double v0 = DBL_MAX;
    
    //DO NOT REMOVE FOLLOWING COMMENT! IT IS USED AS SEARCH TARGET FOR JIT COMPILATION!!!
    switch(con->compile_id){
case 10: v0 = ((159)-(state->variables.store[1].value))/ (1); break;
case 23: v0 = ((159)-(state->variables.store[1].value))/ (1); break;
case 31: v0 = ((159)-(state->variables.store[1].value))/ (1); break;
case 41: v0 = ((159)-(state->variables.store[1].value))/ (1); break;
case 91: v0 = ((30)-(state->variables.store[3].value))/ (1); break;
case 100: v0 = ((30)-(state->variables.store[3].value))/ (1); break;
case 106: v0 = ((30)-(state->variables.store[3].value))/ (1); break;
case 160: v0 = ((159)-(state->variables.store[2].value))/ (1); break;
case 173: v0 = ((159)-(state->variables.store[2].value))/ (1); break;
case 181: v0 = ((159)-(state->variables.store[2].value))/ (1); break;
case 191: v0 = ((159)-(state->variables.store[2].value))/ (1); break;
case 241: v0 = ((30)-(state->variables.store[4].value))/ (1); break;
case 250: v0 = ((30)-(state->variables.store[4].value))/ (1); break;
case 256: v0 = ((30)-(state->variables.store[4].value))/ (1); break;

}


    *is_finite = v0 < DBL_MAX; 
    return v0;
}

CPU GPU double evaluate_expression_node(const expr* expr, state* state)
{
    double v1, v2;
    switch (expr->operand) {
    case expr::literal_ee:
        return expr->value;
    case expr::clock_variable_ee:
        return state->variables.store[expr->variable_id].value;
    case expr::random_ee:
        v1 = state->value_stack.pop();
        return (1.0 - hiprand_uniform_double(state->random)) * v1;
    case expr::plus_ee: 
        v2 = state->value_stack.pop();
        v1 = state->value_stack.pop();
        return v1 + v2;
    case expr::minus_ee:
        v2 = state->value_stack.pop();
        v1 = state->value_stack.pop();
        return v1 - v2;
    case expr::multiply_ee: 
        v2 = state->value_stack.pop();
        v1 = state->value_stack.pop();
        return v1 * v2;
    case expr::division_ee:
        v2 = state->value_stack.pop();
        v1 = state->value_stack.pop();
        return v1 / v2;
    case expr::power_ee: 
        v2 = state->value_stack.pop();
        v1 = state->value_stack.pop();
        return pow(v1, v2);
    case expr::negation_ee: 
        v1 = state->value_stack.pop();
        return -v1;
    case expr::sqrt_ee: 
        v1 = state->value_stack.pop();
        return sqrt(v1);
    case expr::modulo_ee:
        v2 = state->value_stack.pop();
        v1 = state->value_stack.pop();
        return static_cast<double>(static_cast<int>(v1) % static_cast<int>(v2));
    case expr::less_equal_ee: 
        v2 = state->value_stack.pop();
        v1 = state->value_stack.pop();
        return v1 <= v2;
    case expr::greater_equal_ee: 
        v2 = state->value_stack.pop();
        v1 = state->value_stack.pop();
        return v1 >= v2;
    case expr::less_ee: 
        v2 = state->value_stack.pop();
        v1 = state->value_stack.pop();
        return v1 < v2;
    case expr::greater_ee: 
        v2 = state->value_stack.pop();
        v1 = state->value_stack.pop();
        return v1 > v2;
    case expr::equal_ee: 
        v2 = state->value_stack.pop();
        v1 = state->value_stack.pop();
        return abs(v1 - v2) <= DBL_EPSILON;
    case expr::not_equal_ee: 
        v2 = state->value_stack.pop();
        v1 = state->value_stack.pop();
        return abs(v1 - v2) > DBL_EPSILON;
    case expr::not_ee:
        v1 = state->value_stack.pop();
        return (abs(v1) < DBL_EPSILON);
    case expr::conditional_ee:
        v1 = state->value_stack.pop();
        state->value_stack.pop();
        return v1;
    case expr::compiled_ee: return 0.0; break;
    }
    return 0.0;
}

CPU GPU double expr::evaluate_expression(state* state)
{
    if(this->operand == literal_ee)
        return this->value;
    if(this->operand == clock_variable_ee)
        return state->variables.store[this->variable_id].value;
    if(this->operand == compiled_ee)
        return evaluate_compiled_expression(this, state);

    state->expr_stack.clear();
    state->value_stack.clear();
    expr* current = this;
    while (true)
    {
        while(current != nullptr)
        {
            state->expr_stack.push(current);
            
            if(!IS_LEAF(current->operand)) //only push twice if it has children
                state->expr_stack.push(current);
            
            current = current->left;
        }
        if(state->expr_stack.count() == 0)
        {
            break;
        }
        current = state->expr_stack.pop();
        
        if(state->expr_stack.count() > 0 && state->expr_stack.peak() == current)
        {
            current = (current->operand == conditional_ee && abs(state->value_stack.peak()) < DBL_EPSILON)
                ? current->conditional_else
                : current->right;
        }
        else
        {
            double val = evaluate_expression_node(current, state);
            state->value_stack.push(val);
            current = nullptr;
        }
    }

    if(state->value_stack.count() == 0)
    {
        // printf("Expression evaluation ended in no values! PANIC!\n");
        return 0.0;
    }
    
    return state->value_stack.pop();
}


CPU GPU double node::max_progression(state* state, bool* is_finite) const
{
    double max_bound = DBL_MAX;

    for (int i = 0; i < this->invariants.size; ++i)
    {
        const constraint con = this->invariants.store[i];
        double limit;
        
        if(IS_INVARIANT(con.operand))
        {
            if(!con.uses_variable) continue;
            const clock_var var = state->variables.store[con.variable_id];
            if(var.rate == 0) continue;
            limit = (con.expression->evaluate_expression(state) - var.value) / var.rate;
        }
        else if(con.operand == constraint::compiled_c)
        {
            bool finite = false;
            limit = evaluate_compiled_constraint_upper_bound(&con, state, &finite);

            if(!finite) continue;
        }
        else continue;
        max_bound = fmin(max_bound,  limit); //rate is >0.
    }
    *is_finite = max_bound < DBL_MAX;
    return max_bound;
}

CPU GPU bool constraint::evaluate_constraint(state* state) const
{
    if(this->operand == compiled_c)
        return evaluate_compiled_constraint(this, state);
    const double left = this->uses_variable
        ? state->variables.store[this->variable_id].value
        : this->value->evaluate_expression(state);
    const double right = this->expression->evaluate_expression(state);

    switch (this->operand)
    {
    case less_equal_c: return left <= right;
    case less_c: return left < right;
    case greater_equal_c: return left >= right;
    case greater_c: return left > right;
    case equal_c: return left == right;  // NOLINT(clang-diagnostic-float-equal)
    case not_equal_c: return left != right;  // NOLINT(clang-diagnostic-float-equal)
    case compiled_c: return false;
    }
    return false;
}


CPU GPU bool constraint::evaluate_constraint_set(const arr<constraint>& con_arr, state* state)
{
    for (int i = 0; i < con_arr.size; ++i)
    {
        if(!con_arr.store[i].evaluate_constraint(state))
            return false;
    }
    return true;
}

void clock_var::add_time(const double time)
{
    this->value += time*this->rate;
    this->max_value = fmax(this->max_value, this->value);
}

void clock_var::set_value(const double val)
{
    this->value = val;
    this->max_value = fmax(this->max_value, this->value);
}

CPU GPU inline void update::apply_update(state* state) const
{
    const double value = this->expression->evaluate_expression(state);
    state->variables.store[this->variable_id].set_value(value);
}

CPU GPU inline void edge::apply_updates(state* state) const
{
    for (int i = 0; i < this->updates.size; ++i)
    {
        this->updates.store[i].apply_update(state);
    }
}

CPU GPU inline bool edge::edge_enabled(state* state) const
{
    for (int i = 0; i < this->guards.size; ++i)
    {
        if(!this->guards.store[i].evaluate_constraint(state))
            return false;
    }
    return true;
}

CPU GPU void state::traverse_edge(const int process_id, node* dest)
{
    const node* current = this->models.store[process_id];
    
    this->urgent_count = this->urgent_count + IS_URGENT(dest->type) - IS_URGENT(current->type);
    this->committed_count = this->committed_count + (dest->type == node::committed) - (current->type == node::committed);
    
    this->models.store[process_id] = dest;
}

void inline state::broadcast_channel(const int channel, const int process)
{
    if(!IS_BROADCASTER(channel)) return;
    
    for (int p = 0; p < this->models.size; ++p)
    {
        const node* current = this->models.store[p];
        
        if (p == process) continue;
        if(current->type == node::goal) continue;
        if(!constraint::evaluate_constraint_set(current->invariants, this)) continue;
        
        const unsigned offset = hiprand(this->random) % current->edges.size;
        
        for (int e = 0; e < current->edges.size; ++e)
        {
            const edge current_e = current->edges.store[(e + offset) % current->edges.size];
            if(!IS_LISTENER(current_e.channel)) continue;
            if(!CAN_SYNC(channel, current_e.channel)) continue;

            this->traverse_edge(p, current_e.dest);

            current_e.apply_updates(this);
            break;
        }
    }
}

state state::init(void* cache, hiprandState* random, const network* model, const unsigned expr_depth, const unsigned fanout)
{
    node** nodes = static_cast<node**>(cache);
    cache = static_cast<void*>(&nodes[model->automatas.size]);
        
    clock_var* vars = static_cast<clock_var*>(cache);
    cache = static_cast<void*>(&vars[model->variables.size]);
        
    expr** exp = static_cast<expr**>(cache);
    cache = static_cast<void*>(&exp[expr_depth*2+1]);
        
    double* val_store = static_cast<double*>(cache);
    cache = static_cast<void*>(&val_store[expr_depth]);

    state::w_edge* fanout_store = static_cast<state::w_edge*>(cache);
    // cache = static_cast<void*>(&cache[fanout]);
    
    
    return state{
        0,
        0,
        0,
        0,
        0.0,
        arr<node*>{ nodes, model->automatas.size },
        arr<clock_var>{ vars, model->variables.size },
        random,
        my_stack<expr*>(exp, static_cast<int>(expr_depth*2+1)),
        my_stack<double>(val_store, static_cast<int>(expr_depth)),
        my_stack<state::w_edge>(fanout_store, static_cast<int>(fanout))
    };
}

void state::reset(const unsigned sim_id, const network* model)
{
    this->simulation_id = sim_id;
    this->steps = 0;
    this->global_time = 0.0;
    this->urgent_count = 0;
    this->committed_count = 0;
    for (int i = 0; i < model->automatas.size; ++i)
    {
        this->models.store[i] = model->automatas.store[i];
    }

    for (int i = 0; i < model->variables.size; ++i)
    {
        this->variables.store[i] = model->variables.store[i];
    }
}
#pragma once

struct model_size
{
    unsigned network_size = 0;
    unsigned nodes = 0; 
    unsigned edges = 0; 
    unsigned constraints = 0;
    unsigned updates = 0;
    unsigned variables = 0;
    unsigned expressions = 0;

    CPU GPU size_t total_memory_size() const;

    bool operator==(const model_size& rhs) const
    {
        return network_size == rhs.network_size && nodes == rhs.nodes
        && edges == rhs.edges
        && constraints == rhs.constraints
        && updates == rhs.updates
        && variables == rhs.variables
        && expressions == rhs.expressions;
    }
};

class model_oracle
{
public:
    model_oracle(void* point, const model_size& model_count)
    {
        this->initial_point = point;
        this->point = point;
        this->model_counter = model_count;
    }
    
    void* initial_point;
    void* point;
    model_size model_counter;

    template<typename T>
    CPU GPU T* get_diff(void* p1, T* p2, char* source) const;

    CPU GPU network* network_point() const;
    CPU GPU node** network_nodes_point() const;
    CPU GPU node* node_point() const;
    CPU GPU edge* edge_point() const;
    CPU GPU constraint* constraint_point() const;
    CPU GPU update* update_point() const;
    CPU GPU expr* expression_point() const;
    CPU GPU clock_var* variable_point() const;

    GPU network* move_to_shared_memory(char* shared_mem, const int threads) const;
};

template <typename T>
CPU GPU T* model_oracle::get_diff(void* p1, T* p2, char* source) const
{
    const char* c1 = static_cast<char*>(p1);
    const char* c2 = static_cast<char*>(static_cast<void*>(p2));

    return static_cast<T*>(static_cast<void*>(&source[(c2 - c1)]));
}

CPU GPU size_t model_size::total_memory_size() const
{
    return  sizeof(network)
        +   sizeof(void*) * this->network_size
        +   sizeof(node) * this->nodes
        +   sizeof(edge) * this->edges
        +   sizeof(constraint) * this->constraints
        +   sizeof(update) * this->updates
        +   sizeof(expr) * this->expressions
        +   sizeof(clock_var) * this->variables;
}

CPU GPU network* model_oracle::network_point() const
{
    return static_cast<network*>(point);
}

CPU GPU node** model_oracle::network_nodes_point() const
{
    void* p = &network_point()[1];
    return static_cast<node**>(p);
}

CPU GPU node* model_oracle::node_point() const
{
    void* p = &network_nodes_point()[model_counter.network_size];
    return static_cast<node*>(p);
}

CPU GPU edge* model_oracle::edge_point() const
{
    void* p = &node_point()[model_counter.nodes];
    return static_cast<edge*>(p);
}

CPU GPU constraint* model_oracle::constraint_point() const
{
    void* p = &edge_point()[model_counter.edges];
    return static_cast<constraint*>(p);
}

CPU GPU update* model_oracle::update_point() const
{
    void* p = &constraint_point()[model_counter.constraints];
    return static_cast<update*>(p);
}

CPU GPU expr* model_oracle::expression_point() const
{
    void* p = &update_point()[model_counter.updates];
    return static_cast<expr*>(p);
}

CPU GPU clock_var* model_oracle::variable_point() const
{
    void* p = &expression_point()[model_counter.expressions];
    return static_cast<clock_var*>(p);
}


GPU network* model_oracle::move_to_shared_memory(char* shared_mem, const int threads) const
{
    size_t* wide_shared_memory = static_cast<size_t*>(static_cast<void*>(shared_mem));
    const size_t size = this->model_counter.total_memory_size() / sizeof(size_t);

    for (size_t i = 0; i < size; i += threads)
    {
        const size_t idx = i + threadIdx.x;
        if(!(idx < size)) continue;
        wide_shared_memory[idx] = static_cast<size_t*>(this->point)[idx];
    }
    cuda_SYNCTHREADS();

    for (unsigned i = 0; i < this->model_counter.nodes; i += threads)
    {
        const int idx = static_cast<int>(i + threadIdx.x);
        if(idx >= static_cast<int>(this->model_counter.nodes)) continue;

        node* n = get_diff<node>(this->point, &this->node_point()[idx], shared_mem);

        n->edges.store = get_diff<edge>(this->initial_point, n->edges.store, shared_mem);
        n->invariants.store = get_diff<constraint>(this->initial_point, n->invariants.store, shared_mem);
        
        n->lamda = get_diff(this->initial_point, n->lamda, shared_mem);
    }
    cuda_SYNCTHREADS();


    for (unsigned i = 0; i < this->model_counter.edges; i += threads)
    {
        const int idx = static_cast<int>(i + threadIdx.x);
        if(!(idx < static_cast<int>(this->model_counter.edges))) continue;
        
        edge* e = get_diff<edge>(this->point, &this->edge_point()[idx], shared_mem);

        e->dest = get_diff<node>(this->initial_point, e->dest, shared_mem);
        e->guards.store  = get_diff<constraint>(this->initial_point, e->guards.store, shared_mem);
        e->updates.store = get_diff<update>(this->initial_point, e->updates.store, shared_mem);
        e->weight = get_diff<expr>(this->initial_point, e->weight, shared_mem);
    }
    cuda_SYNCTHREADS();

    for (unsigned i = 0; i < this->model_counter.constraints; i += threads)
    {
        const int idx = static_cast<int>(i + threadIdx.x);
        if(!(idx < static_cast<int>(this->model_counter.constraints))) continue;

        constraint* con = get_diff<constraint>(this->point, &this->constraint_point()[idx], shared_mem);

        con->expression = get_diff<expr>(this->initial_point, con->expression, shared_mem);
        if(!con->uses_variable && con->operand != constraint::compiled_c)
            con->value = get_diff<expr>(this->initial_point, con->value, shared_mem);
    }
    cuda_SYNCTHREADS();

    for (unsigned i = 0; i < this->model_counter.updates; i += threads)
    {
        const int idx = static_cast<int>(i + threadIdx.x);
        if(!(idx < static_cast<int>(this->model_counter.updates))) continue;

        update* u = get_diff<update>(this->point, &this->update_point()[idx], shared_mem);
        
        u->expression = get_diff(this->initial_point, u->expression, shared_mem);
    }
    cuda_SYNCTHREADS();

    for (unsigned i = 0; i < this->model_counter.expressions; i += threads)
    {
        const int idx = static_cast<int>(i + threadIdx.x);
        if(!(idx < static_cast<int>(this->model_counter.expressions))) continue;

        expr* ex = get_diff<expr>(this->point, &this->expression_point()[idx], shared_mem);

        if(ex->left != nullptr)
            ex->left = get_diff<expr>(this->initial_point, ex->left, shared_mem);
        if(ex->right != nullptr)
            ex->right = get_diff<expr>(this->initial_point, ex->right, shared_mem);
        if(ex->operand == ex->conditional_ee && ex->conditional_else != nullptr)
            ex->conditional_else = get_diff<expr>(this->initial_point, ex->conditional_else, shared_mem);
    }
    cuda_SYNCTHREADS();

    for (unsigned i = 0; i < this->model_counter.network_size; i += threads)
    {
        const int idx = static_cast<int>(i + threadIdx.x);
        if(idx >= static_cast<int>(this->model_counter.network_size)) continue;

        node** nn = get_diff<node*>(this->point, this->network_nodes_point(), shared_mem);
        nn[idx] = get_diff<node>(this->initial_point, nn[idx], shared_mem);
    }
    cuda_SYNCTHREADS();
    
    if(threadIdx.x == 0)
    {
        network* n = get_diff(this->point, this->network_point(), shared_mem);

        n->automatas.store = get_diff(this->point, this->network_nodes_point(), shared_mem);
        n->variables.store = get_diff(this->point, this->variable_point(), shared_mem);
    }
    cuda_SYNCTHREADS();

    return static_cast<network*>(static_cast<void*>(shared_mem));
}
#pragma once



struct node_results
{
    unsigned reached;
    unsigned total_steps;
    double total_time;

    double avg_steps() const
    {
        if(reached == 0) return 0.0;
        return static_cast<double>(total_steps) / static_cast<double>(reached);
    }

    double avg_time() const
    {
        if(reached == 0) return 0.0;
        return total_time / static_cast<double>(reached);

    }
};

struct variable_result
{
    double total_values;
    double max_value;

    double avg_max_value(const unsigned total_simulations) const
    {
        if(total_simulations == 0) return 0.0;
        return total_values / static_cast<double>(total_simulations);
    }    
};

struct result_pointers
{
private:
    const bool owns_pointers_;
    void* source_p_;
public:
    explicit result_pointers(const bool owns_pointers,
        void* free_p,
        node_results* nodes,
        variable_result* variables,
        int threads,
        unsigned total_simulations);

    node_results* nodes = nullptr;
    variable_result* variables = nullptr;
    int simulations_per_thread = 0;
    int threads = 0;
    unsigned total_simulations = 0;

    unsigned sim_per_thread() const
    {
        return static_cast<unsigned>(ceilf(static_cast<float>(this->total_simulations) / static_cast<float>(threads)));
    }
    
    void free_internals() const;
};

class memory_allocator;


class result_store
{
    friend struct state;
private:
    bool is_cuda_;
    
    unsigned simulations_;
    unsigned node_count_;
    unsigned variables_count_;
    int thread_count_;
    
    
    node_results* node_p_ = nullptr;
    variable_result* variable_p_ = nullptr;
    
    
    size_t total_data_size() const;
    
public:
    explicit result_store(
        unsigned total_sim,
        unsigned variables,
        unsigned node_count,
        int thread_count,
        memory_allocator* helper);

    result_pointers load_results() const;

    //This must be in .h for RDC=false to be used.
    CPU GPU void write_output(const unsigned idx,  const state* sim) const
    {
        const int offset = static_cast<int>(this->node_count_ * idx);
        for (int i = 0; i < sim->models.size; ++i)
        {
            const int index = offset + sim->models.store[i]->id - 1;
            this->node_p_[index].reached++;
            this->node_p_[index].total_steps += sim->steps;
            this->node_p_[index].total_time += sim->global_time;
        }

        const int var_offset = static_cast<int>(this->variables_count_ * idx);
        for (int i = 0, j = 0; i < sim->variables.size; ++i)
        {
            if(!sim->variables.store[i].should_track) continue;
            const int index = var_offset + j++;
            this->variable_p_[index].total_values += sim->variables.store[i].max_value;
            this->variable_p_[index].max_value = fmax(
                this->variable_p_[index].max_value,
                sim->variables.store[i].max_value);
        }
    }
};


CPU GPU size_t thread_heap_size(const sim_config* config)
{
    const size_t size =
          static_cast<size_t>(config->max_expression_depth*2+1) * sizeof(void*) + //this is a expression*, but it doesnt like sizeof(expression*)
          config->max_expression_depth * sizeof(double) +
          config->network_size * sizeof(node) +
          config->variable_count * sizeof(clock_var) +
          config->max_edge_fanout * sizeof(state::w_edge);

    const unsigned long long int padding = (8 - (size % 8));

    return padding < 8 ? size + padding : size;
}  

CPU GPU double determine_progress(const node* node, state* state)
{
    bool is_finite = true;
    const double random_val = hiprand_uniform_double(state->random);
    const double max = node->max_progression(state, &is_finite);
    const double lambda = node->lamda->evaluate_expression(state);

    if(is_finite)
    {
        return (1.0 - random_val) * max;
    }
    else
    {
        // return lambda > 0.0 ? -log(random_val) / (lambda) : lambda;
        return (-log(random_val)) / (lambda - (lambda == 0.0));
    }
}

CPU GPU inline bool can_progress(const node* n)
{
    //#No brackets gang!
    for (int i = 0; i < n->edges.size; ++i)
        if(!IS_LISTENER(n->edges.store[i].channel))
            return true;
    return false;
} 

#define NO_PROCESS (-1)
#define IS_NO_PROCESS(x) ((x) < 0)
CPU GPU int progress_sim(state* sim_state, const sim_config* config)
{
    //determine if sim is done

    // if(config->use_max_steps * sim_state->steps  >= config->max_steps_pr_sim
    //     + !config->use_max_steps * sim_state->global_time >= config->max_global_progression)
    
    if((config->use_max_steps && sim_state->steps  >= config->max_steps_pr_sim)
        || (!config->use_max_steps && sim_state->global_time >= config->max_global_progression) )
            return NO_PROCESS;

    //progress number of steps
    sim_state->steps++;

    // const double max_progression_time = config->use_max_steps
    //                                         ? DBL_MAX
    //                                         : config->max_global_progression - sim_state->global_time;

    const double max_progression_time = ((config->use_max_steps) * DBL_MAX)
                + ((!config->use_max_steps) * (config->max_global_progression - sim_state->global_time));

    double min_progression_time = max_progression_time;
    int winning_process = NO_PROCESS;
    // node** winning_model = nullptr;
    for (int i = 0; i < sim_state->models.size; ++i)
    {
        const node* current = sim_state->models.store[i];
        
        //if goal is reached, dont bother
        if(current->type == node::goal) continue;
        
        //If all channels that are left is listeners, then dont bother
        //This also ensures that current_node has edges
        if(!can_progress(current)) continue;
        
        //if it is not in a valid state, then it is disabled 
        if(!constraint::evaluate_constraint_set(current->invariants, sim_state)) continue;

        
        //determine current models progress
        const double local_progress = determine_progress(current, sim_state);

        // printf("progress %lf\n", local_progress);
        //If negative progression, skip. Represents NO_PROGRESS
        //Set current as winner, if it is the earliest active model.
        if(
            local_progress >= 0.0
            && local_progress < min_progression_time
            && (sim_state->committed_count == 0
                || (sim_state->committed_count > 0
                    && current->type == node::committed)))
        {
            min_progression_time = local_progress;
            winning_process = i;
            // winning_model = &sim_state->models.store[i];
        }
    }
    // printf(" I WON! Node: %d \n", winning_model->current_node->get_id());
    if(min_progression_time < max_progression_time && sim_state->urgent_count == 0)
    {
        for (int i = 0; i < sim_state->variables.size; ++i)
        {
            sim_state->variables.store[i].add_time(min_progression_time);
        }
        sim_state->global_time += min_progression_time;
    }

    return winning_process;
}

CPU GPU edge* pick_next_edge_stack(const arr<edge>& edges, state* state)
{
    state->edge_stack.clear();
    int valid_count = 0;
    state::w_edge valid_edge = {nullptr, 0.0};
    double weight_sum = 0.0;
    
    for (int i = 0; i < edges.size; ++i)
    {
        edge* e = &edges.store[i];
        if(IS_LISTENER(e->channel)) continue;
        if(!constraint::evaluate_constraint_set(e->guards, state)) continue;
        
        const double weight = e->weight->evaluate_expression(state);
        //only consider edge if it its weight is positive.
        //Negative edge value is semantically equivalent to disabled.
        if(weight <= 0.0) continue;
        valid_edge = state::w_edge{ e, weight };
        valid_count++;
        weight_sum += weight;
        state->edge_stack.push(valid_edge);
    }

    if(valid_count == 0) return nullptr;
    if(valid_count == 1) return valid_edge.e;

    const double r_val = (1.0 - hiprand_uniform_double(state->random)) * weight_sum;
    double r_acc = 0.0;

    //pick the weighted random value.
    valid_edge = { nullptr, 0.0 }; //reset valid edge !IMPORTANT
    for (int i = 0; i < valid_count; ++i)
    {
        valid_edge = state->edge_stack.pop();
        r_acc += valid_edge.w;
        if(r_val < r_acc) break;
    }

    return valid_edge.e;
}

CPU GPU void simulate_automata(
    const unsigned idx,
    const network* model,
    const result_store* output,
    const sim_config* config)
{
    void* cache = static_cast<void*>(&static_cast<char*>(config->cache)[(idx*thread_heap_size(config)) / sizeof(char)]);
    hiprandState* r_state = &config->random_state_arr[idx];
    hiprand_init(config->seed, idx, idx, r_state);
    state sim_state = state::init(cache, r_state, model, config->max_expression_depth, config->max_edge_fanout);
    
    for (unsigned i = 0; i < config->simulation_amount; ++i)
    {
        const unsigned int sim_id = i + config->simulation_amount * static_cast<unsigned int>(idx);
        sim_state.reset(sim_id, model);
        
        //run simulation
        while (true)
        {
            const int process = progress_sim(&sim_state, config);
            if(IS_NO_PROCESS(process)) break;
            
            do
            {
                const node* current = sim_state.models.store[process];
                const edge* e = pick_next_edge_stack(current->edges, &sim_state);
                if(e == nullptr) break;
                
                sim_state.traverse_edge(process, e->dest);
                e->apply_updates(&sim_state);
                sim_state.broadcast_channel(e->channel, process);
            } while (sim_state.models.store[process]->type == node::branch);
        }
        output->write_output(idx, &sim_state);
    }
}

__global__ void simulator_gpu_kernel(
    const model_oracle* oracle,
    const result_store* output,
    const sim_config* config)
{
    // ReSharper disable once CppTooWideScope
    extern __shared__ char shared_mem[];
    const unsigned long idx = threadIdx.x + blockDim.x * blockIdx.x;
    
    network* model;
    if(config->use_shared_memory)
    {
        model = oracle->move_to_shared_memory(shared_mem, static_cast<int>(config->threads));
    }
    else
    {
        model = oracle->network_point();
    }
    cuda_SYNCTHREADS();

    simulate_automata(idx, model, output, config);
}