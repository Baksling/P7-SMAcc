#include "hip/hip_runtime.h"
﻿#include "cuda_simulator.h"

#define GPU __device__
#define CPU __host__
#define NOT_GOAL_STATE -1
#include "uneven_list.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include ""
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <chrono>
#include <iostream>
#include <unordered_map>

#include "stochastic_model.h"

using namespace std::chrono;
using namespace std;




GPU bool validate_guards(const array_info<guard_d>* guards, const array_info<timer_d>* timers)
{
    for (int j = 0; j < guards->size; j++)
    {
        //get timer required by guard.
        const int timer_id = guards->arr[j].get_timer_id();
        //validate guard using required timer.
        if(guards->arr[j].validate(timers->arr[timer_id].get_value())) continue;

        //if validate fails, return false. Also breaks loop
        return false;
    }
    return true;
}

GPU array_info<edge_d> validate_edges(const array_info<edge_d>* edges, const stochastic_model* model, const array_info<timer_d>* timers)
{
    //count of valid edges
    int validated_i = 0;
    
    //buffer of all possible valid edges using validated_i
    const auto valid_edges = static_cast<edge_d*>(malloc(sizeof(edge_d) * edges->size)); 

    //go through all edges from current node.
    for(int i = 0; i < edges->size; i++)
    {
        //find all guards of current edge and validate its guards
        array_info<guard_d> guards = model->get_edge_guards(edges->arr[i].get_id());
        bool validated =  validate_guards(&guards, timers);
        guards.free_arr();

        //only continue if all guards are valid.
        if(!validated) continue;

        //check all guards of destination node.
        guards = model->get_node_invariants(edges->arr[i].get_dest_node());
        validated =  validate_guards(&guards, timers);
        guards.free_arr();

        //add to valid_edges if both check succeed.
        if (validated)
        {
            valid_edges[validated_i] = edges->arr[i];
            validated_i++;
        }
    }

    //copy all valid edges into appropriate sized array.
    const auto result_arr = static_cast<edge_d*>(malloc(sizeof(edge_d) * validated_i));
    for (int i = 0; i < validated_i; i++)
    {
        result_arr[i] = valid_edges[i];
    }
    //free old buffer
    free(valid_edges);

    const array_info<edge_d> result { result_arr, validated_i};
    return result;
}

GPU edge_d* choose_next_edge(const array_info<edge_d>* edges, hiprandState* states, const unsigned int thread_id)
{
    //if no possible edges, return null pointer
    if(edges->size == 0) return nullptr;

    //summed weight
    float weight_sum = 0.0f;
    for(int i = 0; i < edges->size; i++)
        weight_sum += edges->arr[i].get_weight();

    //hiprand_uniform return ]0.0f, 1.0f], but we require [0.0f, 1.0f[
    //conversion from float to int is floored, such that a array of 10 (index 0..9) will return valid index.
    const float r_val = (1.0f - hiprand_uniform(&states[thread_id]))*weight_sum;
    float r_acc = 0.0; 

    //pick the weighted random value.
    for (int i = 0; i < edges->size; ++i)
    {
        r_acc += edges->arr[i].get_weight();
        if(r_val < r_acc) return &edges->arr[i];
    }

    //This should be handled in for loop.
    //This is for safety :)
    return &edges->arr[edges->size - 1];
}

GPU void progress_time(const array_info<timer_d>* timers, const double difference, hiprandState* states, const unsigned int thread_id)
{
    //Get random uniform value between ]0.0f, 0.1f] * difference gives a random uniform range of ]0, diff]
    const double time_progression = difference * hiprand_uniform_double(&states[thread_id]);

    //update all timers by adding time_progression to each
    for(int i = 0; i < timers->size; i++)
    {
        timers->arr[i].add_time(time_progression);
    }
}

//Finds the furthest possible time it is possible to progress in current step.
GPU double find_least_difference(const array_info<guard_d>* invariants, const array_info<timer_d>* timers,
    const int max_value = 100)
{
    double least_difference = max_value;
    

    //check all guards of current node
    for (int i = 0; i < invariants->size; i++)
    {
        const logical_operator guard_type = invariants->arr[i].get_type();
        //only relevant if it is upper bounded logical operator.
        if(guard_type != logical_operator::less_equal && guard_type != logical_operator::less) continue;

        //find difference in upper bounded guard value and current time.
        const double diff = invariants->arr[i].get_value() - timers->arr[invariants->arr[i].get_timer_id()].get_value();
        //if equal or higher than 0 and its smallest value, find newest lower bound.
        if (diff >= 0 && diff < least_difference)
            least_difference = diff;
    }

    //free index.
    return least_difference;
}

struct model_options
{
    int simulation_amount;
    int max_steps_pr_sim;
    unsigned long seed;
};

__global__ void simulate_d_2(
    const stochastic_model* model,
    const model_options* options,
    hiprandState* r_state,
    int* output
    )
{
    // printf("HELOL\n");
    // model->pretty_print();
    // printf("HELOLo");
    //init variables and random state
    const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    hiprand_init(options->seed, idx, idx, &r_state[idx]);

    // init local timers.
    const array_info<timer_d> internal_timers = model->copy_timers();

    for (int i = 0; i < options->simulation_amount; i++)
    {
        //reset current location
        const int sim_id = i + options->simulation_amount * static_cast<int>(idx);
        output[sim_id] = NOT_GOAL_STATE;
        
        //reset timers through each simulation
        model->reset_timers(&internal_timers);
        
        int current_node = model->get_start_node();
        int steps = 0;

        while (true)
        {
            if(steps >= options->max_steps_pr_sim)
            {
                break;
            }
            steps++;

            const array_info<guard_d> invariants = model->get_node_invariants(current_node);
            if (!validate_guards(&invariants, &internal_timers))
            {
                invariants.free_arr();
                break;
            }

            if (!model->is_branchpoint(current_node))
            {
                const double difference = find_least_difference(&invariants, &internal_timers);
                progress_time(&internal_timers, difference, r_state, idx);
            }
            
            invariants.free_arr();
            
            const array_info<edge_d> edges = model->get_node_edges(current_node);
            if (edges.size <= 0) //current edge has no outgoing edges.
            {
                edges.free_arr();
                break;
            }

            const array_info<edge_d> valid_edges = validate_edges(&edges, model, &internal_timers);
            edge_d* edge = choose_next_edge(&valid_edges, r_state, idx);
            
            if(edge == nullptr) //no traversal can be done in current step
            {
                // printf("Stopped at node: %d \n", current_node);
                edges.free_arr();
                valid_edges.free_arr();
                continue;
            }

            model->traverse_edge_update(edge->get_id(), &internal_timers);
            current_node = edge->get_dest_node();
            edges.free_arr();
            valid_edges.free_arr();

            if(model->is_goal_node(current_node))
            {
                break;
            }
        }

        //record final location
        output[sim_id] = current_node;
    }

    internal_timers.free_arr();
}

cuda_simulator::cuda_simulator() = default;

void copy_to_device(void* dest, const void* src, const int size)
{
    hipMalloc(((void**)&dest), size);
    hipMemcpy(dest, src, size, hipMemcpyHostToDevice);
}

float calc_percentage(const int counter, const int divisor)
{
    return (static_cast<float>(counter)/static_cast<float>(divisor))*100;
} 

void print_results(unordered_map<int,int>* result_map, const int result_size)
{
    for (auto& it : (*result_map))
    {
        if(it.first == NOT_GOAL_STATE) continue;
        const float percentage = calc_percentage(it.second, result_size);
        cout << "Node: " << it.first << " reached " << it.second << " times. (" << percentage << ")%\n";
    }
    const float percentage = calc_percentage((*result_map)[NOT_GOAL_STATE], result_size);
    cout << "No goal state was reached " << (*result_map)[NOT_GOAL_STATE] << " times. (" << percentage << ")%\n";
    cout << "Nr of simulations: " << result_size << "\n";
}

void count_results(const int total_simulations,
    const int* local_results,
    unordered_map<int, int>* node_results)
{
    node_results->insert_or_assign(NOT_GOAL_STATE, 0);
    for (int i = 0; i < total_simulations; i++)
    {
        const int key = local_results[i];
        const int value = node_results->count(key) == 1
                              ? (*node_results)[key]+1
                              : 1;
        node_results->insert_or_assign(key, value);
    }
}


//Expects all parameters to be on the host.
void cuda_simulator::simulate(const stochastic_model* model, const simulation_strategy* strategy, list<void*>* free_list) const
{
    const steady_clock::time_point start = steady_clock::now();

    const int total_simulations = strategy->total_simulations();
    
    hiprandState* state;
    hipMalloc(&state, sizeof(hiprandState) * strategy->parallel_degree * strategy->threads_n);
    
    int* results = nullptr;
    hipMalloc(&results, sizeof(int)*total_simulations);
    
    //move model to device
    
    stochastic_model* model_d = nullptr;
    model->cuda_allocate(&model_d, free_list);
    //move options to device
    model_options* options_d = nullptr;
    const model_options options = {
        strategy->simulation_amounts,
        strategy->max_sim_steps,
        static_cast<unsigned long>(time(nullptr))
    };
    hipMalloc(&options_d, sizeof(model_options));
    hipMemcpy(options_d, &options, sizeof(model_options), hipMemcpyHostToDevice);
    
    //run simulations
    simulate_d_2<<<strategy->parallel_degree, strategy->threads_n>>>(
        model_d, options_d, state, results);

    //wait for all processes to finish
    hipDeviceSynchronize();

    //report time spent simulating
    cout << "I ran for: " << duration_cast<milliseconds>(steady_clock::now() - start).count() << "[ms] \n";

    //copy results from device to host
    int* local_results = static_cast<int*>(malloc(sizeof(int)*total_simulations));
    hipMemcpy(local_results, results, sizeof(int)*total_simulations, hipMemcpyDeviceToHost);

    //count the results
    unordered_map<int, int> node_results;
    count_results(total_simulations, local_results, &node_results);

    print_results(&node_results, total_simulations);

    //free heap allocated and cuda memory
    //model is not handled here, as it is added to the free list.
    //only the local instantiated memory is freed here.
    free(local_results);
    hipFree(results);
    hipFree(options_d);
    hipFree(state);
}




