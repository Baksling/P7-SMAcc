#include "hip/hip_runtime.h"
﻿#include "cuda_simulator.h"

#define GPU __device__
#define CPU __host__
#define NOT_GOAL_STATE -1
#include "uneven_list.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include ""
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <list>
#include <stdio.h>
#include <math.h>
#include <chrono>
#include <iostream>
#include <map>
#include <unordered_map>

#include "stochastic_model.h"

using namespace std::chrono;
using namespace std;




GPU bool validate_guards(const array_info<guard_d>* guards, const array_info<timer_d>* timers)
{
    for (int j = 0; j < guards->size; j++)
    {
        //get timer required by guard.
        const int timer_id = guards->arr[j].get_timer_id();
        //validate guard using required timer.
        if(guards->arr[j].validate(timers->arr[timer_id].get_value())) continue;

        //if validate fails, return false. Also breaks loop
        return false;
    }
    return true;
}

GPU array_info<edge_d> validate_edges(const array_info<edge_d>* edges, const stochastic_model* model, const array_info<timer_d>* timers)
{
    //count of valid edges
    int validated_i = 0;
    
    //buffer of all possible valid edges using validated_i
    const auto valid_edges = static_cast<edge_d*>(malloc(sizeof(edge_d) * edges->size)); 

    //go through all edges from current node.
    for(int i = 0; i < edges->size; i++)
    {
        //find all guards of current edge and validate its guards
        array_info<guard_d> guards = model->get_edge_guards(edges->arr[i].get_id());
        bool validated =  validate_guards(&guards, timers);
        guards.free_arr();

        //only continue if all guards are valid.
        if(!validated) continue;

        //check all guards of destination node.
        guards = model->get_node_invariants(edges->arr[i].get_dest_node());
        validated =  validate_guards(&guards, timers);
        guards.free_arr();

        //add to valid_edges if both check succeed.
        if (validated)
        {
            valid_edges[validated_i] = edges->arr[i];
            validated_i++;
        }
    }

    //copy all valid edges into appropriate sized array.
    const auto result_arr = static_cast<edge_d*>(malloc(sizeof(edge_d) * validated_i));
    for (int i = 0; i < validated_i; i++)
    {
        result_arr[i] = valid_edges[i];
    }
    //free old buffer
    free(valid_edges);

    const array_info<edge_d> result { result_arr, validated_i};
    return result;
}

GPU edge_d* choose_next_edge(const array_info<edge_d>* edges, hiprandState* states, const unsigned int thread_id)
{
    //if no possible edges, return null pointer
    if(edges->size == 0) return nullptr;

    //hiprand_uniform return ]0.0f, 1.0f], but we require [0.0f, 1.0f[
    //conversion from float to int is floored, such that a array of 10 (index 0..9) will return valid index.
    float weight_sum = 0.0f;
    for(int i = 0; i < edges->size; i++)
        weight_sum += edges->arr[i].get_weight();

    
    const float r_val = (1.0f - hiprand_uniform(&states[thread_id]))*weight_sum;
    float r_acc = 0.0; 
    
    for (int i = 0; i < edges->size; ++i)
    {
        r_acc += edges->arr[i].get_weight();
        if(r_val < r_acc) return &edges->arr[i];
    }

    //This should be handled in for loop.
    //This is for safety :)
    return &edges->arr[edges->size - 1];
}

GPU void progress_time(const array_info<timer_d>* timers, const double difference, hiprandState* states, const unsigned int thread_id)
{
    //Get random uniform value between ]0.0f, 0.1f] * difference gives a random uniform range of ]0, diff]
    const double time_progression = difference * hiprand_uniform_double(&states[thread_id]);

    //update all timers by adding time_progression to each
    for(int i = 0; i < timers->size; i++)
    {
        timers->arr[i].add_time(time_progression);
    }
}

//Finds the furthest possible time it is possible to progress in current step.
GPU double find_least_difference(const array_info<guard_d>* invariants, const array_info<timer_d>* timers,
    const int max_value = 100)
{
    double least_difference = max_value;
    

    //check all guards of current node
    for (int i = 0; i < invariants->size; i++)
    {
        const logical_operator guard_type = invariants->arr[i].get_type();
        //only relevant if it is upper bounded logical operator.
        if(guard_type != logical_operator::less_equal && guard_type != logical_operator::less) continue;

        //find difference in upper bounded guard value and current time.
        const double diff = invariants->arr[i].get_value() - timers->arr[invariants->arr[i].get_timer_id()].get_value();
        //if equal or higher than 0 and its smallest value, find newest lower bound.
        if (diff >= 0 && diff < least_difference)
            least_difference = diff;
    }

    //free index.
    return least_difference;
}

GPU void reset_timers(const array_info<timer_d>* timers, const array_info<timer_d>* original_time)
{
    
}


struct model_options
{
    int simulation_amount;
    int max_steps_pr_sim;
    unsigned long seed;
};

__global__ void simulate_d_2(
    const stochastic_model* model,
    const model_options* options,
    hiprandState* r_state,
    int* output
    )
{
    //init variables and random state
    const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    hiprand_init(options->seed, idx, idx, &r_state[idx]);

    // init local timers.
    const array_info<timer_d> internal_timers = model->copy_timers();

    for (int i = 0; i < options->simulation_amount; i++)
    {
        //reset current location
        const int sim_id = i + options->simulation_amount * static_cast<int>(idx);
        output[sim_id] = NOT_GOAL_STATE;
        
        //reset timers through each simulation
        model->reset_timers(&internal_timers);
        
        int current_node = model->get_start_node();
        int steps = 0;

        while (true)
        {
            if(steps >= options->max_steps_pr_sim)
            {
                output[sim_id] = NOT_GOAL_STATE;
                break;
            }
            steps++;

            const array_info<guard_d> invariants = model->get_node_invariants(current_node);
            if (!validate_guards(&invariants, &internal_timers))
            {
                invariants.free_arr();
                output[sim_id] = NOT_GOAL_STATE;
                break;
            }

            
            const double difference = find_least_difference(&invariants, &internal_timers);
            progress_time(&internal_timers, difference, r_state, idx);
            invariants.free_arr();
            
            const array_info<edge_d> edges = model->get_node_edges(current_node);
            if (edges.size <= 0)
            {
                edges.free_arr();
                continue;
            }

            const array_info<edge_d> valid_edges = validate_edges(&edges, model, &internal_timers);
            edge_d* edge = choose_next_edge(&valid_edges, r_state, idx);
            
            if(edge == nullptr)
            {
                // printf("Stopped at node: %d \n", current_node);
                edges.free_arr();
                valid_edges.free_arr();
                continue;
            }
            
            current_node = edge->get_dest_node();
            edges.free_arr();
            valid_edges.free_arr();

            if(model->is_goal_node(current_node))
            {
                output[sim_id] = current_node;
                break;
            }
        }
    }

    internal_timers.free_arr();
}

cuda_simulator::cuda_simulator()
{
}

void copy_to_device(void* dest, const void* src, const int size)
{
    hipMalloc(((void**)&dest), size);
    hipMemcpy(dest, src, size, hipMemcpyHostToDevice);
}

float calc_percentage(const int counter, const int divisor)
{
    return (static_cast<float>(counter)/static_cast<float>(divisor))*100;
} 

void print_results(map<int,int>* result_map, const int result_size)
{
    for (auto& it : (*result_map))
    {
        if(it.first == NOT_GOAL_STATE) continue;
        const float percentage = calc_percentage(it.second, result_size);
        cout << "Node: " << it.first << " reached " << it.second << " times. (" << percentage << ")%\n";
    }
    const float percentage = calc_percentage((*result_map)[NOT_GOAL_STATE], result_size);
    cout << "No goal state was reached " << (*result_map)[NOT_GOAL_STATE] << " times. (" << percentage << ")%\n";
    cout << "Nr of simulations: " << result_size << "\n";
}

void cuda_simulator::simulate_2(uneven_list<edge_d> *node_to_edge, uneven_list<guard_d> *node_to_invariant,
    uneven_list<guard_d> *edge_to_guard, uneven_list<update_d> *edge_to_update, int timer_amount, timer_d *timers) const
{
    const steady_clock::time_point start = steady_clock::now();

    constexpr int parallel_degree = 32;
    constexpr int threads_n = 80;
    constexpr int simulation_amounts = 100;
    // constexpr int sim_count = 1;

    const int result_size = parallel_degree*threads_n*simulation_amounts;
    hiprandState* state;
    hipMalloc(&state, sizeof(hiprandState)*parallel_degree*threads_n);

    
    int* results = nullptr;
    int* local_results = static_cast<int*>(malloc(sizeof(int)*result_size));
    if(local_results == nullptr)
        throw exception();
    if(hipMalloc(&results, sizeof(int)*result_size) != hipSuccess)
        throw exception();
    hipMemcpy(results, local_results, sizeof(int)*result_size, hipMemcpyHostToDevice);
    
    //move model to decive
    stochastic_model* model_d = nullptr;
    const stochastic_model model(node_to_edge, node_to_invariant, edge_to_guard,
        edge_to_update, timers, timer_amount);
    model.cuda_allocate(&model_d);

    //move options to device
    model_options* options_d = nullptr;
    const model_options options = { simulation_amounts,20000, static_cast<unsigned long>(time(nullptr)) };
    hipMalloc(&options_d, sizeof(model_options));
    hipMemcpy(options_d, &options, sizeof(model_options), hipMemcpyHostToDevice);
    
    //run simulations
    simulate_d_2<<<parallel_degree, threads_n>>>(model_d, options_d, state, results);
    hipDeviceSynchronize();

    cout << "I ran for: " << duration_cast<milliseconds>(steady_clock::now() - start).count() << "[ms] \n";

    hipMemcpy(local_results, results, sizeof(int)*result_size, hipMemcpyDeviceToHost);

    map<int, int> node_results = map<int,int>();
    node_results.insert_or_assign(NOT_GOAL_STATE, 0);
    for (int i = 0; i < result_size; i++)
    {
        const int key = local_results[i];
        const int value = node_results.count(key) == 1
            ? node_results[key]+1
            : 1;
        node_results.insert_or_assign(key, value);
    }
    
    print_results(&node_results, result_size);

    
    free(local_results);
    hipFree(results);
    hipFree(options_d);
    hipFree(model_d);
    hipFree(state);
    return;
}




