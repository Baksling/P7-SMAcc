﻿#include "main.h"
#define GPU __device__
#define CPU __host__
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <list>
#include <stdio.h>

#include "uneven_list.h"
#include "node_d.h"
#include "edge_d.h"
#include "guard_d.h"
#include "update_d.h"
#include "timer_d.h"
#include "cuda_simulator.h"

int main()
{
    // node_d nodes[2] = {node_d(1), node_d(2)};
    // edge_d edges[2] = {edge_d(1, 2), edge_d(2, 1)};
    // guard_d guards[1] = {guard_d(1, logical_operator::greater_equal, 10)};
    // update_d updates[1] = {update_d(1, 0)};
    // timer_d timers[1] = {timer_d(1,0)};
    //
    // array_info<node_d> n {nodes, 2};
    // array_info<edge_d> e {edges, 2};
    // array_info<guard_d> g {guards, 1};
    // array_info<update_d> u {updates, 1};
    // array_info<timer_d> t {timers, 1};
    //
    // cuda_simulator sim(&n, &e, &g, &u, &t);
    // sim.simulate(10);

    // node = [node]
    // edge = [[edge]], hvor index af første = node index
    // invarient = [[guard]], hvor index af først = node index
    // guard = [guard], hvor index = edge id
    // update = [update], hvor index = edge id
    // timer = [timer], index = id.

    // -----------------------------------------------------------

    //Nodes
    list<node_d> nodes__;
    nodes__.emplace_back(0);
    nodes__.emplace_back(1);
    nodes__.emplace_back(2);

    // Edges
    list<edge_d> edges_1_;
    edges_1_.emplace_back(0, 1);
    edges_1_.emplace_back(0,2);

    list<edge_d> edges_2_;
    edges_2_.emplace_back(1, 2);

    list<edge_d> edges_3_;

    list<list<edge_d>> edge_list;
    edge_list.push_back(edges_1_);
    edge_list.push_back(edges_2_);
    edge_list.push_back(edges_3_);

    //Invariants for nodes
    list<guard_d> invariant_1_;
    invariant_1_.emplace_back(0, logical_operator::less, 10);

    list<guard_d> invariant_2_;
    invariant_2_.emplace_back(0, logical_operator::less_equal, 10);

    list<guard_d> invariant_3_;
    invariant_3_.emplace_back(0, logical_operator::greater_equal, 10);

    list<list<guard_d>> invariant_list;
    invariant_list.push_back(invariant_1_);
    invariant_list.push_back(invariant_2_);
    invariant_list.push_back(invariant_3_);

    // Guard List for edges
    list<guard_d> guard_1_;
    guard_1_.emplace_back(0, logical_operator::less, 10);

    list<guard_d> guard_2_;
    guard_2_.emplace_back(0, logical_operator::less_equal, 10);

    list<guard_d> guard_3_;
    guard_3_.emplace_back(0, logical_operator::greater_equal, 10);

    list<list<guard_d>> guard_list;
    guard_list.push_back(invariant_1_);
    guard_list.push_back(invariant_2_);
    guard_list.push_back(invariant_3_);

    //Update list for edges
    list<update_d> update_1_;
    update_1_.emplace_back(0, 0);
    update_1_.emplace_back(1,0);
    
    list<update_d> update_2_;
    update_2_.emplace_back(0, 0);

    list<update_d> update_3_;

    list<list<update_d>> update_list;
    update_list.push_back(update_1_);
    update_list.push_back(update_2_);
    update_list.push_back(update_3_);

    // Timers
    timer_d* timer_list;
    timer_list = (timer_d*)malloc(sizeof(timer_d) * 2);
    timer_list[0] = timer_d(0, 0);
    timer_list[1] = timer_d(1, 0);
    
    uneven_list<edge_d> node_to_edge(&edge_list, 3);
    uneven_list<guard_d> node_to_invariant(&invariant_list, 3);
    uneven_list<guard_d> edge_to_guard(&guard_list, 3);
    uneven_list<update_d> edge_to_update(&update_list, 3);

    // NOW ALLOCATE MEMORY ON DEVICE FOR ALL THIS SHIT!

    uneven_list<edge_d> *node_to_edge_d;
    uneven_list<guard_d> *node_to_invariant_d;
    uneven_list<guard_d> *edge_to_guard_d;
    uneven_list<update_d> *edge_to_update_d;

    timer_d* timers_d;
    
    hipMalloc((void**)&node_to_edge_d, sizeof(uneven_list<edge_d>));
    hipMalloc((void**)&node_to_invariant_d, sizeof(uneven_list<guard_d>));
    hipMalloc((void**)&edge_to_guard_d, sizeof(uneven_list<guard_d>));
    hipMalloc((void**)&edge_to_update_d, sizeof(uneven_list<update_d>));
    hipMalloc((void**)&timers_d, sizeof(timer_d) * 2);

    // Copy memory to device
    node_to_edge.allocate_memory();
    node_to_invariant.allocate_memory();
    edge_to_guard.allocate_memory();
    edge_to_update.allocate_memory();

    hipMemcpy(node_to_edge_d, &node_to_edge, sizeof(uneven_list<edge_d>), hipMemcpyHostToDevice);
    hipMemcpy(node_to_invariant_d, &node_to_invariant, sizeof(uneven_list<guard_d>), hipMemcpyHostToDevice);
    hipMemcpy(edge_to_guard_d, &edge_to_guard, sizeof(uneven_list<guard_d>), hipMemcpyHostToDevice);
    hipMemcpy(edge_to_update_d, &edge_to_update, sizeof(uneven_list<update_d>), hipMemcpyHostToDevice);
    hipMemcpy(timers_d, timer_list, sizeof(timer_d) * 2, hipMemcpyHostToDevice);

    //printf("yasss girl: %d %d %d %d\n", node_to_edge.max_elements_, node_to_edge.max_index_, node_to_edge_d->max_elements_, node_to_edge_d->max_index_);
    
    cuda_simulator sim;
    sim.simulate_2(node_to_edge_d, node_to_invariant_d, edge_to_guard_d, edge_to_update_d, timers_d);
    
    
    // array_info<guard_d> hej = node_to_invariant.get_index(0);
    //
    // for(int i = 0; i < hej.size; i++) {
    //     printf("%d -> %d", hej.arr[i].get_timer_id(), (int)hej.arr[i].get_value());
    // }

    return 0;
}


