﻿#include "stochastic_model.h"

#include <assert.h>
#include <ctime>

using namespace std;

stochastic_model::stochastic_model(uneven_list<edge_d>* node_to_edge, uneven_list<guard_d>* node_to_invariant,
                                   uneven_list<guard_d>* edge_to_guard, uneven_list<update_d>* edge_to_update,
                                   timer_d* timers, const int timer_count)
{
    this->timer_count_ = timer_count;
    this->timers_ = timers;
    this->node_to_edge_ = node_to_edge;
    this->node_to_invariant_ = node_to_invariant;
    this->edge_to_guard_ = edge_to_guard;
    this->edge_to_update_ = edge_to_update;
}


GPU array_info<edge_d> stochastic_model::get_node_edges(const int node_id) const
{
    return this->node_to_edge_->get_index(node_id);
}

GPU array_info<guard_d> stochastic_model::get_node_invariants(const int node_id) const
{
    return this->node_to_invariant_->get_index(node_id);

}

GPU array_info<guard_d> stochastic_model::get_edge_guards(const int edge_id) const
{
    return this->edge_to_guard_->get_index(edge_id);

}

GPU array_info<update_d> stochastic_model::get_updates(const int edge_id) const
{
    return this->edge_to_update_->get_index(edge_id);
}


GPU void stochastic_model::traverse_edge_update(const int edge_id, const array_info<timer_d>* local_timers) const
{
    const array_info<update_d> updates = this->get_updates(edge_id);

    for (int i = 0; i < updates.size; ++i)
    {
        update_d update = updates.arr[i];
        const int timer_id = update.get_timer_id();
        timer_d timer = local_timers->arr[timer_id];

        timer.set_time(update.get_value());
    }
    
    updates.free_arr();
}

GPU int stochastic_model::get_start_node() const
{
    return 0;
}

GPU bool stochastic_model::is_goal_node(int node_id) const
{
    return node_id == 2;
}

GPU array_info<timer_d> stochastic_model::copy_timers() const
{
    const int size = this->timer_count_;
    timer_d* internal_timers_arr = static_cast<timer_d*>(malloc(sizeof(timer_d) * size));
    const array_info<timer_d> internal_timers{ internal_timers_arr, size};
    
    for (int i = 0; i < internal_timers.size; i++)
    {
        internal_timers.arr[i] = this->timers_[i].copy();
    }
    
    return internal_timers;
}

GPU void stochastic_model::reset_timers(const array_info<timer_d>* timers) const
{
    assert(timers->size == this->timer_count_);
    for (int i = 0; i < timers->size; i++)
    {
        timers->arr[i].set_time(this->timers_[i].get_value());
    }
    
}

void stochastic_model::cuda_allocate(stochastic_model** p) const
{
    hipMalloc(p, sizeof(stochastic_model));
    hipMemcpy(*p, this, sizeof(stochastic_model), hipMemcpyHostToDevice);
}
