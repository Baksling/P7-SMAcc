﻿#include "stochastic_model.h"
#include <assert.h>
#include <ctime>
#include <map>
#include <string>

using namespace std;

stochastic_model::stochastic_model(uneven_list<edge_d>* node_to_edge, uneven_list<guard_d>* node_to_invariant,
                                   uneven_list<guard_d>* edge_to_guard, uneven_list<update_d>* edge_to_update,
                                   timer_d* timers, int* branchpoint_nodes, int branchpoint_count, const int timer_count)
{
    this->timer_count_ = timer_count;
    this->timers_ = timers;
    this->node_to_edge_ = node_to_edge;
    this->node_to_invariant_ = node_to_invariant;
    this->edge_to_guard_ = edge_to_guard;
    this->edge_to_update_ = edge_to_update;
    this->branchpoint_nodes_ = branchpoint_nodes;
    this->branchpoint_count_= branchpoint_count;
}


GPU array_info<edge_d> stochastic_model::get_node_edges(const int node_id) const
{
    return this->node_to_edge_->get_index(node_id);
}

GPU array_info<guard_d> stochastic_model::get_node_invariants(const int node_id) const
{
    return this->node_to_invariant_->get_index(node_id);
}

GPU array_info<guard_d> stochastic_model::get_edge_guards(const int edge_id) const
{
    return this->edge_to_guard_->get_index(edge_id);

}

GPU array_info<update_d> stochastic_model::get_updates(const int edge_id) const
{
    return this->edge_to_update_->get_index(edge_id);
}

GPU bool stochastic_model::is_branchpoint(int node_id) const
{
    for (int i = 0; i<branchpoint_count_;i++)
    {
        if (node_id == branchpoint_nodes_[i])
            return true;
    }
    return false;
}


GPU void stochastic_model::traverse_edge_update(const int edge_id, const array_info<timer_d>* local_timers) const
{
    const array_info<update_d> updates = this->get_updates(edge_id);

    for (int i = 0; i < updates.size; ++i)
    {
        update_d* update = &updates.arr[i];
        const int timer_id = update->get_timer_id();
        timer_d* timer = &local_timers->arr[timer_id];

        timer->set_time(update->get_value());
    }
    
    updates.free_arr();
}

GPU int stochastic_model::get_start_node() const
{
    return 0;
}

GPU bool stochastic_model::is_goal_node(int node_id) const
{
    return node_id == 3;
}

GPU array_info<timer_d> stochastic_model::copy_timers() const
{
    const int size = this->timer_count_;
    timer_d* internal_timers_arr = static_cast<timer_d*>(malloc(sizeof(timer_d) * size));
    const array_info<timer_d> internal_timers{ internal_timers_arr, size};
    
    for (int i = 0; i < internal_timers.size; i++)
    {
        internal_timers.arr[i] = this->timers_[i].copy();
    }
    
    return internal_timers;
}

GPU void stochastic_model::reset_timers(const array_info<timer_d>* timers) const
{
    assert(timers->size == this->timer_count_);
    for (int i = 0; i < timers->size; i++)
    {
        timers->arr[i].set_time(this->timers_[i].get_value());
    }
    
}

void stochastic_model::cuda_allocate(stochastic_model** p, list<void*>* free_list) const
{
    //move internal lists to cuda    
    uneven_list<edge_d>* node_to_edge_d = nullptr;
    this->node_to_edge_->cuda_allocate(&node_to_edge_d, free_list);
    
    uneven_list<guard_d>* node_to_invariant_d = nullptr;
    this->node_to_invariant_->cuda_allocate(&node_to_invariant_d, free_list);

    uneven_list<guard_d>* edge_to_guard_d = nullptr;
    this->edge_to_guard_->cuda_allocate(&edge_to_guard_d, free_list);

    uneven_list<update_d>* edge_to_update_d = nullptr;
    this->edge_to_update_->cuda_allocate(&edge_to_update_d, free_list);

    //move timers to cuda
    timer_d* timers_d = nullptr;
    hipMalloc(&timers_d, sizeof(timer_d)*this->timer_count_);
    free_list->push_back(timers_d);
    hipMemcpy(timers_d, this->timers_, sizeof(timer_d)*this->timer_count_, hipMemcpyHostToDevice);

    int* branchpoint_nodes_d = nullptr;
    hipMalloc(&branchpoint_nodes_d, sizeof(int)*this->branchpoint_count_);
    free_list->push_back(branchpoint_nodes_d);
    hipMemcpy(branchpoint_nodes_d, this->branchpoint_nodes_, sizeof(int)*this->branchpoint_count_,hipMemcpyHostToDevice);

    //create model with cuda pointers
    const stochastic_model model = stochastic_model(
        node_to_edge_d, node_to_invariant_d,
        edge_to_guard_d, edge_to_update_d,
        timers_d, branchpoint_nodes_d, this->branchpoint_count_, this->timer_count_);

    //move model with cuda pointers to device. Add to free list.
    hipMalloc(p, sizeof(stochastic_model));
    free_list->push_back((*p));
    hipMemcpy((*p), &model, sizeof(stochastic_model), hipMemcpyHostToDevice);
}

CPU GPU void stochastic_model::pretty_print() const
{
    // std::map<logical_operator, string> logical_guys;
    // logical_guys.insert_or_assign(logical_operator::equal,"==");
    // logical_guys.insert_or_assign(logical_operator::greater, ">");
    // logical_guys.insert_or_assign(logical_operator::greater_equal, ">=");
    // logical_guys.insert_or_assign(logical_operator::less_equal, "<=");
    // logical_guys.insert_or_assign(logical_operator::less, "<");
    // logical_guys.insert_or_assign(logical_operator::not_equal, "!=");
    //
    //
    printf("Branchpoint nodes: ");
    for (int b_nodes = 0; b_nodes<branchpoint_count_; b_nodes++)
    {
        printf("%d ", branchpoint_nodes_[b_nodes]);
    }
    
    for (int node_id = 0; node_id < this->node_to_edge_->get_index_size(); ++node_id)
    {
        printf("\nNode: %d \n", node_id);
        array_info<edge_d> edges = this->node_to_edge_->get_index(node_id);

        array_info<guard_d> invariants = this->node_to_invariant_->get_index(node_id);
        printf("    Invariant amount: %d \n", invariants.size);
        for (int invariant_id=0; invariant_id < invariants.size; invariant_id++)
        {
            auto invariant = static_cast<guard_d>(invariants.arr[invariant_id]);
            printf("                %d %d %f\n", invariant.get_timer_id(),invariant.get_type(),invariant.get_value());
        }
        
        for (int edge_id = 0; edge_id < edges.size; ++edge_id)
        {
            int id_of_edge = edges.arr[edge_id].get_id();
            array_info<guard_d> guards = this->edge_to_guard_->get_index(id_of_edge);
            array_info<update_d> updates = this->edge_to_update_->get_index(id_of_edge);
            printf("    Edge id: %d, %d -> %d \n", id_of_edge, node_id, edges.arr[edge_id].get_dest_node());
            printf("        Guards amount: %d \n", guards.size);
            for (int guard_id = 0; guard_id<guards.size; guard_id++)
            {
                auto guard = static_cast<guard_d>(guards.arr[guard_id]);
                printf("                %d %d %f\n", guard.get_timer_id(),guard.get_type(),guard.get_value());
            }
            printf("        Updates: %d \n", updates.size);
            for (int update_id = 0; update_id<updates.size; update_id++)
            {
                auto update = static_cast<update_d>(updates.arr[update_id]);
                printf("                %d = %f\n", update.get_timer_id(),update.get_value());
            }
            printf("        Weight %f \n", edges.arr[edge_id].get_weight());
            for (int update_id = 0; update_id < updates.size; ++update_id)
            {
                printf("            Clock: %d, Value: %f \n", updates.arr[update_id].get_timer_id(), updates.arr[update_id].get_value());
            }
            updates.free_arr();
            guards.free_arr();
        }
        edges.free_arr();
    }
    printf("Clocks %d \n", this->timer_count_);
    for (int clock_id = 0; clock_id < this->timer_count_; ++clock_id)
    {
        printf("    Clock: %d, Value: %f \n", this->timers_[clock_id].get_id(), this->timers_->get_value());       
    }
}
