﻿#include "stochastic_model.h"
#include <assert.h>
#include <ctime>

using namespace std;

stochastic_model::stochastic_model(uneven_list<edge_d>* node_to_edge, uneven_list<guard_d>* node_to_invariant,
                                   uneven_list<guard_d>* edge_to_guard, uneven_list<update_d>* edge_to_update,
                                   timer_d* timers, const int timer_count)
{
    this->timer_count_ = timer_count;
    this->timers_ = timers;
    this->node_to_edge_ = node_to_edge;
    this->node_to_invariant_ = node_to_invariant;
    this->edge_to_guard_ = edge_to_guard;
    this->edge_to_update_ = edge_to_update;
}


GPU array_info<edge_d> stochastic_model::get_node_edges(const int node_id) const
{
    return this->node_to_edge_->get_index(node_id);
}

GPU array_info<guard_d> stochastic_model::get_node_invariants(const int node_id) const
{
    return this->node_to_invariant_->get_index(node_id);
}

GPU array_info<guard_d> stochastic_model::get_edge_guards(const int edge_id) const
{
    return this->edge_to_guard_->get_index(edge_id);

}

GPU array_info<update_d> stochastic_model::get_updates(const int edge_id) const
{
    return this->edge_to_update_->get_index(edge_id);
}


GPU void stochastic_model::traverse_edge_update(const int edge_id, const array_info<timer_d>* local_timers) const
{
    const array_info<update_d> updates = this->get_updates(edge_id);

    for (int i = 0; i < updates.size; ++i)
    {
        update_d* update = &updates.arr[i];
        const int timer_id = update->get_timer_id();
        timer_d* timer = &local_timers->arr[timer_id];

        timer->set_time(update->get_value());
    }
    
    updates.free_arr();
}

GPU int stochastic_model::get_start_node() const
{
    return 0;
}

GPU bool stochastic_model::is_goal_node(int node_id) const
{
    return node_id == 2;
}

GPU array_info<timer_d> stochastic_model::copy_timers() const
{
    const int size = this->timer_count_;
    timer_d* internal_timers_arr = static_cast<timer_d*>(malloc(sizeof(timer_d) * size));
    const array_info<timer_d> internal_timers{ internal_timers_arr, size};
    
    for (int i = 0; i < internal_timers.size; i++)
    {
        internal_timers.arr[i] = this->timers_[i].copy();
    }
    
    return internal_timers;
}

GPU void stochastic_model::reset_timers(const array_info<timer_d>* timers) const
{
    assert(timers->size == this->timer_count_);
    for (int i = 0; i < timers->size; i++)
    {
        timers->arr[i].set_time(this->timers_[i].get_value());
    }
    
}

void stochastic_model::cuda_allocate(stochastic_model** p, list<void*>* free_list) const
{
    //move internal lists to cuda    
    uneven_list<edge_d>* node_to_edge_d = nullptr;
    this->node_to_edge_->cuda_allocate(&node_to_edge_d, free_list);
    
    uneven_list<guard_d>* node_to_invariant_d = nullptr;
    this->node_to_invariant_->cuda_allocate(&node_to_invariant_d, free_list);

    uneven_list<guard_d>* edge_to_guard_d = nullptr;
    this->edge_to_guard_->cuda_allocate(&edge_to_guard_d, free_list);

    uneven_list<update_d>* edge_to_update_d = nullptr;
    this->edge_to_update_->cuda_allocate(&edge_to_update_d, free_list);

    //move timers to cuda
    timer_d* timers_d = nullptr;
    hipMalloc(&timers_d, sizeof(timer_d)*this->timer_count_);
    free_list->push_back(timers_d);
    hipMemcpy(timers_d, this->timers_, sizeof(timer_d)*this->timer_count_, hipMemcpyHostToDevice);

    //create model with cuda pointers
    const stochastic_model model = stochastic_model(
        node_to_edge_d, node_to_invariant_d,
        edge_to_guard_d, edge_to_update_d,
        timers_d, this->timer_count_);

    //move model with cuda pointers to device. Add to free list.
    hipMalloc(p, sizeof(stochastic_model));
    free_list->push_back((*p));
    hipMemcpy((*p), &model, sizeof(stochastic_model), hipMemcpyHostToDevice);

}