﻿#include "thread_pool.h"

void thread_pool::start()
{
    const uint32_t num_threads = std::thread::hardware_concurrency(); // Max # of threads the system supports
    threads_.resize(num_threads);
    for (uint32_t i = 0; i < num_threads; i++) {
        threads_.at(i) = std::thread([this]() {this->thread_loop();});
    }
}

void thread_pool::queue_job(const std::function<void()>& job)
{
    {
        std::unique_lock<std::mutex> lock(this->queue_mutex_);
        this->jobs_.push(job);
    }
    this->mutex_condition_.notify_one();
}

void thread_pool::stop()
{
    {
        std::unique_lock<std::mutex> lock(this->queue_mutex_);
        this->should_terminate_ = true;
    }
    this->mutex_condition_.notify_all();
    for (std::thread& active_thread : this->threads_) {
        active_thread.join();
    }
    this->threads_.clear();
}

bool thread_pool::is_busy()
{
    bool pool_busy;
    {
        std::unique_lock<std::mutex> lock(this->queue_mutex_);
        pool_busy = !this->jobs_.empty();
    }
    return pool_busy;
}

void thread_pool::thread_loop()
{
    while (true) {
        std::function<void()> job;
        {
            std::unique_lock<std::mutex> lock(this->queue_mutex_);
            mutex_condition_.wait(lock, [this] {
                return !this->jobs_.empty() || this->should_terminate_;
            });
            if (this->should_terminate_) {
                return;
            }
            job = this->jobs_.front();
            this->jobs_.pop();
        }
        job();
    }
}
