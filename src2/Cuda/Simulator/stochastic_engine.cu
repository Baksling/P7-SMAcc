#include "hip/hip_runtime.h"
﻿#include "stochastic_engine.h"
#include "stochastic_simulator.h"
#include "thread_pool.h"
#include ""
#include <chrono>
#include "simulator_tools.h"
#include "../Domain/edge_t.h"
#include "../Domain/stochastic_model_t.h"
#include "writers/result_manager.h"


using namespace std::chrono;


CPU GPU void run_simulator(simulator_state* state, result_manager* trace_tracker, const model_options* options)
{
    while (true)
    {
        model_state* current_model = state->progress_sim(options);

        if(current_model == nullptr || current_model->reached_goal)
        {
            break;
        }

        do //repeat as long as current node is branch node
        {
            lend_array<edge_t> outgoing_edges =  current_model->current_node->get_edges();
            if(outgoing_edges.size() == 0) break;

            const edge_t* edge = simulator_tools::choose_next_edge_bit(state, &outgoing_edges, state->random);
            if(edge == nullptr)
            {
                break;
            }
            
            current_model->current_node = edge->get_dest();
            edge->execute_updates(state);
            state->broadcast_channel(current_model, edge->get_channel(), trace_tracker);
        }
        while (current_model->current_node->is_branch_point());

        trace_tracker->write_step_trace(current_model, state);
        
        if(current_model->current_node->is_goal_node())
        {
            current_model->reached_goal = true;
        }
    }
}

CPU GPU void simulate_stochastic_model(
    const stochastic_model_t* model,
    const model_options* options,
    hiprandState* random_states,
    result_manager* output,
    const unsigned long idx,
    void* memory_heap
)
{
    hiprandState* r_state = &random_states[idx];
    hiprand_init(options->seed, idx, idx, r_state);
    
    simulator_state state = simulator_state::init(model, options, r_state, memory_heap);
    
    
    for (unsigned i = 0; i < options->simulation_amount; ++i)
    {
        const unsigned int sim_id = i + options->simulation_amount * static_cast<unsigned int>(idx);
        state.reset(sim_id, model);

        //run simulation
        run_simulator(&state, output, options);

        output->write_result(&state);
    }
}

__global__ void gpu_simulate(
    const stochastic_model_t* model,
    const model_options* options,
    hiprandState* r_state,
    result_manager* output,
    void* total_memory_heap
    )
{
    const unsigned long idx = threadIdx.x + blockDim.x * blockIdx.x;
    const unsigned long long int thread_memory_size = options->get_cache_size();
    const unsigned long long int offset = (idx * thread_memory_size) / sizeof(char);
    simulate_stochastic_model(model, options, r_state, output, idx, static_cast<void*>(&static_cast<char*>(total_memory_heap)[offset]));
}

bool stochastic_engine::run_gpu(
    const stochastic_model_t* model,
    const model_options* options,
    result_manager* output,
    const simulation_strategy* strategy,
    void* total_memory_heap)
{
    hiprandState* random_states = nullptr;
    hipMalloc(&random_states, sizeof(hiprandState)*strategy->block_n*strategy->threads_n);
    
    //simulate on device
    gpu_simulate<<<strategy->block_n, strategy->threads_n>>>(model, options, random_states, output, total_memory_heap);
        
    //wait for all processes to finish
    hipDeviceSynchronize();
    
    const hipError_t success = hipPeekAtLastError();
    if(success != hipSuccess) printf("\nAn error of code '%d' occured in cuda :( \n", success);
    hipFree(random_states);

    return success == hipSuccess;
}

bool stochastic_engine::run_cpu(
    const stochastic_model_t* model,
    const model_options* options,
    result_manager* output,
    const simulation_strategy* strategy,
    void* total_memory_heap)
{
    hiprandState* random_states = static_cast<hiprandState*>(malloc(sizeof(hiprandState)*strategy->degree_of_parallelism()));

    const unsigned long long int thread_memory_size = options->get_cache_size();
    //init thread pool
    thread_pool pool(strategy->cpu_threads_n);

    //add all jobs
    for (unsigned i = 0; i < strategy->degree_of_parallelism(); i++)
    {
        unsigned long long int offset = (i * thread_memory_size) / sizeof(char);
        pool.queue_job([model, options, random_states, output, i, total_memory_heap, offset]()
        {
            simulate_stochastic_model(model, options, random_states, output, i, static_cast<void*>(&static_cast<char*>(total_memory_heap)[offset]));
        });
    }
    //Start processing jobs in pool.
    pool.start();
    
    while(pool.is_busy()) //wait for pool to process all tasks
    {
        std::this_thread::yield();
    }

    //stop pool
    pool.stop();
    free(random_states);
    
    return true;
}
