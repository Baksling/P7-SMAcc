#include "hip/hip_runtime.h"
﻿#include "stochastic_engine.h"
#include "stochastic_simulator.h"
#include "thread_pool.h"
#include ""
#include <chrono>
#include "simulator_tools.h"
#include "../Domain/edge_t.h"
#include "../Domain/stochastic_model_t.h"
#include "../Domain/channel_medium.h"

using namespace std::chrono;


CPU GPU void run_simulator(simulator_state* state, hiprandState* r_state, const model_options* options)
{
    while (true)
    {
        model_state* current_model = state->progress_sim(options, r_state);

        if(current_model == nullptr || current_model->reached_goal)
        {
            break;
        }

        do //repeat as long as current node is branch node
        {
            lend_array<edge_t*> outgoing_edges =  current_model->current_node->get_edges();
            if(outgoing_edges.size() == 0) break;

            const edge_t* edge = simulator_tools::choose_next_edge_bit(state, &outgoing_edges, r_state);
            if(edge == nullptr)
            {
                break;
            }
            
            current_model->current_node = edge->get_dest();
            edge->execute_updates(state);
            state->broadcast_channel(current_model, edge->get_channel(), r_state);
        }
        while (current_model->current_node->is_branch_point());
        
        if(current_model->current_node->is_goal_node())
        {
            current_model->reached_goal = true;
        }
    }
}

CPU GPU void simulate_stochastic_model(
    const stochastic_model_t* model,
    const model_options* options,
    hiprandState* random_states,
    const simulation_result_container* output,
    const unsigned long idx,
    void* memory_heap
)
{
    hiprandState* r_state = &random_states[idx];
    hiprand_init(options->seed, idx, idx, r_state);
    
    simulator_state state = simulator_state::from_multi_model(model, options, memory_heap);
    
    
    for (unsigned i = 0; i < options->simulation_amount; ++i)
    {
        const unsigned int sim_id = i + options->simulation_amount * static_cast<unsigned int>(idx);
        state.reset(sim_id, model);

        //run simulation
        run_simulator(&state, r_state, options);

        state.write_result(output);
    }
    
    state.free_internals();
}

__global__ void gpu_simulate(
    const stochastic_model_t* model,
    const model_options* options,
    hiprandState* r_state,
    const simulation_result_container* output,
    void* total_memory_heap
    )
{
    const unsigned long idx = threadIdx.x + blockDim.x * blockIdx.x;
    const unsigned long long int thread_memory_size = options->get_cache_size();
    const unsigned long long int offset = (idx * thread_memory_size) / sizeof(char);
    simulate_stochastic_model(model, options, r_state, output, idx, static_cast<void*>(&static_cast<char*>(total_memory_heap)[offset]));
}

bool stochastic_engine::run_gpu(
    const stochastic_model_t* model,
    const model_options* options,
    const simulation_result_container* output,
    const simulation_strategy* strategy,
    void* total_memory_heap)
{
    hiprandState* random_states = nullptr;
    hipMalloc(&random_states, sizeof(hiprandState)*strategy->block_n*strategy->threads_n);

    // const unsigned long long int thread_memory_size = options->get_cache_size();
    // void* original_store = malloc(thread_memory_size);
    
    
    if(hipSuccess != hipDeviceSetLimit(hipLimitMallocHeapSize, 8589934592))
    {
        printf("Could not allocate heap space on cuda device\n");
        return false;
    }
    
    //simulate on device
    gpu_simulate<<<strategy->block_n, strategy->threads_n>>>(model, options, random_states, output, total_memory_heap);
        
    //wait for all processes to finish
    hipDeviceSynchronize();
    
    const hipError_t success = hipPeekAtLastError();
    if(success != hipSuccess) printf("\nAn error of code '%d' occured in cuda :( \n", success);
    hipFree(random_states);

    hipFree(total_memory_heap);

    return success == hipSuccess;
}

bool stochastic_engine::run_cpu(
    const stochastic_model_t* model,
    const model_options* options,
    simulation_result_container* output,
    const simulation_strategy* strategy,
    void* total_memory_heap)
{
    hiprandState* random_states = static_cast<hiprandState*>(malloc(sizeof(hiprandState)*strategy->degree_of_parallelism()));

    const unsigned long long int thread_memory_size = options->get_cache_size();
    //init thread pool
    thread_pool pool(strategy->cpu_threads_n);

    //add all jobs
    for (unsigned i = 0; i < strategy->degree_of_parallelism(); i++)
    {
        unsigned long long int offset = (i * thread_memory_size) / sizeof(char);
        pool.queue_job([model, options, random_states, output, i, total_memory_heap, offset]()
        {
            simulate_stochastic_model(model, options, random_states, output, i, static_cast<void*>(&static_cast<char*>(total_memory_heap)[offset]));
        });
    }
    //Start processing jobs in pool.
    pool.start();
    
    while(pool.is_busy()) //wait for pool to process all tasks
    {
    }

    //stop pool
    pool.stop();
    free(random_states);
    
    return true;
}
