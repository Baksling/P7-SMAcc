﻿#include "result_store.h"
#include <iostream>
#include <string>

result_pointers::result_pointers(const bool owns_pointers,
    void* free_p,
    node_results* nodes,
    variable_result* variables,
    const int threads,
    const unsigned total_simulations): owns_pointers_(owns_pointers), source_p_(free_p)
{
    this->nodes = nodes;
    this->variables = variables;
    this->total_simulations = total_simulations;
    this->threads = threads;
}

void result_pointers::free_internals() const
{
    if(!owns_pointers_) return;
    free(this->source_p_);
}

size_t result_store::total_data_size() const
{
    return sizeof(node_results)    * this->node_count_ * this->thread_count_
         + sizeof(variable_result) * this->variables_count_ * this->thread_count_;
}


result_store::result_store(const unsigned total_sim,
    const unsigned variables,
    const unsigned node_count,
    const int thread_count, memory_allocator* helper)
{
    this->is_cuda_ = helper->use_cuda;
    this->simulations_ = total_sim;
    this->node_count_ = node_count;
    this->variables_count_ = variables;
    this->thread_count_ = thread_count;

    const size_t total_data = this->total_data_size();

    void* store;
    CUDA_CHECK(helper->allocate(&store, total_data));
    
    this->node_p_ = static_cast<node_results*>(store);
    store = static_cast<void*>(&this->node_p_[this->node_count_ * this->thread_count_]);
        
    this->variable_p_ = static_cast<variable_result*>(store);
}

result_pointers result_store::load_results() const
{
    if(!this->is_cuda_) return result_pointers{
        false,
        nullptr,
        this->node_p_,
        this->variable_p_,
        this->thread_count_,
        this->simulations_
    };

    const size_t size = this->total_data_size();
    const void* source = static_cast<void*>(this->node_p_); //this is the source of the array. nodes and variables are just offsets from here
    void* init_store = malloc(size);
    void* store = init_store;
    CUDA_CHECK(hipMemcpy(store, source , size, hipMemcpyDeviceToHost));

    node_results* nodes = static_cast<node_results*>(store);
    const int offset = static_cast<int>(this->node_count_) * this->thread_count_;
    store = static_cast<void*>(&nodes[offset]);
        
    variable_result* vars = static_cast<variable_result*>(store);
    return result_pointers{
        true,
        init_store,
        nodes,
        vars,
        this->thread_count_,
        this->simulations_
    };
}
