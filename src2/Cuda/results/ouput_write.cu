#include "hip/hip_runtime.h"
﻿#include "output_writer.h"

float output_writer::calc_percentage(const unsigned long long counter, const unsigned long long divisor)
{
    return (static_cast<float>(counter) / static_cast<float>(divisor)) * 100.0f;
}

void output_writer::write_to_file(const result_pointers* results,
    std::chrono::steady_clock::duration sim_duration) const
{
    std::ofstream file_node, file_variable;
    
    file_node.open(this->file_path_ + "_node_data_" + std::to_string(output_counter_) + ".csv" );
    file_variable.open(this->file_path_ + "_variable_data_" + std::to_string(output_counter_) + ".csv");
    
    file_node << "Simulation,Model,Node,Steps,Time\n";
    file_variable << "Simulation,Variable,Value\n";
    
    
    for (unsigned i = 0; i < total_simulations_; ++i)
    {
        const sim_metadata local_result = results->meta_results[i];
        for (unsigned  j = 0; j < this->model_count_; ++j)
        {
            file_node << i << "," << j << "," << results->nodes[this->model_count_ * i + j] << ","
            << local_result.steps << "," << local_result.global_time << "\n";
        }
    
        for (int k = 0; k < this->variable_summaries_.size; ++k)
        {
            file_variable << i << "," << k << "," << results->variables[this->variable_summaries_.size * i + k] << "\n";
        }
    }

    file_node.flush();
    file_variable.flush();
     
    file_node.close();
    file_variable.close();
}

void insert_name(const int id, const std::unordered_map<int, std::string>& name_map, std::ostream& stream)
{
    stream << abs(id) << ": ";
    if(name_map.count(abs(id)))
    {
        stream << name_map.at(abs(id));
    }
    else
    {
        stream << '_';
    }
    stream << (HAS_HIT_MAX_STEPS(id) ? " (not goal)" : "");
}

void output_writer::write_summary_to_stream(std::ostream& stream,
                                            std::chrono::steady_clock::duration sim_duration) const
{
    stream << "\nAverage maximum value of each variable: \n";
    
    for (int j = 0; j < this->variable_summaries_.size; ++j)
    {
        const variable_summary result = this->variable_summaries_.store[j];
        stream << "variable " << result.variable_id << " = " << result.avg_max_value << "\n";
    }
    
    stream << "\n\nReachability: \n";
    bool has_seen_hit = false;
    
    // node_summary no_hit = node_summary{0, 0};
    std::map<int, node_summary> no_hit_map{};
    stream << "Not reached:\n";
    
    for (const std::pair<const int, node_summary>& it : this->node_summary_map_)
    {
        if(HAS_HIT_MAX_STEPS(it.first))
        {
            // no_hit.cumulative(it.second);
            int network = this->node_network_.count(abs(it.first))
            ? this->node_network_.at(abs(it.first))
            : 0;
            
            if(no_hit_map.count(network))
                no_hit_map[network].cumulative(it.second);
            else
                no_hit_map[network] = it.second;
        }
        else  if(!has_seen_hit)
        {
            has_seen_hit = true;
            stream << "Goals: \n";
        }
        
        const float percentage = calc_percentage(it.second.reach_count, total_simulations_);
        insert_name(it.first, this->node_names_, stream);
        stream << " reached " << it.second.reach_count << " times (" << percentage << "%)\n";
        stream << "    " << " avg steps: " << it.second.avg_steps << " | avg sim time: " << it.second.avg_time << "t.\n";
    }

    stream << "\nPer process\n";
    node_summary global_no_hit = {0,0,0};
    for (int i = 0; i < static_cast<int>(this->model_count_); ++i)
    {
        const node_summary no_hit = no_hit_map.count(i)
                                        ? no_hit_map[i]
                                        : node_summary{0, 0, 0} ;
        global_no_hit.cumulative(no_hit);
        const float percentage = calc_percentage(no_hit.reach_count, total_simulations_);

        stream << "Process " << i << " did not reach goal " << no_hit.reach_count << " times ("<< percentage <<"%)\n";
        stream << "    " <<" avg steps: " << no_hit.avg_steps << " | avg sim time: " << no_hit.avg_time << "t.\n";
    }

    stream << '\n';
    stream << "Probability of false negative results (alpha): " << this->alpha_*100 << "%\n";  
    stream << "Probability uncertainty (-+epsilon) of results: " << this->epsilon_*100 << "%\n";  
    stream << "Nr of simulations: " << total_simulations_ << "\n\n";
    stream << "Simulation ran for: " << std::chrono::duration_cast<std::chrono::milliseconds>(sim_duration).count()
           << "[ms]" << "\n";
}

void output_writer::write_lite(std::chrono::steady_clock::duration sim_duration) const
{
    std::ofstream lite;
    const std::string file_path = file_path_ + "_lite_summary.txt";
    lite.open(file_path);

    lite << std::chrono::duration_cast<std::chrono::milliseconds>(sim_duration).count();

    lite.flush();
    lite.close();
}

void output_writer::write_hit_file(const std::chrono::steady_clock::duration sim_duration) const
{
    if (this->node_summary_map_.empty()) return;
    std::ofstream file = std::ofstream(this->file_path_ + "_results.tsv", std::ofstream::out|std::ofstream::trunc);
    
    bool any = false;
    for (const auto& pair : this->node_summary_map_)
    {
        if (HAS_HIT_MAX_STEPS(pair.first)) continue;
        const float percentage = this->calc_percentage(pair.second.reach_count, total_simulations_);
        file << percentage << "\t" << std::chrono::duration_cast<std::chrono::milliseconds>(sim_duration).count();
        any = true;
    }
    if(!any)
    {
        file << "0.0\t" << std::chrono::duration_cast<std::chrono::milliseconds>(sim_duration).count();
    }

    file.flush();
    file.close();
}

output_writer::output_writer(const sim_config* config, const network* model)
{
    this->file_path_ = config->paths->output_path;
    this->write_mode_ = config->write_mode;
    this->node_summary_map_ = std::map<int, node_summary>();
    this->model_count_ = config->network_size;
    this->total_simulations_ = static_cast<unsigned>(config->total_simulations()) * config->simulation_repetitions;
    this->alpha_ = config->alpha;
    this->epsilon_ = config->epsilon;
    this->node_names_ = std::unordered_map<int, std::string>(*config->properties->node_names);
    this->node_network_ = std::unordered_map<int, int>(*config->properties->node_network);
    
    this->variable_summaries_ = arr<variable_summary>{
        static_cast<variable_summary*>(malloc(sizeof(variable_summary)*config->tracked_variable_count)),
        static_cast<int>(config->tracked_variable_count)
    };
    
    for (int i = 0, j = 0; i < model->variables.size; ++i)
    {
        if(!model->variables.store[i].should_track) continue;
        this->variable_summaries_.store[j++] = variable_summary{static_cast<unsigned>(i), 0.0, 0};
    }

}

void output_writer::write(const result_store* sim_result, std::chrono::steady_clock::duration sim_duration)
{
    if (this->write_mode_ & lite_sum)
    {
        this->write_lite(sim_duration);
    }

    if(this->write_mode_ & (console_sum | file_sum | file_data | hit_file))
    {
        const result_pointers pointers = sim_result->load_results();

        for (unsigned i = 0; i < this->total_simulations_; ++i)
        {
            const sim_metadata x = pointers.meta_results[i];
            for (unsigned j = 0; j < model_count_; ++j)
            {
                int n = pointers.nodes[i*model_count_ + j];
                if(this->node_summary_map_.count(n)) //exists
                    this->node_summary_map_[n].add_reach(x.steps, x.global_time);
                else
                    this->node_summary_map_.insert(
                        std::pair<int, node_summary>(
                            n, node_summary{ 1, static_cast<double>(x.steps), x.global_time }
                            ));
            }
            for (int j = 0; j < this->variable_summaries_.size; ++j)
            {
                const double v = pointers.variables[i*this->variable_summaries_.size + j];
                this->variable_summaries_.store[j].update_count(v);
            }
        }
        if(this->write_mode_ & file_data) write_to_file(&pointers, sim_duration);
        pointers.free_internals();
    }
    output_counter_++;
}

void output_writer::write_summary(std::chrono::steady_clock::duration sim_duration) const
{
    if (this->write_mode_ & console_sum)
    {
        this->write_summary_to_stream(std::cout, sim_duration);
    }
    if (this->write_mode_ & file_sum)
    {
        const std::string path = this->file_path_ + "_summary.csv";
    
        std::ofstream summary;
        summary.open(path);
        this->write_summary_to_stream(summary, sim_duration);

        summary.flush();
        summary.close();
    }
    if (this->write_mode_ & hit_file)
    {
        this->write_hit_file(sim_duration);   
    }
}

void output_writer::clear()
{
    this->node_summary_map_.clear();
    for (int i = 0; i < this->variable_summaries_.size; ++i)
    {
        this->variable_summaries_.store[i] = variable_summary{
            this->variable_summaries_.store[i].variable_id,
            0.0,
            0,
        };
    }
}
