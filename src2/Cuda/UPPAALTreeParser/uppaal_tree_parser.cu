﻿#include "uppaal_tree_parser.h"


#define ALPHA "abcdefghijklmnopqrstuvwxyz"



char get_constraint_op(const string& expr)
{
    if(expr.find("<=") != std::string::npos)
        return '=';
    if(expr.find(">=") != std::string::npos)
        return '=';
    if(expr.find("==") != std::string::npos)
        return '=';
    if(expr.find("!=") != std::string::npos)
        return '=';
    if(expr.find('<') != std::string::npos)
        return '<';
    if(expr.find('>') != std::string::npos)
        return '>';
    THROW_LINE("Operand in " + expr + " not found, sad..");
}

constraint_t* get_constraint(const string& expr, const int timer_id, expression* value)
{
    if(expr.find("<=") != std::string::npos)
        return constraint_t::less_equal_v(timer_id, value);
    if(expr.find(">=") != std::string::npos)
        return constraint_t::greater_equal_v(timer_id,value);
    if(expr.find("==") != std::string::npos)
        return constraint_t::equal_v(timer_id,value);
    if(expr.find("!=") != std::string::npos)
        return constraint_t::not_equal_v(timer_id,value);
    if(expr.find('<') != std::string::npos)
        return constraint_t::less_v(timer_id,value);
    if(expr.find('>') != std::string::npos)
        return constraint_t::greater_v(timer_id,value);
    THROW_LINE("Operand in " + expr + " not found, sad..");
}

constraint_t* get_constraint(const string& expr, const int timer_id_1, const int timer_id_2)
{
    if(expr.find("<=") != std::string::npos)
        return constraint_t::less_equal_t(timer_id_1, timer_id_2);
    if(expr.find(">=") != std::string::npos)
        return constraint_t::greater_equal_t(timer_id_1, timer_id_2);
    if(expr.find("==") != std::string::npos)
        return constraint_t::equal_t(timer_id_1, timer_id_2);
    if(expr.find('<') != std::string::npos)
        return constraint_t::less_t(timer_id_1, timer_id_2);
    if(expr.find('>') != std::string::npos)
        return constraint_t::greater_t(timer_id_1, timer_id_2);
    THROW_LINE("Operand in " + expr + " not found, sad..")
}


template <typename T> T* list_to_arr(list<T> l)
{
    T* arr = static_cast<T*>(malloc(sizeof(T) * l.size()));
    int k = 0;
    for (T const &i: l) {
        arr[k++] = i;
    }
    
    return arr;
}


int uppaal_tree_parser::get_timer_id(const string& expr) const
{
    const string expr_wout_spaces = replace_all(expr, string(" "), string(""));
    int index = 0;

    while (true)
    {
        if (static_cast<int>(expr.size()) == index)
        {
            THROW_LINE("sum tin wong")
        }
        
        if (in_array(expr_wout_spaces[++index], {'<','>','='}))
        {
            break;
        }
    }

    const string sub = expr_wout_spaces.substr(0, index);

    if ( vars_map_.count(sub))
    {
        return vars_map_.at(sub);
    }
    
    if (global_vars_map_.count(sub))
    {
        return global_vars_map_.at(sub);
    }
    
    THROW_LINE("sum tin wong")
}

template <typename T>
void uppaal_tree_parser::get_guys(const list<string>& expressions, list<T>* t)
{
    for(const auto& expr: expressions)
    {
        if (expr.empty())
            continue;

        const string right_side = take_after(expr, get_constraint_op(expr));
        //TODO fix this plz
        //Constraint is heap allocated, and is then copied here.
        //Results in dead memory.
        t->push_back(*get_constraint(expr, get_timer_id(expr), update_parser::parse(right_side, &vars_map_, &global_vars_map_)));
    }
}


void uppaal_tree_parser::init_clocks(const xml_document* doc)
{
    string global_decl = doc->child("nta").child("declaration").child_value();
    global_decl = replace_all(global_decl, " ", "");
    const list<declaration> decls = dp_.parse(global_decl);
        
    for (declaration d : decls)
    {
        //global declarations
        if(d.get_type() == clock_type)
        {
            global_vars_map_.insert_or_assign(d.get_name(),clock_id_);
            timers_map_.insert_or_assign(d.get_name(), clock_id_);
            timer_list_->push_back(clock_variable(clock_id_++, d.get_value()));
        }
        else if(d.get_type() == chan_type)
        {
            global_vars_map_.insert_or_assign(d.get_name(), chan_id_++);
        }
        else
        {
            global_vars_map_.insert_or_assign(d.get_name(), var_id_);
            var_list_->push_back(clock_variable(var_id_++, d.get_value()));
        }
    }
    
    
    for (pugi::xml_node templates: doc->child("nta").children("template"))
    {
        string decl = templates.child("declaration").child_value();
        decl = replace_all(decl, " ", "");
        list<declaration> declarations = dp_.parse(decl);
        
        for (declaration d : declarations)
        {
            //local declarations
            if(d.get_type() == clock_type)
            {
                vars_map_.insert_or_assign(d.get_name(),clock_id_);
                timers_map_.insert_or_assign(d.get_name(), clock_id_);
                timer_list_->push_back(clock_variable(clock_id_++, d.get_value()));
            }
            else if(d.get_type() == chan_type)
            {
                vars_map_.insert_or_assign(d.get_name(), chan_id_++);
            }
            else
            {
                vars_map_.insert_or_assign(d.get_name(), var_id_);
                var_list_->push_back(clock_variable(var_id_++, d.get_value()));
            }
        }
    }
}

uppaal_tree_parser::uppaal_tree_parser()
= default;

node_t* uppaal_tree_parser::get_node(const int target_id, const list<node_t*>* arr) const
{
    for(node_t* node: *arr)
    {
        if(node->get_id() == target_id)
            return node;
    }
    return arr->front();
}


__host__ stochastic_model_t uppaal_tree_parser::parse_xml(char* file_path)
{
    string path = file_path;
    xml_document doc;
    unordered_map<int, list<edge_t>> node_edge_map; //TODO edge is no longer pointer
    declaration_parser dp;
    
    // load the XML file
    if (!doc.load_file(file_path))
    {
        THROW_LINE("The specified file does not exist.. stupid.")
    }

    int edge_id = 0;
    int update_id = 0;

    init_clocks(&doc);
    
    for (pugi::xml_node templates: doc.child("nta").children("template"))
    {
        string init_node = templates.child("init").attribute("ref").as_string();
        init_node_id_ = xml_id_to_int(init_node);
        for (pugi::xml_node locs: templates.children("location"))
        {
            string string_id = locs.attribute("id").as_string();
            string string_name = locs.child("name").child_value();
            const int node_id = xml_id_to_int(string_id);
            bool is_goal = false;
            node_edge_map.insert_or_assign(node_id, list<edge_t>());
            
            list<constraint_t> invariants; //TODO no longer pointer
            expression* expo_rate = nullptr;
            
            if (string_name != "")
            {
                is_goal = string_name == "Goal" ? true : false;
                node_names_->emplace(node_id, string_name);
            }
            
            string kind = locs.child("label").attribute("kind").as_string();
            string expr_string = locs.child("label").child_value();

            list<string> expressions = split_expr(expr_string);
            
            if (kind == "exponentialrate")
            {
                expo_rate = update_parser::parse(expr_string, &vars_map_, &global_vars_map_);
            }
            
            if (kind == "invariant")
            {
                get_guys(expressions, &invariants);
            }
            if (init_node_id_ == node_id)
                start_nodes_.push_back(nodes_->size());
            
            node_t* node = new node_t(node_id, to_array(&invariants),false, is_goal, expo_rate);
            nodes_->push_back(node);
            nodes_map_->emplace(node->get_id(), node_with_system_id(node, this->system_count));
        }

        for (pugi::xml_node locs: templates.children("branchpoint"))
        {
            string string_id = locs.attribute("id").as_string();
            const int node_id = xml_id_to_int(string_id);
            node_edge_map.insert_or_assign(node_id, list<edge_t>()); //TODO removed pointer edge pointer
            nodes_->push_back(new node_t(node_id,array_t<constraint_t>(0), true));
        }

        
        
        for (pugi::xml_node trans: templates.children("transition"))
        {
            string source = trans.child("source").attribute("ref").as_string();
            string target = trans.child("target").attribute("ref").as_string();

            int source_id = xml_id_to_int(source);
            int target_id = xml_id_to_int(target);
            
            list<constraint_t> guards; //TODO removed pointer part
            list<update_t> updates; //TODO removed pointer part
            expression* probability = nullptr;
            edge_channel* ec = nullptr;
            
            for (pugi::xml_node labels: trans.children("label"))
            {
                string kind = labels.attribute("kind").as_string();
                string expr_string = labels.child_value();

                
                if(kind == "guard")
                {
                    list<string> expressions = split_expr(expr_string);
                    get_guys(expressions, &guards);
                }
                else if (kind == "assignment")
                {
                    list<string> expressions = split_expr(expr_string, ',');
                    for(const auto& expr: expressions)
                    {
                        if (expr.empty())
                            continue;
                        
                        expression* e = update_parser::parse(expr, &vars_map_, &global_vars_map_);
                        string keyword = take_while(expr, '=');
                        bool is_clock = false;

                        if(timers_map_.count(keyword) > 0)
                        {
                            is_clock = true;
                        }

                        
                        updates.push_back(update_t(update_id++, get_timer_id(expr), is_clock, e));
                    }
                }
                else if (kind == "synchronisation")
                {
                    ec = new edge_channel();
                    // printf("\n !=!=!==!=!=!== %s \n | %d", expr_string.c_str(), !does_not_contain(expr_string, "!"));
                    if (!does_not_contain(expr_string, "!"))
                    {
                        ec->is_listener = false;
                    }
                    else
                    {
                        ec->is_listener = true;
                    }
                    string sync_keyword = replace_all(expr_string, "!", "");
                    sync_keyword = replace_all(sync_keyword, "?", "");
                    sync_keyword = replace_all(sync_keyword, " ", "");
                    
                    if (vars_map_.count(sync_keyword))
                    {
                        ec->channel_id = vars_map_.at(sync_keyword);
                    }
                    else if (global_vars_map_.count(sync_keyword))
                    {
                        ec->channel_id = global_vars_map_.at(sync_keyword);
                    }
                    else
                    {
                        THROW_LINE(sync_keyword + " NOT IN LOCAL, NOR GLOBAL MAP, comeon dude..");
                    }
                    
                }
                else if (kind == "probability")
                {
                    probability = update_parser::parse(expr_string, &vars_map_, &global_vars_map_);
                }
            }

            if (probability == nullptr) probability = expression::literal_expression(1.0);
            
            node_t* target_node = get_node(target_id, nodes_);
            edge_t result_edge = ec == nullptr
                ? edge_t(edge_id++, probability, target_node, to_array(&guards), to_array(&updates))
                : edge_t(edge_id++, probability, target_node, to_array(&guards), to_array(&updates), *ec);
            
            node_edge_map.at(source_id).push_back(result_edge);
        }
        vars_map_.clear();
        this->system_count++;
    }

    for(node_t* node: *nodes_)
    {
        node->set_edges(&node_edge_map.at(node->get_id()));
    }
    
    //TODO i broke plz fix :)
    //The stochastic model now expects an array of objects, rather than a array of object pointers.
    //This helps cut down on the number of times pointers need to be followed in the simulation.
    //Only reason it was like that before, was because we didnt know how to make the cuda-allocation code without it :)
    // - Bak ඞ
    array_t<node_t*> start_nodes = array_t<node_t*>(start_nodes_.size());

    

    int number_of_start_nodes = 0;
    for (int i : start_nodes_)
    {
        auto n_front = nodes_->begin();
        std::advance(n_front, i);
        start_nodes.arr()[number_of_start_nodes++] = *n_front;
    }

    return stochastic_model_t(start_nodes, to_array(timer_list_), to_array(var_list_));
}

__host__ stochastic_model_t uppaal_tree_parser::parse(string file_path)
{
    try
    {
        char* writeable = new char[file_path.size() + 1];
        std::copy(file_path.begin(), file_path.end(), writeable);
        writeable[file_path.size()] = '\0';
        auto model = parse_xml(writeable);
        delete[] writeable;
        return model;
    }
    catch (const std::runtime_error &ex)
    {
        cout << "Parse error: " << ex.what() << "\n";
        throw runtime_error("parse error");
    }
}