﻿#include "uppaal_tree_parser.h"

#include <fstream>


#define ALPHA "abcdefghijklmnopqrstuvwxyz"

char get_constraint_op(const string& expr)
{
    if(expr.find("<=") != std::string::npos)
        return '=';
    if(expr.find(">=") != std::string::npos)
        return '=';
    if(expr.find("==") != std::string::npos)
        return '=';
    if(expr.find("!=") != std::string::npos)
        return '=';
    if(expr.find('<') != std::string::npos)
        return '<';
    if(expr.find('>') != std::string::npos)
        return '>';
    THROW_LINE("Operand in " + expr + " not found, sad..");
}

constraint_t* get_constraint(const string& expr, const int timer_id, expression* value)
{
    if(expr.find("<=") != std::string::npos)
        return constraint_t::less_equal_v(timer_id, value);
    if(expr.find(">=") != std::string::npos)
        return constraint_t::greater_equal_v(timer_id,value);
    if(expr.find("==") != std::string::npos)
        return constraint_t::equal_v(timer_id,value);
    if(expr.find("!=") != std::string::npos)
        return constraint_t::not_equal_v(timer_id,value);
    if(expr.find('<') != std::string::npos)
        return constraint_t::less_v(timer_id,value);
    if(expr.find('>') != std::string::npos)
        return constraint_t::greater_v(timer_id,value);
    THROW_LINE("Operand in " + expr + " not found, sad..");
}

constraint_t* get_constraint(const string& expr, const int timer_id_1, const int timer_id_2)
{
    if(expr.find("<=") != std::string::npos)
        return constraint_t::less_equal_t(timer_id_1, timer_id_2);
    if(expr.find(">=") != std::string::npos)
        return constraint_t::greater_equal_t(timer_id_1, timer_id_2);
    if(expr.find("==") != std::string::npos)
        return constraint_t::equal_t(timer_id_1, timer_id_2);
    if(expr.find('<') != std::string::npos)
        return constraint_t::less_t(timer_id_1, timer_id_2);
    if(expr.find('>') != std::string::npos)
        return constraint_t::greater_t(timer_id_1, timer_id_2);
    THROW_LINE("Operand in " + expr + " not found, sad..")
}


template <typename T> T* list_to_arr(list<T> l)
{
    T* arr = static_cast<T*>(malloc(sizeof(T) * l.size()));
    int k = 0;
    for (T const &i: l) {
        arr[k++] = i;
    }
    
    return arr;
}


int uppaal_tree_parser::get_timer_id(const string& expr) const
{
    const string expr_wout_spaces = replace_all(expr, string(" "), string(""));
    int index = 0;
    bool is_unary = false;

    while (true)
    {
        if (static_cast<int>(expr.size()) == index)
        {
            THROW_LINE("sum tin wong")
        }
        
        if (in_array(expr_wout_spaces[index], {'<','>','='}))
        {
            break;
        }
        cout << "\nHUH?:"<<expr_wout_spaces[index]<<":"<<index;
        cout << "\nHUH?2:"<<expr_wout_spaces<<":"<<index;
        if (expr_wout_spaces[index] == '-' && expr_wout_spaces[index+1] == '-' || expr_wout_spaces[index] == '+' && expr_wout_spaces[index+1] == '+')
        {
            if (expr_wout_spaces.front() != '-' && expr_wout_spaces.front() != '+') break;
            
            index = index + 2;
            is_unary = true;
            break;
        }
        index++;
    }

    const string sub = is_unary ? expr_wout_spaces.substr(index) : expr_wout_spaces.substr(0, index);
    cout << "\nSUB!:"<<sub<<":";
    cout.flush();

    if ( vars_map_.count(sub))
    {
        return vars_map_.at(sub);
    }
    
    if (global_vars_map_.count(sub))
    {
        return global_vars_map_.at(sub);
    }
    
    THROW_LINE("sum tin wong")
}

string uppaal_tree_parser::get_assignment_keyword(const string& ass)
{
    if(ass.substr(0,2) == "--" || ass.substr(0,2) == "++")
        return ass.substr(2);

    return take_while(ass, '=');
}

template <typename T>
void uppaal_tree_parser::get_guys(const list<string>& expressions, list<T>* t)
{
    for(const auto& expr: expressions)
    {
        if (expr.empty())
            continue;

        const string right_side = take_after(expr, get_constraint_op(expr));
        t->push_back(get_constraint(expr, get_timer_id(expr), update_parser::parse(right_side, &vars_map_, &global_vars_map_)));
    }
}


void uppaal_tree_parser::init_clocks(const xml_document* doc)
{
    string global_decl = doc->child("nta").child("declaration").child_value();
    //cout << "\nGLOBAL GUYS: " << global_decl << " :NICE\n";
    global_decl = replace_all(global_decl, " ", "");
    const list<declaration> decls = dp_.parse(global_decl);
    //cout << "\nSIZE: " << decls.size() << "\n";
        
    for (declaration d : decls)
    {
        //global declarations
        if(d.get_type() == clock_type)
        {
            global_vars_map_.insert_or_assign(d.get_name(),clock_id_);
            timers_map_.insert_or_assign(d.get_name(), clock_id_);
            timer_list_->push_back(clock_variable(clock_id_++, d.get_value()));
        }
        else if(d.get_type() == chan_type)
        {
            global_vars_map_.insert_or_assign(d.get_name(), chan_id_++);
        }
        else
        {
            global_vars_map_.insert_or_assign(d.get_name(), var_id_);
            var_list_->push_back(clock_variable(var_id_++, d.get_value()));
        }
    }
    
    
    for (pugi::xml_node templates: doc->child("nta").children("template"))
    {
        string decl = templates.child("declaration").child_value();
        decl = replace_all(decl, " ", "");
        list<declaration> declarations = dp_.parse(decl);
        //cout << "\nSIZE: " << declarations.size() << "\n";
        
        for (declaration d : declarations)
        {
            //local declarations
            if(d.get_type() == clock_type)
            {
                vars_map_.insert_or_assign(d.get_name(),clock_id_);
                timers_map_.insert_or_assign(d.get_name(), clock_id_);
                timer_list_->push_back(clock_variable(clock_id_++, d.get_value()));
            }
            else if(d.get_type() == chan_type)
            {
                vars_map_.insert_or_assign(d.get_name(), chan_id_++);
            }
            else
            {
                vars_map_.insert_or_assign(d.get_name(), var_id_);
                var_list_->push_back(clock_variable(var_id_++, d.get_value()));
            }
        }
    }
}

uppaal_tree_parser::uppaal_tree_parser()
= default;

node_t* uppaal_tree_parser::get_node(const int target_id, const list<node_t*>* arr) const
{
    for(node_t* node: *arr)
    {
        if(node->get_id() == target_id)
            return node;
    }
    return arr->front();
}


__host__ stochastic_model_t uppaal_tree_parser::parse_xml(char* file_path)
{
    string path = file_path;
    xml_document doc;
    map<int, list<edge_t*>> node_edge_map;
    declaration_parser dp;
    
    // load the XML file
    if (!doc.load_file(file_path))
    {
        THROW_LINE("The specified file does not exist.. stupid.")
    }

    int edge_id = 0;
    int update_id = 0;

    init_clocks(&doc);
    
    for (pugi::xml_node templates: doc.child("nta").children("template"))
    {
        string init_node = templates.child("init").attribute("ref").as_string();
        init_node_id_ = xml_id_to_int(init_node);
        for (pugi::xml_node locs: templates.children("location"))
        {
            string string_id = locs.attribute("id").as_string();
            string string_name = locs.child("name").child_value();
            const int node_id = xml_id_to_int(string_id);
            bool is_goal = false;
            node_edge_map.insert_or_assign(node_id, list<edge_t*>());
            
            list<constraint_t*> invariants;
            expression* expo_rate = nullptr;
            
            if (string_name == "Goal")
                is_goal = true;
            
            string kind = locs.child("label").attribute("kind").as_string();
            string expr_string = locs.child("label").child_value();

            list<string> expressions = split_expr(expr_string);
            
            if (kind == "exponentialrate")
            {
                expo_rate = update_parser::parse(expr_string, &vars_map_, &global_vars_map_);
            }
            
            if (kind == "invariant")
            {
                get_guys(expressions, &invariants);
            }
            if (init_node_id_ == node_id)
                start_nodes_.push_back(nodes_->size());
            nodes_->push_back(new node_t(node_id, to_array(&invariants),false, is_goal, expo_rate));
            
        }

        for (pugi::xml_node locs: templates.children("branchpoint"))
        {
            string string_id = locs.attribute("id").as_string();
            const int node_id = xml_id_to_int(string_id);
            node_edge_map.insert_or_assign(node_id, list<edge_t*>());
            nodes_->push_back(new node_t(node_id,array_t<constraint_t*>(0), true));
        }

        
        
        for (pugi::xml_node trans: templates.children("transition"))
        {
            string source = trans.child("source").attribute("ref").as_string();
            string target = trans.child("target").attribute("ref").as_string();

            int source_id = xml_id_to_int(source);
            int target_id = xml_id_to_int(target);
            
            list<constraint_t*> guards;
            list<update_t*> updates;
            expression* probability = nullptr;
            edge_channel* ec = nullptr;
            
            for (pugi::xml_node labels: trans.children("label"))
            {
                string kind = labels.attribute("kind").as_string();
                string expr_string = labels.child_value();

                
                if(kind == "guard")
                {
                    list<string> expressions = split_expr(expr_string);
                    get_guys(expressions, &guards);
                }
                else if (kind == "assignment")
                {
                    list<string> expressions = split_expr(expr_string, ',');
                    //cout << "\nASS0: " << expressions.size() << " " << expr_string <<"\n";
                    for(const auto& expr: expressions)
                    {
                        if (expr.empty())
                            continue;
                        
                        expression* e = update_parser::parse(expr, &vars_map_, &global_vars_map_);

                        string keyword = get_assignment_keyword(expr);
                        bool is_clock = false;

                        if(timers_map_.count(keyword) > 0)
                        {
                            is_clock = true;
                        }

                        
                        updates.push_back(new update_t(update_id++, get_timer_id(expr), is_clock, e));
                    }
                }
                else if (kind == "synchronisation")
                {
                    ec = new edge_channel();
                    // printf("\n !=!=!==!=!=!== %s \n | %d", expr_string.c_str(), !does_not_contain(expr_string, "!"));
                    if (!does_not_contain(expr_string, "!"))
                    {
                        ec->is_listener = false;
                    }
                    else
                    {
                        ec->is_listener = true;
                    }
                    string sync_keyword = replace_all(expr_string, "!", "");
                    sync_keyword = replace_all(sync_keyword, "?", "");
                    sync_keyword = replace_all(sync_keyword, " ", "");
                    
                    if (vars_map_.count(sync_keyword))
                    {
                        ec->channel_id = vars_map_.at(sync_keyword);
                    }
                    else if (global_vars_map_.count(sync_keyword))
                    {
                        ec->channel_id = global_vars_map_.at(sync_keyword);
                    }
                    else
                    {
                        THROW_LINE(sync_keyword + " NOT IN LOCAL, NOR GLOBAL MAP, comeon dude..");
                    }
                    
                }
                else if (kind == "probability")
                {
                    probability = update_parser::parse(expr_string, &vars_map_, &global_vars_map_);
                }
            }

            if (probability == nullptr) probability = expression::literal_expression(1.0);
            
            node_t* target_node = get_node(target_id, nodes_);
            edge_t* result_edge = nullptr;
            if (ec == nullptr)
            {
               result_edge = new edge_t(edge_id++, probability, target_node, to_array(&guards), to_array(&updates));
            }
            else
            {
                result_edge = new edge_t(edge_id++, probability, target_node, to_array(&guards), to_array(&updates), *ec);
            }

            //cout << "guard size: " << guards.size() << "\n";
            
            // if (guards.empty())
            //     result_edge = new edge_t(edge_id, probability, target_node, array_t<constraint_t*>(0));
            //result_edge->set_updates(&updates);
            
            node_edge_map.at(source_id).push_back(result_edge);
        }
        vars_map_.clear();
    }

    for(node_t* node: *nodes_)
    {
        //cout << "\n" << node->get_id() <<" HELLLLO!";
        node->set_edges(&node_edge_map.at(node->get_id()));
    }
    
    //TODO i broke plz fix :)
    //The stochastic model now expects an array of objects, rather than a array of object pointers.
    //This helps cut down on the number of times pointers need to be followed in the simulation.
    //Only reason it was like that before, was because we didnt know how to make the cuda-allocation code without it :)
    // - Bak ඞ
    array_t<node_t> start_nodes = array_t<node_t>(start_nodes_.size());

    

    int number_of_start_nodes = 0;
    for (int i : start_nodes_)
    {
        auto n_front = nodes_->begin();
        std::advance(n_front, i);
        start_nodes.arr()[number_of_start_nodes++] = **n_front;
    }

    return stochastic_model_t(start_nodes, to_array(timer_list_), to_array(var_list_), chan_id_);
}



__host__ stochastic_model_t uppaal_tree_parser::parse(char* file_path)
{
    try
    {
        return parse_xml(file_path);
    }
    catch (const std::runtime_error &ex)
    {
        cout << "Parse error: " << ex.what() << "\n";
        throw runtime_error("parse error");
    }
}