#include "hip/hip_runtime.h"

#include <iostream>
#include <filesystem>
#include <fstream>

#include "Visitors/domain_analysis_visitor.h"
#include "Visitors/pretty_visitor.h"
#include "UPPAALTreeParser/uppaal_tree_parser.h"
#include "Simulator/simulation_strategy.h"
#include "Simulator/stochastic_simulator.h"
#include "common/argparser.h"

#include "Domain/edge_t.h"
#include "Simulator/result_writer.h"

using namespace argparse;


int main(int argc, const char* argv[])
{
    hipFree(nullptr); //done to load cuda assembly, in case of dynamic linking 
    simulation_strategy strategy = {};
    
    ArgumentParser parser("supa_pc_strikes_argina.exe/cuda", "Argument parser example");

    parser.add_argument("-m", "--model", "Model xml file path", false);
    parser.add_argument("-b", "--block", "Number of block", false);
    parser.add_argument("-t", "--threads", "Number of threads", false);
    parser.add_argument("-a", "--amount", "Number of simulations", false);
    parser.add_argument("-c", "--count", "number of times to repeat simulations", false);
    parser.add_argument("-s", "--steps", "maximum number of steps per simulation", false);
    parser.add_argument("-p", "--maxtime", "Maximum number to progress in time (default=100)", false );
    parser.add_argument("-d", "--device", "What simulation to run (GPU (0) / CPU (1) / BOTH (2))", false);
    parser.add_argument("-u", "--cputhread", "The number of threads to use on the CPU", false);
    parser.add_argument("-w", "--write", "Write to file (\n / No output (0) \n / Console Summary (1) \n / File summary (2) \n / Console and File summary (3) \n / Console summary and File data (4) \n / File summary and File data (5) \n / Console summary, File summary, and File data (6) \n / Lite summary (7))", false);
    parser.add_argument("-o", "--output", "The path to output result file", false);
    parser.add_argument("-y", "--max", "Use max steps or time for limit simulation. (max steps (0) / max time (1) )", false);
    parser.add_argument("-v", "--verbose", "Enable pretty print of model (print model (0) / silent(1))", false);
    parser.enable_help();
    auto err = parser.parse(argc, argv);
    
    if (err) {
        std::cout << err << std::endl;
        return -1;
    }

    if (parser.exists("help")) {
        parser.print_help();
        return 0;
    }

    int mode = 0; // 0 = GPU, 1 = CPU, 2 = BOTH
    string o_path = std::filesystem::current_path();
    writer_modes write_mode; // 0 = file, 1 = console, 2 = both
    bool verbose = true;

    if (parser.exists("b")) strategy.block_n = parser.get<int>("b");
    if (parser.exists("t")) strategy.threads_n = parser.get<int>("t");
    if (parser.exists("a")) strategy.simulations_per_thread = parser.get<unsigned int>("a");
    if (parser.exists("c")) strategy.simulation_runs = parser.get<unsigned int>("c");
    if (parser.exists("s")) strategy.max_sim_steps = parser.get<unsigned int>("s");
    if (parser.exists("p")) strategy.max_time_progression = parser.get<double>("p");
    if (parser.exists("u")) strategy.cpu_threads_n = parser.get<unsigned int>("u");
    if (parser.exists("d")) mode = parser.get<int>("d");
    if (parser.exists("o")) o_path = o_path + "/" + parser.get<string>("o");
    if (parser.exists("w")) write_mode = static_cast<writer_modes>(parser.get<int>("w"));
    if (parser.exists("y")) strategy.use_max_steps = parser.get<int>("y") == 0;
    if (parser.exists("v")) verbose = parser.get<int>("v") == 0;
    
    
    stochastic_model_t model(array_t<node_t>(0), array_t<clock_variable>(0), array_t<clock_variable>(0),  0);
    
    if (parser.exists("m"))
    {
        uppaal_tree_parser tree_parser;
        string temp = parser.get<string>("m"); 
        char* writeable = new char[temp.size() + 1]; //TODO Move this fuckery inside parser
        std::copy(temp.begin(), temp.end(), writeable);
        writeable[temp.size()] = '\0';
        
        model = tree_parser.parse(writeable);

        delete[] writeable;
    }
    else
    {
        //TODO remove default model
        array_t<clock_variable> variable_arr = array_t<clock_variable>(2);
        variable_arr.arr()[0] = clock_variable(0, 10);
        variable_arr.arr()[1] = clock_variable(1, 5);
        
        array_t<constraint_t*> con0_arr = array_t<constraint_t*>(1);
        con0_arr.arr()[0] = constraint_t::less_equal_v(0, expression::literal_expression(10) );

        node_t node0 = node_t(0, con0_arr, false,false);
        node_t node1 = node_t(1, con0_arr, false,false);
        node_t node2 = node_t(2, con0_arr,false,true);

        std::list<update_t*> update_lst;
        array_t<update_t*> update_arr = to_array(&update_lst);
        
        edge_t* edge0_1 = new edge_t(0, expression::literal_expression(1), &node1, con0_arr, update_arr);
        edge_t* edge0_2 = new edge_t(1, expression::literal_expression(1), &node2, array_t<constraint_t*>(0), update_arr);
        edge_t* edge1_0 = new edge_t(2, expression::literal_expression(1), &node0, array_t<constraint_t*>(0), update_arr);

        array_t<clock_variable> timer_arr = array_t<clock_variable>(2);
        timer_arr.arr()[0] = clock_variable(0, 0.0);
        timer_arr.arr()[1] = clock_variable(1, 0.0);
        
        std::list<edge_t*> node0_lst;
        std::list<edge_t*> node1_lst;
        
        node0_lst.push_back(edge0_1);
        node0_lst.push_back(edge0_2);
        node0.set_edges(&node0_lst);

        node1_lst.push_back(edge1_0);
        node1.set_edges(&node1_lst);
        
        array_t<node_t> start_nodes = array_t<node_t>(1);
        start_nodes.arr()[0] = node0;

        model = stochastic_model_t(start_nodes, timer_arr, variable_arr, 5);
    }
    result_writer r_writer = result_writer(
        &o_path ,strategy,
        model.get_models_count(),
        model.get_variable_count(),
        write_mode);
    
    //Computers were not meant to speak.
    //You can speak when spoken to.
    if (verbose)
    {
        pretty_visitor p_visitor;
        domain_analysis_visitor d_visitor;
        p_visitor.visit(&model);
        d_visitor.visit(&model);
        printf("Max exp: %d | Max updates: %d\n", d_visitor.get_max_expression_depth(), d_visitor.get_max_update_width());
    }

    //0 == GPU, 1 == CPU, 2 == BOTH
    if (mode == 2 || mode == 0)
    {
        if (verbose) cout << "GPU SIMULATIONS STARTED! \n";
        stochastic_simulator::simulate_gpu(&model, &strategy, &r_writer, verbose);
        if (verbose) cout << "GPU SIMULATION DONE! \n";
    }
    if (mode > 0)
    {
        if (verbose) cout << "CPU SIMULATION STARTED! \n";
        stochastic_simulator::simulate_cpu(&model, &strategy, &r_writer, verbose);
        if(verbose) cout << "CPU SIMULATION DONE! \n";
    }
    return 0;
}
