#include "hip/hip_runtime.h"

#include <iostream>
#include <filesystem>
#include <fstream>

#include "Visitors/domain_analysis_visitor.h"
#include "Visitors/pretty_visitor.h"
#include "UPPAALTreeParser/uppaal_tree_parser.h"
#include "Simulator/simulation_strategy.h"
#include "Simulator/stochastic_simulator.h"
#include "common/argparser.h"

#include "Domain/edge_t.h"
#include "Simulator/writers/result_writer.h"
#include "Simulator/writers/result_manager.h"

using namespace argparse;

int main(int argc, const char* argv[])
{
    hipFree(nullptr); //done to load cuda assembly, in case of dynamic linking 
    simulation_strategy strategy = {};
    
    ArgumentParser parser("supa_pc_strikes_argina.exe/cuda", "Argument parser example");

    parser.add_argument("-a", "--amount", "Number of simulations", false);
    parser.add_argument("-b", "--block", "Number of block", false);
    parser.add_argument("-c", "--count", "number of times to repeat simulations", false);
    parser.add_argument("-d", "--device", "What simulation to run (GPU (0) / CPU (1) / BOTH (2))", false);
    parser.add_argument("-f", "--pretty", "Pretty print output file", false);
    parser.add_argument("-i", "--interval", "Interval to store trace. e.g. 10t = trace every 10 simulation seconds / 10s = every 10 step  (default = 1s). \n Parameter '-s' is used to specify the max number of steps to store.", false);
    parser.add_argument("-m", "--model", "Model xml file path", false);
    parser.add_argument("-o", "--output", "The path to output result file", false);
    parser.add_argument("-p", "--max_time", "Maximum number to progress in time (default=100)", false );
    parser.add_argument("-s", "--steps", "maximum number of steps per simulation", false);
    parser.add_argument("-t", "--threads", "Number of threads", false);
    parser.add_argument("-u", "--cputhread", "The number of threads to use on the CPU", false);
    parser.add_argument("-v", "--verbose", "Enable pretty print of model (print model (0) / silent(1))", false);
    parser.add_argument("-w", "--write", "Write mode \n / c = console summary  \n / f = file summary \n / d = file data dump \n / t = trace \n / l = lite summary \n / m = write model to file \n / r = write hit percentage to file", false);
    parser.add_argument("-y", "--max", "Use max steps or time for limit simulation. (max steps (0) / max time (1) )", false);
    parser.enable_help();
    auto err = parser.parse(argc, argv);
    
    if (err) {
        std::cout << err << std::endl;
        return -1;
    }

    if (parser.exists("help")) {
        parser.print_help();
        return 0;
    }

    int mode = 0; // 0 = GPU, 1 = CPU, 2 = BOTH
    string o_path = "./"; // std::filesystem::current_path();
    int write_mode = 0; // 0 = file, 1 = console, 2 = both
    bool verbose = true;

    if (parser.exists("a")) strategy.simulations_per_thread = parser.get<unsigned int>("a");
    if (parser.exists("b")) strategy.block_n = parser.get<int>("b");
    if (parser.exists("c")) strategy.simulation_runs = parser.get<unsigned int>("c");
    if (parser.exists("d")) mode = parser.get<int>("d");
    if (parser.exists("p")) strategy.max_time_progression = parser.get<double>("p");
    if (parser.exists("s")) strategy.max_sim_steps = parser.get<unsigned int>("s");
    if (parser.exists("t")) strategy.threads_n = parser.get<int>("t");
    if (parser.exists("u")) strategy.cpu_threads_n = parser.get<unsigned int>("u");
    if (parser.exists("v")) verbose = parser.get<int>("v") == 0;
    if (parser.exists("w")) write_mode = result_writer::parse_mode(parser.get<std::string>("w"));
    if (parser.exists("y")) strategy.use_max_steps = parser.get<int>("y") == 0;
    if (parser.exists("o")) o_path = o_path + parser.get<string>("o");
    else o_path = o_path + "output";

    if(write_mode & trace) //Trace settings, only if trace is enabled
    {
        strategy.trace_settings =  result_manager::parse_interval(parser.exists("i")
            ? parser.get<std::string>("i")
            : "1s");
    }
    else strategy.trace_settings.mode = trace_interval::disabled;

    uppaal_tree_parser tree_parser = uppaal_tree_parser();
    stochastic_model_t model(array_t<node_t*>(0), array_t<clock_variable>(0), array_t<clock_variable>(0));
    
    if (parser.exists("m"))
    {
        string input_file_path = parser.get<string>("m"); 
        model = tree_parser.parse(input_file_path);
    }
    else
    {
        //TODO remove default model
        array_t<clock_variable> variable_arr = array_t<clock_variable>(2);
        variable_arr.arr()[0] = clock_variable(0, 10);
        variable_arr.arr()[1] = clock_variable(1, 5);
        
        array_t<constraint_t> con0_arr = array_t<constraint_t>(1);
        con0_arr.arr()[0] = *constraint_t::less_equal_v(0, expression::literal_expression(10) );

        node_t node0 = node_t(0, con0_arr, false,false);
        node_t node1 = node_t(1, con0_arr, false,false);
        node_t node2 = node_t(2, con0_arr,false,true);

        std::list<update_t> update_lst;
        array_t<update_t> update_arr = to_array(&update_lst);
        
        edge_t edge0_1 = edge_t(0, expression::literal_expression(1), &node1, con0_arr, update_arr);
        edge_t edge0_2 = edge_t(1, expression::literal_expression(1), &node2, array_t<constraint_t>(0), update_arr);
        edge_t edge1_0 = edge_t(2, expression::literal_expression(1), &node0, array_t<constraint_t>(0), update_arr);

        array_t<clock_variable> timer_arr = array_t<clock_variable>(2);
        timer_arr.arr()[0] = clock_variable(0, 0.0);
        timer_arr.arr()[1] = clock_variable(1, 0.0);
        
        std::list<edge_t> node0_lst;
        std::list<edge_t> node1_lst;
        
        node0_lst.push_back(edge0_1);
        node0_lst.push_back(edge0_2);
        node0.set_edges(&node0_lst);

        node1_lst.push_back(edge1_0);
        node1.set_edges(&node1_lst);
        
        array_t<node_t*> start_nodes = array_t<node_t*>(1);
        start_nodes.arr()[0] = &node0;

        model = stochastic_model_t(start_nodes, timer_arr, variable_arr);
    }
    result_writer r_writer = result_writer(&o_path ,strategy,
        model.get_models_count(),
        model.get_variable_count(),
        write_mode);

    if(write_mode & trace || write_mode & model_out)
    {
        r_writer.write_model(tree_parser.get_nodes_with_name(),
            tree_parser.get_subsystems());
    }
    
    //Computers were not meant to speak.
    //You can speak when spoken to.
    if (write_mode & pretty_out || verbose)
    {
        pretty_visitor p_visitor = pretty_visitor(verbose, write_mode & pretty_out, o_path + "_pretty_model.txt");
        p_visitor.visit(&model);
    }

    //0 == GPU, 1 == CPU, 2 == BOTH
    if (mode == 2 || mode == 0)
    {
        if (verbose) cout << "GPU SIMULATIONS STARTED! \n";
        r_writer.clear();
        stochastic_simulator::simulate_gpu(&model, &strategy, &r_writer, verbose);
        if (verbose) cout << "GPU SIMULATION DONE! \n";
    }
    if (mode > 0)
    {
        if (verbose) cout << "CPU SIMULATION STARTED! \n";
        r_writer.clear();
        stochastic_simulator::simulate_cpu(&model, &strategy, &r_writer, verbose);
        if(verbose) cout << "CPU SIMULATION DONE! \n";
    }
    return 0;
}
