#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "dog.h"

dog::dog(int id) {
    this->id_ = id;
}

__host__ __device__ int dog::get_id() {
    return this->id_;
}
__host__ __device__ void dog::bork() {
    printf("%d BORKed!", this->get_id());
}

__global__ void print_dog(dog* dog_, int* out_d) {
    *out_d = dog_->get_id();
    dog_->bork();
}

int main() {

    dog *test; // Host variable
    dog *test_d; // Device Variable

    test = new dog(1);
    
    int *out; // Host variable 
    int *out_d; // Device Variable
    
    out = (int*)malloc(sizeof(int));
    hipMalloc((void**)&test_d, sizeof(dog));
    hipMalloc((void**)&out_d, sizeof(int));

    hipMemcpy(test_d, test, sizeof(dog), hipMemcpyHostToDevice);

    print_dog<<<1,1>>>(test_d, out_d);

    hipMemcpy(out, out_d, sizeof(int), hipMemcpyDeviceToHost);
    
    printf("%d", *out);
    
    hipFree(test_d);
    hipFree(out_d);

    free(out);

    

    return 0;
}