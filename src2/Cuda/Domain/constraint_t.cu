﻿#include "constraint_t.h"


GPU CPU double cuda_abs(const double f)
{
    return f < 0 ? -f : f;
}


constraint_t::constraint_t(const logical_operator_t type, const int timer_id1, const int timer_id2, const float value)
{
    this->type_ = type;
    this->timer_id1_ = timer_id1;
    this->timer_id2_ = timer_id2;
    this->value_ = value;
}

CPU GPU bool constraint_t::evaluate(const lend_array<clock_variable>* timers) const
{
    const double v1 = timers->at(this->timer_id1_)->get_temp_time();
    const double v2 = this->timer_id2_ == NO_ID
                ? static_cast<double>(this->value_)
                : timers->at(this->timer_id2_)->get_temp_time();
    
    switch(this->type_)
    {
        case logical_operator_t::less_equal_t: return v1 <= v2;
        case logical_operator_t::greater_equal_t: return v1 >= v2;
        case logical_operator_t::less_t: return v1 < v2;
        case logical_operator_t::greater_t: return v1 > v2;
        case logical_operator_t::equal_t: return cuda_abs(v1 - v2) < 0.005; //v1 == v2;
        case logical_operator_t::not_equal_t: return cuda_abs(v1 - v2) >= 0.005; //v1 != v2;
    }
    return false;
}

CPU GPU bool constraint_t::check_max_time_progression(const lend_array<clock_variable>* timer_arr, double* out_max_progression) const
{
    if(this->timer_id2_ != NO_ID) return false;
    if(this->type_ == logical_operator_t::less_t || this->type_ == logical_operator_t::less_equal_t)
    {
        const double time = timer_arr->at(this->timer_id1_)->get_time();
        const double value = static_cast<double>(this->value_);

        double diff = value - time;
        if(diff < 0) diff = 0.0;
        (*out_max_progression) = diff;
        return true;
    }
    
    return false;
}

void constraint_t::accept(visitor* v)
{
    return;
}

void constraint_t::pretty_print() const
{
    printf("Constraint type: %s | Timer 1 id: %3d | Timer 2 id: %3d | value: %10f \n", constraint_t::logical_operator_to_string(this->type_).c_str(),
           this->timer_id1_, this->timer_id2_, this->value_);
}

void constraint_t::cuda_allocate(constraint_t** pointer, const allocation_helper* helper) const
{
    hipMalloc(pointer, sizeof(constraint_t));
    helper->free_list->push_back(*pointer);
    const constraint_t con = constraint_t(this->type_, this->timer_id1_, this->timer_id2_, this->value_);
    hipMemcpy(*pointer, &con, sizeof(constraint_t), hipMemcpyHostToDevice);
}

void constraint_t::cuda_allocate_2(constraint_t* cuda_pointer, const allocation_helper* helper) const
{
    const constraint_t con = constraint_t(this->type_, this->timer_id1_, this->timer_id2_, this->value_);
    hipMemcpy(cuda_pointer, &con, sizeof(constraint_t), hipMemcpyHostToDevice);
}

std::string constraint_t::logical_operator_to_string(const logical_operator_t op)
{
    switch (op)
    {
    case logical_operator_t::less_equal_t:
        return "<=";
    case logical_operator_t::greater_equal_t:
        return ">=";
    case logical_operator_t::less_t:
        return "<";
    case logical_operator_t::greater_t:
        return ">";
    case logical_operator_t::equal_t:
        return "==";
    case logical_operator_t::not_equal_t:
        return "!=";
    default:
        return "not a boolean operator";
    }
}

//! LESS THAN OR EQUAL
constraint_t* constraint_t::less_equal_v(const int timer_id, const float value)
{
    return new constraint_t{logical_operator_t::less_equal_t, timer_id, NO_ID, value};
}

constraint_t* constraint_t::less_equal_t(const int timer_id, const int timer_id2)
{
    return new constraint_t{logical_operator_t::less_equal_t, timer_id, timer_id2, UNUSED_VALUE};

}

//! GREATER THAN OR EQUAL
constraint_t* constraint_t::greater_equal_v(const int timer_id, const float value)
{
    return new constraint_t{logical_operator_t::greater_equal_t, timer_id, NO_ID, value};

}

constraint_t* constraint_t::greater_equal_t(const int timer_id, const int timer_id2)
{
    return new constraint_t{logical_operator_t::greater_equal_t, timer_id, timer_id2, UNUSED_VALUE};
}

//! LESS THAN
constraint_t* constraint_t::less_v(const int timer_id, const float value)
{
    return new constraint_t{logical_operator_t::less_t, timer_id, NO_ID, value};

}

constraint_t* constraint_t::less_t(const int timer_id, const int timer_id2)
{
    return new constraint_t{logical_operator_t::less_t, timer_id, timer_id2, UNUSED_VALUE};
}

//! GREATER THAN
constraint_t* constraint_t::greater_v(const int timer_id, const float value)
{
    return new constraint_t{logical_operator_t::greater_t, timer_id, NO_ID, value};

}

constraint_t* constraint_t::greater_t(const int timer_id, const int timer_id2)
{
    return new constraint_t{logical_operator_t::greater_t, timer_id, timer_id2, UNUSED_VALUE};
}

//! equal
constraint_t* constraint_t::equal_v(const int timer_id, const float value)
{
    return new constraint_t{logical_operator_t::equal_t, timer_id, NO_ID, value};

}

constraint_t* constraint_t::equal_t(const int timer_id, const int timer_id2)
{
    return new constraint_t{logical_operator_t::equal_t, timer_id, timer_id2, UNUSED_VALUE};
}

//! NOT EQUAL
constraint_t* constraint_t::not_equal_v(const int timer_id, const float value)
{
    return new constraint_t{logical_operator_t::not_equal_t, timer_id, NO_ID, value};
}

constraint_t* constraint_t::not_equal_t(const int timer_id, const int timer_id2)
{
    return new constraint_t{logical_operator_t::not_equal_t, timer_id, timer_id2, UNUSED_VALUE};
}
