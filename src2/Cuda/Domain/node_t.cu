﻿#include "node_t.h"

#include "CudaSimulator.h"

node_t::node_t(node_t* source, constraint_t* invariant, array_t<edge_t*> edges)
{
    this->id_ = source->id_;
    this->is_branch_point_ = source->is_branch_point_;
    this->invariant_ = invariant;
    this->is_goal_ = source->is_goal_;
    this->edges_ = edges;
}

node_t::node_t(const int id, const bool is_branch_point, constraint_t* invariant, const bool is_goal)
{
    this->id_ = id;
    this->is_goal_ = is_goal;
    this->invariant_ = invariant;
    this->is_branch_point_ = is_branch_point;
    this->edges_ = array_t<edge_t*>(0);
}

GPU CPU int node_t::get_id() const
{
    return this->id_;
}

void node_t::set_edges(std::list<edge_t*>* list)
{
    this->edges_ = to_array(list);
}

CPU GPU lend_array<edge_t*> node_t::get_edges()
{
    return lend_array<edge_t*>(&this->edges_);
}

CPU GPU bool node_t::is_goal_node() const
{
    return this->is_goal_;
}

GPU bool node_t::evaluate_invariants(const lend_array<clock_timer_t>* timers) const
{
    if(this->invariant_ == nullptr) return true;
    return this->invariant_->evaluate(timers);
}

void node_t::accept(visitor* v)
{
    const lend_array<edge_t*> edges = this->get_edges();
    v->visit(this->invariant_);
    for (int i = 0; i < edges.size(); ++i)
    {
        v->visit(*edges.at(i));
    }
    for (int i = 0; i < edges.size(); ++i)
    {
        v->visit(edges.get(i)->get_dest());
    }
}

void node_t::cuda_allocate(node_t** pointer, const allocation_helper* helper)
{
    if(helper->node_map->count(this) == 1) return;
    hipMalloc(pointer, sizeof(node_t));
    helper->free_list->push_back(*pointer);
    helper->node_map->insert(std::pair<node_t*, node_t*>(this, *pointer) );
    
    std::list<edge_t*> edge_lst;
    for (int i = 0; i < this->edges_.size(); ++i)
    {
        edge_t* edge_p = nullptr;
        this->edges_.get(i)->cuda_allocate(&edge_p, helper);
        edge_lst.push_back(edge_p);
    }
    constraint_t* invariant_p = nullptr;
    if (this->invariant_ != nullptr)
    {
        this->invariant_->cuda_allocate(&invariant_p, helper);
    }
    
    const node_t result(this, invariant_p, cuda_to_array(&edge_lst, helper->free_list));
    hipMemcpy(*pointer, &result, sizeof(node_t), hipMemcpyHostToDevice);
}

CPU GPU bool node_t::is_branch_point() const
{
    return this->is_branch_point_;
}

GPU double node_t::max_time_progression(const lend_array<clock_timer_t>* timers, double max_progression) const
{
    if(this->invariant_ == nullptr)
    {
        return max_progression;
    }
    
    return this->invariant_->max_time_progression(timers, max_progression); 
}
