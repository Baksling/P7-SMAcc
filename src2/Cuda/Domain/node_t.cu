﻿#include "node_t.h"

node_t::node_t(const int id, constraint_t* invariant, const bool is_goal)
{
    this->id_ = id;
    this->is_goal_ = is_goal;
    this->invariant_ = invariant;
    this->edges_ = array_t<edge_t>(0);
}

void node_t::set_edges(std::list<edge_t>* list)
{
    this->edges_ = to_array(list);
}

GPU lend_array<edge_t> node_t::get_edges()
{
    return lend_array<edge_t>(&this->edges_);
}

GPU bool node_t::is_goal_node() const
{
    return this->is_goal_;
}


GPU bool node_t::evaluate_invariants(const lend_array<timer_t>* timers) const
{
    if(this->invariant_ == nullptr) return true;
    return this->invariant_->evaluate(timers);
}
