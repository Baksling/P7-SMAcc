﻿#include "node_t.h"

node_t::node_t(const int id, constraint_t* invariant, const bool is_goal, const bool is_branch)
{
    this->id_ = id;
    this->is_goal_ = is_goal;
    this->is_branch_ = is_branch;
    this->invariant_ = invariant;
    this->edges_ = array_t<edge_t>(0);
}

void node_t::set_edges(std::list<edge_t>* list)
{
    this->edges_ = to_array(list);
}

GPU lend_array<edge_t> node_t::get_edges()
{
    return lend_array<edge_t>(&this->edges_);
}

GPU bool node_t::is_goal_node() const
{
    return this->is_goal_;
}


GPU bool node_t::evaluate_invariants(const lend_array<timer_t>* timers) const
{
    if(this->invariant_ == nullptr) return true;
    return this->invariant_->evaluate(timers);
}

void node_t::accept(visistor& v)
{
    const lend_array<edge_t> edges = this->get_edges();
    for (int i = 0; i < edges.size(); ++i)
    {
        v.visit(edges.at(i));
    }
    v.visit(this->invariant_);
}

int node_t::get_id() const
{
    return this->id_;
}

bool node_t::is_branch() const
{
    return this->is_branch_;
}
