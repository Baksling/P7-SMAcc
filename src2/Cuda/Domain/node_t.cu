﻿#include "node_t.h"

node_t::node_t(node_t* source, const array_t<constraint_t*> invariant, const array_t<edge_t*> edges)
{
    this->id_ = source->id_;
    this->is_branch_point_ = source->is_branch_point_;
    this->invariants_ = invariant;
    this->is_goal_ = source->is_goal_;
    this->edges_ = edges;
}

node_t::node_t(const int id, const array_t<constraint_t*> invariants, const bool is_branch_point, const bool is_goal)
{
    this->id_ = id;
    this->is_goal_ = is_goal;
    this->invariants_ = invariants;
    this->is_branch_point_ = is_branch_point;
    this->edges_ = array_t<edge_t*>(0);
}

GPU CPU int node_t::get_id() const
{
    return this->id_;
}

void node_t::set_edges(std::list<edge_t*>* list)
{
    this->edges_ = to_array(list);
}

CPU GPU lend_array<edge_t*> node_t::get_edges()
{
    return lend_array<edge_t*>(&this->edges_);
}

CPU GPU bool node_t::is_goal_node() const
{
    return this->is_goal_;
}

CPU GPU bool node_t::evaluate_invariants(const lend_array<clock_timer_t>* timers) const
{
    for (int i = 0; i < this->invariants_.size(); ++i)
    {
        if(!this->invariants_.get(i)->evaluate(timers))
            return false;
    }

    return true;
}

void node_t::accept(visitor* v) const
{
    //visit node constraints
    for (int i = 0; i < this->invariants_.size(); ++i)
    {
        printf("    ");
        v->visit(this->invariants_.get(i));
    }

    //visit edges
    for (int i = 0; i < this->edges_.size(); ++i)
    {
        v->visit(this->edges_.get(i));
    }

    //visit edge destinations
    for (int i = 0; i < this->edges_.size(); ++i)
    {
        v->visit(this->edges_.get(i)->get_dest());
    }
}

void node_t::cuda_allocate(node_t** pointer, const allocation_helper* helper)
{
    if(helper->node_map->count(this) == 1) return;
    hipMalloc(pointer, sizeof(node_t));
    helper->free_list->push_back(*pointer);
    helper->node_map->insert(std::pair<node_t*, node_t*>(this, *pointer) );
    
    std::list<edge_t*> edge_lst;
    for (int i = 0; i < this->edges_.size(); ++i)
    {
        edge_t* edge_p = nullptr;
        this->edges_.get(i)->cuda_allocate(&edge_p, helper);
        edge_lst.push_back(edge_p);
    }

    std::list<constraint_t*> invariant_lst;
    for (int i = 0; i < this->invariants_.size(); ++i)
    {
        constraint_t* invariant_p = nullptr;
        this->invariants_.get(i)->cuda_allocate(&invariant_p, helper);
        invariant_lst.push_back(invariant_p);
    }
    
    const node_t result(this,
        cuda_to_array(&invariant_lst, helper->free_list),
        cuda_to_array(&edge_lst, helper->free_list));
    hipMemcpy(*pointer, &result, sizeof(node_t), hipMemcpyHostToDevice);
}


void node_t::cuda_allocate_2(node_t* cuda_p, const allocation_helper* helper) const
{
    edge_t* edges = nullptr; 
    hipMalloc(&edges,sizeof(edge_t)*this->edges_.size());
    helper->free_list->push_back(edges);

    for (int i = 0; i < this->edges_.size(); ++i)
    {
        this->edges_.get(i)->cuda_allocate_2(&edges[i], helper);
    }

    constraint_t* invariants = nullptr; 
    hipMalloc(&invariants,sizeof(constraint_t)*this->invariants_.size());
    helper->free_list->push_back(cuda_p);
    for (int i = 0; i < this->invariants_.size(); ++i)
    {
        this->invariants_.get(i)->cuda_allocate_2(&invariants[i], helper);
    }
    
    // const node_t result(this,
    //     array_t<constraint_t>(invariants, this->invariants_.size()),
    //     array_t<edge_t>(edges, this->edges_.size()));
    
    // hipMemcpy(cuda_p, &result, sizeof(node_t), hipMemcpyHostToDevice);
}

CPU GPU bool node_t::is_branch_point() const
{
    return this->is_branch_point_;
}

CPU GPU double node_t::max_time_progression(const lend_array<clock_timer_t>* timers, double max_progression) const
{
    if(this->invariants_.size() <= 0) return max_progression;

    for (int i = 0; i < this->invariants_.size(); ++i)
    {
        const double temp_progression = this->invariants_.get(i)->max_time_progression(timers, max_progression);
        max_progression = temp_progression < max_progression ? temp_progression : max_progression;
    }
    
    return max_progression; 
}
