﻿#include "node_t.h"

node_t::node_t(const int id, constraint_t* invariant, const bool is_goal)
{
    this->id_ = id;
    this->is_goal_ = is_goal;
    this->invariant_ = invariant;
    this->edges_ = array_t<edge_t>(0);
}

GPU CPU int node_t::get_id() const
{
    return this->id_;
}

void node_t::set_edges(std::list<edge_t>* list)
{
    this->edges_ = to_array(list);
}

GPU lend_array<edge_t> node_t::get_edges()
{
    return lend_array<edge_t>(&this->edges_);
}

GPU bool node_t::is_goal_node() const
{
    return this->is_goal_;
}


GPU bool node_t::evaluate_invariants(const lend_array<clock_timer_t>* timers) const
{
    if(this->invariant_ == nullptr) return true;
    return this->invariant_->evaluate(timers);
}

GPU double node_t::max_time_progression(const lend_array<clock_timer_t>* timers, double max_progression) const
{
    if(this->invariant_ == nullptr)
    {
        return max_progression;
    }
    
    return this->invariant_->max_time_progression(timers, max_progression); 
}