﻿// ReSharper disable CppClangTidyBugproneSizeofExpression
#include "stochastic_model_t.h"

stochastic_model_t::stochastic_model_t(
    const array_t<node_t*> models,
    const array_t<clock_variable> timers,
    const array_t<clock_variable> variables,
    const unsigned channel_count)
{
    this->models_ = models;
    this->timers_ = timers;
    this->variables_ = variables;
    this->channel_count_ = channel_count;
}

CPU GPU unsigned stochastic_model_t::get_channel_count() const
{
    return this->channel_count_;
}

unsigned stochastic_model_t::get_variable_count() const
{
    return this->variables_.size();
}

unsigned stochastic_model_t::get_timer_count() const
{
    return this->timers_.size();
}

unsigned stochastic_model_t::get_models_count() const
{
    return this->models_.size();
}

void stochastic_model_t::cuda_allocate(stochastic_model_t* device, allocation_helper* helper) const
{
    //allocate models!
    node_t* node_store = nullptr;
    node_t** node_store_p = nullptr;
    node_t** local_node_store_p = static_cast<node_t**>(malloc(sizeof(node_t*)*this->models_.size()));
    helper->allocate(&node_store, sizeof(node_t)*this->models_.size());
    helper->allocate(&node_store_p, sizeof(node_t*)*this->models_.size());
    const array_t<node_t*> node_arr = array_t<node_t*>(node_store_p, this->models_.size());

    for (int i = 0; i < this->models_.size(); ++i)
    {
        this->models_.get(i)->cuda_allocate(&node_store[i], helper);
        local_node_store_p[i] = &node_store[i];
    }
    hipMemcpy(node_store_p, local_node_store_p, sizeof(node_t*)*this->models_.size(), hipMemcpyHostToDevice);
    free(local_node_store_p);

    //allocate clocks
    clock_variable* clock_store = nullptr;
    helper->allocate(&clock_store, sizeof(clock_variable)*this->timers_.size());
    const array_t<clock_variable> clock_arr = array_t<clock_variable>(clock_store, this->timers_.size());
    for (int i = 0; i < this->timers_.size(); ++i)
    {
        this->timers_.at(i)->cuda_allocate(&clock_store[i], helper);
    }

    //allocate clocks
    clock_variable* variable_store = nullptr;
    helper->allocate(&variable_store, sizeof(clock_variable)*this->variables_.size());
    const array_t<clock_variable> variable_arr = array_t<clock_variable>(variable_store, this->variables_.size());
    for (int i = 0; i < this->variables_.size(); ++i)
    {
        this->variables_.at(i)->cuda_allocate(&variable_store[i], helper);
    }
    
    const stochastic_model_t result = stochastic_model_t(node_arr, clock_arr, variable_arr, this->channel_count_);
    hipMemcpy(device, &result, sizeof(stochastic_model_t), hipMemcpyHostToDevice);
}

void stochastic_model_t::accept(visitor* v) const
{
    //visit timers
    for (int i = 0; i < this->timers_.size(); ++i)
    {
        v->visit(this->timers_.at(i));
    }

    //visit variables
    for (int i = 0; i < this->variables_.size(); ++i)
    {
        v->visit(this->variables_.at(i));
    }
    
    //visit models
    for (int i = 0; i < this->models_.size(); ++i)
    {
        v->visit(this->models_.get(i));
    }
}

// ReSharper disable once CppMemberFunctionMayBeStatic
void stochastic_model_t::pretty_print() const
{
    //TODO fix plz :)
    return;
} 
