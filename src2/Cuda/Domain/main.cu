﻿
#include "common.h"
#include <iostream>
#include "pretty_visitor.h"
#include "../UPPAALTreeParser/uppaal_tree_parser.h"

#include "../Simulator/simulation_strategy.h"
#include "../Simulator/stochastic_simulator.h"
#include "argparser.h"

using namespace argparse;

int main(int argc, const char* argv[])
{
    simulation_strategy strategy = {};
    
    ArgumentParser parser("supa_pc_strikes_argina.exe/cuda", "Argument parser example");

    parser.add_argument("-m", "--model", "Model xml file path", false);
    parser.add_argument("-b", "--block", "Number of block", false);
    parser.add_argument("-t", "--threads", "Number of threads", false);
    parser.add_argument("-a", "--amount", "Number of simulations", false);
    parser.add_argument("-c", "--count", "number of times to repeat simulations", false);
    parser.add_argument("-s", "--steps", "maximum number of steps per simulation", false);
    parser.add_argument("-p", "--maxtime", "Maximum numper to progrss in time", false );
    parser.enable_help();
    auto err = parser.parse(argc, argv);
    
    if (err) {
        std::cout << err << std::endl;
        return -1;
    }

    if (parser.exists("help")) {
        parser.print_help();
        return 0;
    }

    


    if (parser.exists("b")) strategy.block_n = parser.get<int>("b");
    if (parser.exists("t")) strategy.threads_n = parser.get<int>("t");
    if (parser.exists("a")) strategy.simulation_amounts = parser.get<unsigned int>("a");
    if (parser.exists("c")) strategy.sim_count = parser.get<int>("c");
    if (parser.exists("s")) strategy.max_sim_steps = parser.get<unsigned int>("s");
    if (parser.exists("p")) strategy.max_time_progression = parser.get<double>("p");
    
    std::cout << "Fuck you\n";

    constraint_t* con0 = constraint_t::less_equal_v(0, 10.0f);
    constraint_t* con1 = constraint_t::greater_equal_t(0, 10.0f);
    // constraint_t con1 = constraint_t::less_equal_v(1, 10.0f);
    // constraint_t con2 = constraint_t::greater_equal_v(0, 0.0f);

    array_t<constraint_t*> con0_arr = array_t<constraint_t*>(1);
    array_t<constraint_t*> con1_arr = array_t<constraint_t*>(1);

    con0_arr.arr()[0] = con0;
    con1_arr.arr()[0] = con1;
    node_t node0 = node_t(0, con0_arr, false,false);
    node_t node1     = node_t(1, con0_arr, false,false);
    node_t node2 = node_t(2, array_t<constraint_t*>(0),false,true);

    edge_t* edge0_1 = new edge_t(0, 1, &node1, con1_arr);
    edge_t* edge0_2 = new edge_t(1, 1, &node2, array_t<constraint_t*>(0));
    edge_t* edge1_0 = new edge_t(2, 1, &node0, array_t<constraint_t*>(0));

    clock_timer_t timer1 = clock_timer_t(0, 0.0);
    clock_timer_t timer2 = clock_timer_t(1, 0.0);

    std::list<clock_timer_t*> clock_lst;
    clock_lst.push_back(&timer1);
    clock_lst.push_back(&timer2);
    
    std::list<edge_t*> node0_lst;
    std::list<edge_t*> node1_lst;
    
    node0_lst.push_back(edge0_1);
    node0_lst.push_back(edge0_2);
    node0.set_edges(&node0_lst);

    node1_lst.push_back(edge1_0);
    node1.set_edges(&node1_lst);

    


    pretty_visitor visitor;
    stochastic_model_t model(&node0, to_array(&clock_lst));
    if (parser.exists("m"))
    {
        printf("USING PARSER\n");
        uppaal_tree_parser tree_parser;
        string temp = parser.get<string>("m");
        char* writeable = new char[temp.size() + 1];
        std::copy(temp.begin(), temp.end(), writeable);
        writeable[temp.size()] = '\0';
        
        printf("Fuck thi");
        model = tree_parser.parse(writeable);

        delete[] writeable;
    }
    visitor.visit(&model);

    // stochastic_simulator::simulate_cpu(&model, &strategy);
    stochastic_simulator::simulate_gpu(&model, &strategy);
    
    std::cout << "pully porky\n";

    return 0;
}
