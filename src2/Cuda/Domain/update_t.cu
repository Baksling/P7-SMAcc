﻿#include "update_t.h"

update_t::update_t(const int id, const int timer_id, const double timer_value)
{
    this->id_ = id;
    this->timer_id_ = timer_id;
    this->timer_value_ = timer_value;
}

GPU void update_t::update_timer(const lend_array<timer_t>* timers) const
{
    timers->at(this->timer_id_)->set_time(this->timer_value_);
}

void update_t::accept(visistor& v)
{
    return;
}

int update_t::get_timer_id() const
{
    return this->timer_id_;
}

float update_t::get_timer_value() const
{
    return this->timer_value_;
}

int update_t::get_id() const
{
    return this->id_;
}
