﻿#include "update_t.h"

update_t::update_t(const update_t* source, expression* expression)
{
    this->id_ = source->id_;
    this->variable_id_ = source->variable_id_;
    this->is_clock_update_ = source->is_clock_update_;
    this->expression_ = expression;
}

update_t::update_t(const int id, const int variable_id, const bool is_clock_update, expression* expression)
{
    this->id_ = id;
    this->variable_id_ = variable_id;
    this->is_clock_update_ = is_clock_update;
    this->expression_ = expression;
}

CPU GPU void update_t::apply_update(simulator_state* state) const
{
    const double value = state->evaluate_expression(this->expression_);
    if(this->is_clock_update_)
    {
        state->get_timers().at(this->variable_id_)->set_time(value);
    }
    else
    {
        //value is rounded correctly by adding 0.5. casting always rounds down.
        state->get_variables().at(this->variable_id_)->set_time(value);  // NOLINT(bugprone-incorrect-roundings)
    }
}

CPU GPU void update_t::apply_temp_update(simulator_state* state) const
{
    const double value = state->evaluate_expression(this->expression_);
    if(this->is_clock_update_)
    {
        state->get_timers().at(this->variable_id_)->set_temp_time(value);
    }
    else
    {
        //value is rounded correctly by adding 0.5. casting always rounds down.
        state->get_variables().at(this->variable_id_)
            ->set_temp_time(value);  // NOLINT(bugprone-incorrect-roundings)
    }
}

void update_t::reset_temp_update(const simulator_state* state) const
{
    if(this->is_clock_update_)
    {
        state->get_timers().at(this->variable_id_)->reset_temp_time();
    }
    else
    {
        //value is rounded correctly by adding 0.5. casting always rounds down.
        state->get_variables().at(this->variable_id_)->reset_temp_time();  // NOLINT(bugprone-incorrect-roundings)
    }
}

void update_t::accept(visitor* v) const
{
    v->visit(this->expression_);
}

void update_t::pretty_print(std::ostream& os) const
{
    std::string temp, temp2;
    if (is_clock_update_) temp = "Clock " + std::to_string(this->variable_id_) + " = ";
    else temp = "Variable " + std::to_string(this->variable_id_) + " = ";

    temp2 = this->expression_->to_string();

    os << temp + temp2 + "\n";
    
    //printf("%s %s\n", temp.c_str(), temp2.c_str());
    
    //std::cout << expression_->to_string();
    //printf("Update id: %3d | Timer id: %3d\n", this->id_, this->variable_id_);
}

void update_t::cuda_allocate(update_t* cuda, const allocation_helper* helper) const
{
    expression* expr = nullptr;
    hipMalloc(&expr, sizeof(expression));
    helper->free_list->push_back(expr);
    this->expression_->cuda_allocate(expr, helper);
    
    const update_t copy = update_t(this, expr);
    hipMemcpy(cuda, &copy, sizeof(update_t), hipMemcpyHostToDevice);
}


unsigned update_t::get_expression_depth() const
{
    return this->expression_->get_depth();
}
