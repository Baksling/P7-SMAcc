﻿#include "edge_t.h"


edge_t::edge_t(const int id, const float weight, node_t* dest, constraint_t* guard)
{
    this->id_ = id;
    this->dest_ = dest;
    this->weight_ = weight;
    this->updates_ = array_t<update_t>(0);
    this->guard_ = guard;
}

GPU float edge_t::get_weight() const
{
    return this->weight_;
}

void edge_t::set_updates(std::list<update_t>* updates)
{
    this->updates_ = to_array(updates);
}

GPU bool edge_t::evaluate_constraints(const lend_array<timer_t>* timers) const
{
    if(this->guard_ == nullptr) return true;
    return this->guard_->evaluate(timers);
}

void edge_t::accept(visistor& v)
{
    v.visit(this->guard_);
    const lend_array<update_t> updates = this->get_updates();
    for (int i = 0; i < updates.size(); ++i)
    {
        v.visit(updates.at(i));
    }
}

node_t* edge_t::get_dest_node() const
{
    return this->dest_;
}

int edge_t::get_id() const
{
    return this->id_;
}

lend_array<update_t> edge_t::get_updates()
{
    const lend_array<update_t> result (&this->updates_);
    return result;
}
