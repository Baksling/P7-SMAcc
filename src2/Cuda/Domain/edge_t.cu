﻿#include "edge_t.h"


edge_t::edge_t(const int id, const float weight, node_t* dest, constraint_t* guard)
{
    this->id_ = id;
    this->dest_ = dest;
    this->weight_ = weight;
    this->updates_ = array_t<update_t>(0);
    this->guard_ = guard;
}

GPU float edge_t::get_weight() const
{
    return this->weight_;
}

void edge_t::set_updates(std::list<update_t>* updates)
{
    this->updates_ = to_array(updates);
}

GPU bool edge_t::evaluate_constraints(const lend_array<clock_timer_t>* timers) const
{
    if(this->guard_ == nullptr) return true;
    return this->guard_->evaluate(timers);
}
