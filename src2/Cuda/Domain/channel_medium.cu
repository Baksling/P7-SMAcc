﻿#include "channel_medium.h"
#include "simulator_state.h"

CPU GPU void channel_listener::broadcast(simulator_state* sim_state) const
{
    sim_state->medium->remove(this->state->current_node);
    
    node_t* dest = edge->get_dest();

    //update state
    this->state->current_node = dest;
    this->state->reached_goal = dest->is_goal_node();
    
    this->edge->execute_updates(sim_state);
    
    sim_state->medium->add(this->state);
}

CPU GPU void channel_stack::add(model_state* state, edge_t* edge)
{
    this->listeners[this->count] = channel_listener{state, edge};
    this->count++;
}

CPU GPU void channel_stack::remove(const node_t* node)
{
    if(this->count == 0) return; //no entries

    const channel_listener* end = &this->listeners[this->count -1];

    if(end->state->current_node == node)
    {
        this->count--;
        return;
    }

    //loop over all listeners except end. if this.count = 1, then this is skipped
    for (unsigned i = 0; i < this->count - 1; ++i)
    {
        if(this->listeners[i].state->current_node != node) continue;

        this->listeners[i] = *end; //switch the listener containing 'node to remove' with the end listener
        this->count--; //remove end node, which is now the node to remove
        return;
    }
}

CPU GPU channel_medium::channel_medium(channel_stack* store, const unsigned channels, const unsigned max_width)
{
    this->store_ = store;
    this->channels_ = channels;
    this->width_ = max_width;
}

CPU GPU channel_medium::channel_medium(const unsigned channels, const unsigned max_width)
{
    this->channels_ = channels;
    this->width_ = max_width;

    if(channels == 0 || max_width == 0)
    {
        this->store_ = nullptr;
        return;
    }
    this->store_ = static_cast<channel_stack*>(malloc(channels * sizeof(channel_stack)));
    for (unsigned i = 0; i < channels; ++i)
    {
        this->store_[i] = channel_stack{0,
            static_cast<channel_listener*>(malloc(max_width*sizeof(channel_listener))) };
    }
}

CPU GPU void channel_medium::init(const lend_array<model_state>* states) const
{
    for (int i = 0; i < states->size(); ++i)
    {
        this->add(states->at(i));
    }
}

CPU GPU void channel_medium::add(model_state* state) const
{
    node_t* node = state->current_node;
    const lend_array<edge_t*> edges = node->get_edges();
        
    for (int i = 0; i < edges.size(); ++i)
    {
        edge_t* edge = edges.get(i);
        const unsigned channel_id = edge->get_channel();
        if(channel_id == NO_CHANNEL) continue;

        this->store_[channel_id].add(state, edge);
    }
}

CPU GPU void channel_medium::remove(node_t* node) const
{
    const lend_array<edge_t*> edges = node->get_edges();
        
    for (int i = 0; i < edges.size(); ++i)
    {
        const unsigned channel_id = edges.get(i)->get_channel();
        if(channel_id == NO_CHANNEL) continue;

        this->store_[channel_id].remove(node);
    }
}

CPU GPU bool channel_medium::listener_exists(const unsigned channel_id) const
{
    return this->store_[channel_id].count > 0;
}

CPU GPU channel_listener* channel_medium::find_listener(const unsigned channel_id) const
{
    if (!this->listener_exists(channel_id)) return nullptr;
    const channel_stack* stack = &this->store_[channel_id];
    return &stack->listeners[stack->count - 1 ]; //returns end node
}


CPU GPU channel_listener* channel_medium::pick_random_valid_listener(const unsigned channel_id, simulator_state* state, hiprandState* r_state) const
{
    if(!listener_exists(channel_id)) return nullptr;
    const channel_stack* stack = &this->store_[channel_id];
    const unsigned start_index = hiprand(r_state) % stack->count;

    for (unsigned i = 0; i < stack->count; ++i)
    {
        channel_listener* listener = &stack->listeners[(start_index + i) % stack->count];
        if(listener->edge->evaluate_constraints(state))
            return listener;
    }

    return nullptr;
}

CPU GPU void channel_medium::clear() const
{
    for (unsigned i = 0; i < this->channels_; ++i)
    {
        this->store_[i].count = 0;
    }
}



