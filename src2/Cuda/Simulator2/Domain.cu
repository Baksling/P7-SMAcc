#include "hip/hip_runtime.h"
﻿#include "Domain.h"

#define DBL_EPSILON 2.2204460492503131e-016 // smallest such that 1.0+DBL_EPSILON != 1.0

CPU GPU double evaluate_expression_node(const expr* expr, state* state)
{
    double v1, v2;
    switch (expr->operand) {
    case expr::literal_ee:
        return expr->value;
    case expr::clock_variable_ee:
        return state->variables.store[expr->variable_id].temp_value;
    case expr::random_ee:
        v1 = state->value_stack.pop();
        return (1.0 - hiprand_uniform_double(state->random)) * v1;
    case expr::plus_ee: 
        v2 = state->value_stack.pop();
        v1 = state->value_stack.pop();
        return v1 + v2;
    case expr::minus_ee:
        v2 = state->value_stack.pop();
        v1 = state->value_stack.pop();
        return v1 - v2;
    case expr::multiply_ee: 
        v2 = state->value_stack.pop();
        v1 = state->value_stack.pop();
        return v1 * v2;
    case expr::division_ee:
        v2 = state->value_stack.pop();
        v1 = state->value_stack.pop();
        return v1 / v2;
    case expr::power_ee: 
        v2 = state->value_stack.pop();
        v1 = state->value_stack.pop();
        return pow(v1, v2);
    case expr::negation_ee: 
        v1 = state->value_stack.pop();
        return -v1;
    case expr::sqrt_ee: 
        v1 = state->value_stack.pop();
        return sqrt(v1);
    case expr::less_equal_ee: 
        v2 = state->value_stack.pop();
        v1 = state->value_stack.pop();
        return v1 <= v2;
    case expr::greater_equal_ee: 
        v2 = state->value_stack.pop();
        v1 = state->value_stack.pop();
        return v1 >= v2;
    case expr::less_ee: 
        v2 = state->value_stack.pop();
        v1 = state->value_stack.pop();
        return v1 < v2;
    case expr::greater_ee: 
        v2 = state->value_stack.pop();
        v1 = state->value_stack.pop();
        return v1 > v2;
    case expr::equal_ee: 
        v2 = state->value_stack.pop();
        v1 = state->value_stack.pop();
        return abs(v1 - v2) <= DBL_EPSILON;
    case expr::not_equal_ee: 
        v2 = state->value_stack.pop();
        v1 = state->value_stack.pop();
        return abs(v1 - v2) > DBL_EPSILON;
    case expr::not_ee:
        v1 = state->value_stack.pop();
        return abs(v1) < DBL_EPSILON * 1.0;
    case expr::conditional_ee:
        v1 = state->value_stack.pop();
        state->value_stack.pop();
        return v1;
    default: return 0.0;
    }
}

CPU GPU double expr::evaluate_expression(state* state)
{
    state->expr_stack.clear();
    state->value_stack.clear();
    expr* current = this;
    while (true)
    {
        while(current != nullptr)
        {
            state->expr_stack.push(current);
            
            if(!IS_LEAF(current->operand)) //only push twice if it has children
                state->expr_stack.push(current);
            
            current = current->left;
        }
        if(state->expr_stack.count() == 0)
        {
            break;
        }
        current = state->expr_stack.pop();
        
        if(state->expr_stack.count() > 0 && state->expr_stack.peak() == current)
        {
            current = (current->operand == conditional_ee && abs(state->value_stack.peak()) < DBL_EPSILON)
                ? current->conditional_else
                : current->right;
        }
        else
        {
            double val = evaluate_expression_node(current, state);
            state->value_stack.push(val);
            current = nullptr;
        }
    }

    if(state->value_stack.count() == 0)
    {
        printf("Expression evaluation ended in no values! PANIC!\n");
    }
    
    return state->value_stack.pop();
}


CPU GPU double node::max_progression(state* state, bool* is_finite) const
{
    double max_bound = INFINITY;

    for (int i = 0; i < this->invariants.size; ++i)
    {
        const constraint* con = &this->invariants.store[i];
        if(!IS_INVARIANT(con->operand)) continue;
        if(!con->uses_variable) continue;
        const clock_var var = state->variables.store[con->variable_id];
        if(var.rate == 0) continue;
        const double time = state->variables.store[con->variable_id].value;
        const double expr_value = con->expression->evaluate_expression(state);
        
        max_bound = fmin(max_bound,  (expr_value - time) / var.rate); //rate is >0.
    }
    *is_finite = isfinite(max_bound);
    return max_bound;
}

CPU GPU bool constraint::evaluate_constraint(state* state) const
{
    const double left = state->variables.store[this->variable_id].value;
    const double right = this->expression->evaluate_expression(state);

    switch (this->operand)
    {
    case constraint::less_equal_c: return left <= right;
    case constraint::less_c: return left < right;
    case constraint::greater_equal_c: return left >= right;
    case constraint::greater_c: return left < right;
    case constraint::equal_c: return left == right;  // NOLINT(clang-diagnostic-float-equal)
    case constraint::not_equal_c: return left != right;  // NOLINT(clang-diagnostic-float-equal)
    default: return false;
    }
}


CPU GPU bool constraint::evaluate_constraint_set(const arr<constraint>& con_arr, state* state)
{
    for (int i = 0; i < con_arr.size; ++i)
    {
        if(!con_arr.store[i].evaluate_constraint(state))
            return false;
    }
    return true;
}

CPU GPU inline void update::apply_temp_update(state* state) const
{
    const double value = this->expression->evaluate_expression(state);
    state->variables.store[this->variable_id].temp_value = value;
}

CPU GPU inline void update::apply_update(state* state) const
{
    const double value = this->expression->evaluate_expression(state);
    state->variables.store[this->variable_id].set_value(value);
}

CPU GPU inline void edge::apply_updates(state* state) const
{
    for (int i = 0; i < this->updates.size; ++i)
    {
        this->updates.store[i].apply_update(state);
    }
}

CPU GPU inline bool edge::edge_enabled(state* state) const
{
    for (int i = 0; i < this->guards.size; ++i)
    {
        if(!this->guards.store[i].evaluate_constraint(state))
            return false;
    }

    for (int i = 0; i < this->updates.size; ++i)
    {
        this->updates.store[i].apply_temp_update(state);
    }

    bool is_valid = true;
        
    for (int i = 0; i < this->dest->invariants.size; ++i)
    {
        if(this->dest->invariants.store[i].evaluate_constraint(state))
        {
            is_valid = false;
            break;
        }
    }

    //this is always <= state.variables.size
    for (int i = 0; i < this->updates.size; ++i)
    {
        const int id = this->updates.store[i].variable_id;
        state->variables.store[id].reset_temp();
    }

    return is_valid;
}

CPU GPU void inline state::broadcast_channel(const int channel, const node* source)
{
    if(!IS_BROADCASTER(channel)) return;

    for (int i = 0; i < this->models.size; ++i)
    {
        const node* current = this->models.store[i];
        if(current->id == source->id) continue;
        if(current->is_goal) continue;
        if(!constraint::evaluate_constraint_set(current->invariants, this)) continue;
        
        const unsigned offset = hiprand(this->random) % current->edges.size;
        for (int j = 0; j < current->edges.size; ++j)
        {
            const edge* current_e = &current->edges.store[(i + offset) % current->edges.size];
            if(!IS_LISTENER(current_e->channel)) continue;
            if(!CAN_SYNC(channel, current_e->channel)) continue;

            node* dest = current_e->dest;
            
            this->models.store[i] = dest;

            current_e->apply_updates(this);
            break;
        }
    }
}
