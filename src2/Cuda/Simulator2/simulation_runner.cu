﻿#include "simulation_runner.h"
#include "results/output_writer.h"
#include "allocations/cuda_allocator.h"
#include "allocations/memory_allocator.h"
#include "automata_engine.cu"
#include "common/macro.h"
#include "common/thread_pool.h"

using namespace std::chrono;

void simulation_runner::simulate_gpu(const automata* model, sim_config* config)
{
    memory_allocator allocator = memory_allocator(true);
    
    const size_t n_parallelism = static_cast<size_t>(config->blocks)*config->threads;
    const size_t total_simulations = config->total_simulations();

    cuda_allocator av = cuda_allocator(&allocator);
    const automata* model_d = av.allocate_automata(model);
    
    CUDA_CHECK(allocator.allocate_cuda(&config->cache, n_parallelism*thread_heap_size(config)));
    CUDA_CHECK(allocator.allocate_cuda(&config->random_state_arr, n_parallelism*sizeof(hiprandState)));

    sim_config* config_d = nullptr;
    CUDA_CHECK(allocator.allocate(&config_d, sizeof(sim_config)));
    CUDA_CHECK(hipMemcpy(config_d, config, sizeof(sim_config), hipMemcpyHostToDevice));
    
    const result_store store = result_store(
        static_cast<unsigned>(config->total_simulations()),
        config->tracked_variable_count,
        config->network_size,
        &allocator);
    
    result_store* store_d = nullptr;
    CUDA_CHECK(allocator.allocate(&store_d, sizeof(result_store)));
    CUDA_CHECK(hipMemcpy(store_d, &store, sizeof(result_store), hipMemcpyHostToDevice));
    
    output_writer writer = output_writer(
        &config->out_path,
        static_cast<unsigned>(total_simulations),
        config->write_mode,
        model        
        );

    if(config->verbose)  std::cout << "GPU simulation started\n";
    const steady_clock::time_point global_start = steady_clock::now();
    for (unsigned r = 0; r < config->simulation_repetitions; ++r)
    {
        const steady_clock::time_point local_start = steady_clock::now();
        simulator_gpu_kernel<<<config->blocks, config->threads>>>(model_d, store_d, config_d);
        hipDeviceSynchronize();
        if(hipPeekAtLastError() != hipSuccess)
            throw std::runtime_error("An error was encountered while running simulation. Error: " + std::to_string(hipPeekAtLastError()) + ".\n" );

        writer.write(
            &store,
            std::chrono::duration_cast<milliseconds>(steady_clock::now() - local_start));
    }
    if(config->verbose) std::cout << "GPU simulation finished\n";
    writer.write_summary(std::chrono::duration_cast<milliseconds>(steady_clock::now() - global_start));

    allocator.free_allocations();
}


void simulation_runner::simulate_cpu(const automata* model, sim_config* config)
{
    memory_allocator allocator = memory_allocator(false);
    
    const size_t n_parallelism = static_cast<size_t>(config->blocks)*config->threads;
    const size_t total_simulations = config->total_simulations();

    const result_store store = result_store(
    static_cast<unsigned>(total_simulations),
    config->tracked_variable_count,
    config->network_size,
    &allocator);

    output_writer writer = output_writer(
        &config->out_path,
        static_cast<unsigned>(total_simulations),
        config->write_mode,
        model        
        );
    
    CUDA_CHECK(allocator.allocate_host(&config->cache, n_parallelism*thread_heap_size(config)));
    CUDA_CHECK(allocator.allocate_host(&config->random_state_arr, n_parallelism*sizeof(hiprandState)));

    std::cout << "CPU simulation started\n";
    const steady_clock::time_point global_start = steady_clock::now();
    for (unsigned r = 0; r < config->simulation_repetitions; ++r)
    {
        const steady_clock::time_point local_start = steady_clock::now();
        thread_pool pool = {config->cpu_threads};
    
        for (int i = 0; i < n_parallelism; ++i)
        {
            pool.queue_job([model, &store, config, i]()
            {
                simulate_automata(i, model, &store, config);
            });
        }
        pool.await_run();

        writer.write(&store, std::chrono::duration_cast<milliseconds>(steady_clock::now() - local_start));
    }
    writer.write_summary(std::chrono::duration_cast<milliseconds>(steady_clock::now() - global_start));
    
    allocator.free_allocations();
}
