#include "hip/hip_runtime.h"
﻿#include <string>
#include "simulation_runner.h"

#include "../UPPAALXMLParser/uppaal_xml_parser.h"

#include "./results/output_writer.h"
#include "./allocations/argparser.h"
#include "common/io_paths.h"

#include "visitors/domain_optimization_visitor.h"
#include "visitors/model_count_visitor.h"
#include "visitors/pretty_print_visitor.h"


enum parser_state
{
    parsed,
    error,
    help
};

parser_state parse_configs(const int argc, const char* argv[], sim_config* config)
{

    argparse::ArgumentParser parser("Cuda stochastic system simulator", "Argument parser example");

    //model
    parser.add_argument("-m", "--model", "Model xml file path", false);

    //output 
    parser.add_argument("-o", "--output", "The path to output result file without file extension. e.g. './output' ", false);
    parser.add_argument("-w", "--write", "Write mode \n / c = console summary  \n / f = file summary \n / d = file data dump \n / t = trace \n / l = lite summary \n / m = write model to file \n / r = write hit percentage to file", false);
    
    //explicit block/thread specification
    parser.add_argument("-b", "--block", "Specify number of blocks/threads to use.\nIn the format 'blocks,threads' e.g. '32,512' for 32 blocks and 512 threads pr. block", false);

    //number of simulations to run.
    parser.add_argument("-e", "--epsilon", "epsilon value to calculate number of simulations", false);
    parser.add_argument("-a", "--alpha", "Specify number of blocks/threads to use.\nIn the format 'blocks,threads' e.g. '32,512' for 32 blocks and 512 threads pr. block", false);
    parser.add_argument("-n", "--number", "Specify the total number of simulations to run .\nIf this parameter is specified, epsilon and alpha are ignored", false);
    parser.add_argument("-r", "--repeat", "number of times to repeat simulations. Concats results. default = 1", false);

    //device options
    parser.add_argument("-d", "--device", "Where to run simulation. (GPU (0) / CPU (1) / BOTH (2)). default = 0", false);
    parser.add_argument("-c", "--cputhread", "The number of threads to use on the CPU. ", false);

    //simulation options
    parser.add_argument("-x", "--units", "Maximum number of steps or time to simulate. e.g. 100t for 100 time units or 100s for 100 steps", false);
    parser.add_argument("-s", "--shared", "Attempt to use shared memory in cuda simulation. Will only enable if (threads * 32 > model size)", false);
    parser.add_argument("-j", "--jit", "JIT compile the expressions. Only works for GPU, mutually exclusive with --shared.", false);
    
    
    //other
    parser.add_argument("-v", "--verbose", "Enable pretty print of model (print model (0) / silent(1))", false);
    parser.enable_help();

    const auto err = parser.parse(argc, argv);
    if (err) {
        std::cout << err << std::endl;
        return parser_state::error;
    }

    if (parser.exists("help")) {
        parser.print_help();
        return parser_state::help;
    }
    
    config->seed = static_cast<unsigned long long>(time(nullptr));
    size_t total_simulations;
    
    if(parser.exists("m")) config->paths->model_path = parser.get<std::string>("m");
    else throw argparse::arg_exception('m', "No model argument supplied");

    if(parser.exists("o")) config->paths->output_path = parser.get<std::string>("o");
    else config->paths->output_path = "./output";

    if(parser.exists("w")) config->write_mode = output_writer::parse_mode(parser.get<std::string>("w"));
    else config->write_mode = 0;

    if(parser.exists("b"))
    {
        if(!uppaal_xml_parser::try_parse_block_threads(
            parser.get<std::string>("b"),
            &config->blocks,
            &config->threads
            ))
                throw argparse::arg_exception('b', "could not parse block/threads. format: 'blocks,threads'. e.g. '32,512'");
    }
    else throw argparse::arg_exception('b', "no block arg supplied");

    if(parser.exists("n")) total_simulations = parser.get<size_t>("n");
    else if(parser.exists("e") && parser.exists("a"))
    {
        const double epsilon = parser.get<double>("e");
        const double alpha = parser.get<double>("a");
        total_simulations = static_cast<size_t>(ceil((log(2.0) - log(alpha)) / (2*pow(epsilon, 2))));
    }
    else throw argparse::arg_exception('n', "no simulation amount supplied. ");

    if(parser.exists("r")) config->simulation_repetitions = parser.get<unsigned>("r");
    else config->simulation_repetitions = 1;

    if(parser.exists("d")) config->sim_location = static_cast<sim_config::device_opt>(parser.get<int>("d"));
    else config->sim_location = sim_config::device;

    if(parser.exists("c")) config->cpu_threads = parser.get<unsigned>("c");
    else config->cpu_threads = 1;

    config->use_shared_memory = parser.exists("s");
    config->use_jit = parser.exists("j");
    
    if(parser.exists("x"))
    {
        bool is_timer;
        double unit_value = 0.0;
        const bool success = uppaal_xml_parser::try_parse_units(parser.get<std::string>("x"), &is_timer, &unit_value);
        if(!success) throw argparse::arg_exception('x', "could not parse unit format. e.g. 100t or 100s");
        config->use_max_steps = !is_timer;
        config->max_steps_pr_sim = static_cast<unsigned>(floor(unit_value));
        config->max_global_progression = unit_value;
    }
    else
    {
        config->use_max_steps = true;
        config->max_steps_pr_sim = 100;
        config->max_global_progression = 100;
    }

    if(parser.exists("v")) config->verbose = parser.get<int>("v");
    else config->verbose = true;
    
    config->simulation_amount = static_cast<unsigned>(ceil(
            static_cast<double>(total_simulations) /
            static_cast<double>((config->blocks * config->threads))));
    
    return parser_state::parsed;
}

void setup_config(sim_config* config, const network* model, const unsigned max_expr_depth, const unsigned max_fanout)
{
    unsigned track_count = 0;
    for (int i = 0; i < model->variables.size; ++i)
        if(model->variables.store[i].should_track)
            track_count++;

    config->tracked_variable_count = track_count;
    config->network_size = model->automatas.size;
    config->variable_count = model->variables.size;
    config->max_expression_depth = max_expr_depth;
    config->max_edge_fanout = max_fanout;
}

void print_config(const sim_config* config, const size_t model_size)
{
    printf("simulation configuration:\n");
    printf("simulating on model %s\n", config->paths->model_path.c_str());
    printf("running %llu simulations on %d repetitions using parallelism of %d.\n",
        static_cast<unsigned long long>(config->total_simulations()),
        config->simulation_repetitions,
        config->blocks*config->threads);
    printf("Model size: %llu bytes\n", static_cast<unsigned long long>(model_size));
    printf("attempt to use shared memory: %s (possible: %s)\n",
        (config->use_shared_memory ? "Yes" : "No" ),
        (config->can_use_cuda_shared_memory(model_size) ? "Yes" : "No"));
    printf("End criteria: %lf %s\n",
        (config->use_max_steps ? static_cast<double>(config->max_steps_pr_sim) : config->max_global_progression),
        (config->use_max_steps ? "steps" : "time units"));
}

int main(int argc, const char* argv[])
{
    CUDA_CHECK(hipFree(nullptr));

    io_paths paths = {};
    sim_config config = {};
    config.paths = &paths;
    parser_state state = parse_configs(argc, argv, &config);
    if(state == error) return -1;
    if(state == help) return 0;
    
    memory_allocator allocator = memory_allocator(
        config.sim_location == sim_config::device || config.sim_location == sim_config::both
        );
    
    uppaal_xml_parser xml_parser;
    network model = xml_parser.parse(config.paths->model_path);

    if(config.verbose)
        pretty_print_visitor(&std::cout).visit(&model);

    if(config.verbose) printf("Optimizing...\n");
    domain_optimization_visitor optimizer = domain_optimization_visitor();
    optimizer.optimize(&model);

    model_count_visitor count_visitor = model_count_visitor();
    count_visitor.visit(&model);
    
    model_size size_of_model = count_visitor.get_model_size();
    setup_config(&config, &model,
        optimizer.get_max_expr_depth(),
        optimizer.get_max_fanout());
    
    optimizer.clear();
    if(config.verbose) print_config(&config, size_of_model.total_memory_size());
    
    if(config.use_shared_memory)
        config.use_shared_memory = config.can_use_cuda_shared_memory(size_of_model.total_memory_size());

    const bool run_device = (config.sim_location == sim_config::device || config.sim_location == sim_config::both);
    const bool run_host   = (config.sim_location == sim_config::host   || config.sim_location == sim_config::both);

    //run simulation
    if(run_device)
    {
        if(config.use_jit)
            simulation_runner::simulate_gpu_jit(&model, &config);
        else
            simulation_runner::simulate_gpu(&model, &config);
    }
    if(run_host)
    {
        if(config.use_jit) throw std::runtime_error("Cannot run simulation on host, when JIT is enabled. Please disable jit(-j) and try again.");
        simulation_runner::simulate_cpu(&model, &config);
    }

    allocator.free_allocations();
    return 0;
}
