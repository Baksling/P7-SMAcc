﻿#include "thread_pool.h"

thread_pool::thread_pool(const unsigned max_concurrency)
{
    this->max_concurrency_ = max_concurrency == 0 ? std::thread::hardware_concurrency() : max_concurrency;
}

void thread_pool::await_run()
{
    this->start();
    while(this->is_busy())
    {
        std::this_thread::yield();
    }
    this->stop();
}

void thread_pool::start()
{
    // if 0 threads supplied, default to hardware default
    const unsigned num_threads = this->max_concurrency_ == 0 ? 1 : this->max_concurrency_;

    //pick lowest between user parameter and supported concurrency.
    threads_.resize(num_threads);
    for (unsigned i = 0; i < num_threads; i++) {
        threads_.at(i) = std::thread([this]() {this->thread_loop();});
    }
}

void thread_pool::queue_job(const std::function<void()>& job)
{
    {
        std::unique_lock<std::mutex> lock(this->queue_mutex_);
        this->jobs_.push(job);
    }
    this->mutex_condition_.notify_one();
}

void thread_pool::stop()
{
    {
        std::unique_lock<std::mutex> lock(this->queue_mutex_);
        this->should_terminate_ = true;
    }
    this->mutex_condition_.notify_all();
    for (std::thread& active_thread : this->threads_) {
        active_thread.join();
    }
    this->threads_.clear();
}

bool thread_pool::is_busy()
{
    bool pool_busy;
    {
        std::unique_lock<std::mutex> lock(this->queue_mutex_);
        pool_busy = !this->jobs_.empty();
    }
    return pool_busy;
}

void thread_pool::thread_loop()
{
    while (true) {
        std::function<void()> job;
        {
            std::unique_lock<std::mutex> lock(this->queue_mutex_);
            mutex_condition_.wait(lock, [this] {
                return !this->jobs_.empty() || this->should_terminate_;
            });
            if (this->should_terminate_) {
                return;
            }
            job = this->jobs_.front();
            this->jobs_.pop();
        }
        job();
    }
}
