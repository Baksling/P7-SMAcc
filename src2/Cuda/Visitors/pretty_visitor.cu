﻿#include "pretty_visitor.h"

#include "../Domain/expressions/constraint_t.h"
#include "../Domain/edge_t.h"
#include "../Domain/node_t.h"
#include "../Domain/stochastic_model_t.h"

void pretty_visitor::indentation() const
{
    for (int i = 0; i < scope_; ++i)
    {
        printf("    ");
    }

    //printf("-----scope: %d\n", this->scope_);
}

pretty_visitor::pretty_visitor()
{
    this->check_ = static_cast<int*>(malloc(3*sizeof(int)));
    for (int i = 0; i < 3; ++i)
    {
        check_[i] = 0;
    }
}

void pretty_visitor::visit(constraint_t* constraint)
{
    if (constraint == nullptr)
    {
        return;
    }
    indentation();
    constraint->pretty_print();
    scope_++;
    constraint->accept(this);
    scope_--;
}

void pretty_visitor::visit(edge_t* edge)
{
    if (edge == nullptr)
    {
        return;
    }
    indentation();
    edge->pretty_print();
    scope_++;
    edge->accept(this);
    scope_--;
}

void pretty_visitor::visit(node_t* node)
{
    if (node == nullptr)
    {
        return;
    }
    const bool is_new = checker_.insert(node->get_id()).second;
    if(!is_new) return;

    //std::cout << "set count for node :" << checker_.count(node) << "\n";
    // if (checker_.count(node))
    // {
    //     return;
    // }
    //if(check_[node->get_id()] > 0) return;
    //if(checker_.count(node) == 1) return;
    //check_[node->get_id()] = 1;
    //std::cout << "set count for node after insert:" << checker_.count(node) << "\n";
    checker_.insert(node->get_id());
    this->scope_ = 0;
    indentation();
    node->pretty_print();
    scope_++;
    node->accept(this);
    scope_--;

}

void pretty_visitor::visit(stochastic_model_t* model)
{
    if (model == nullptr) return;
    model->pretty_print();
    model->accept(this);
    printf("Model end\n");
    pretty_helper();
}

void pretty_visitor::visit(clock_variable* timer)
{
    if (timer == nullptr)
    {
        return;
    }
    indentation();
    timer->pretty_print();
    scope_++;
    timer->accept(this);
    scope_--;
}

void pretty_visitor::visit(update_t* update)
{
    if (update == nullptr)
    {
        return;
    }
    indentation();
    update->pretty_print();
    scope_++;
    update->accept(this);
    scope_--;
}

void pretty_visitor::visit(expression* expression)
{
    if (expression == nullptr)
    {
        return;
    }
    //indentation();
    //std::cout << expression->to_string();
    scope_++;
    expression->accept(this);
    scope_--;
}

void pretty_visitor::pretty_helper()
{
    // printf("⠀⠀⠀⠀⠀⢰⡿⠋⠁⠀⠀⠈⠉⠙⠻⣷⣄⠀⠀⠀⠀⠀⠀⠀⠀⠀⠀⠀⠀⠀\n");
    // printf("⠀⠀⠀⠀⢀⣿⠇⠀⢀⣴⣶⡾⠿⠿⠿⢿⣿⣦⡀⠀⠀⠀⠀⠀⠀⠀⠀⠀⠀⠀\n");
    // printf("⠀⠀⣀⣀⣸⡿⠀⠀⢸⣿⣇⠀⠀⠀⠀⠀⠀⠙⣷⡀⠀⠀⠀⠀⠀⠀⠀⠀⠀⠀\n");
    // printf("⠀⣾⡟⠛⣿⡇⠀⠀⢸⣿⣿⣷⣤⣤⣤⣤⣶⣶⣿⠇⠀⠀⠀⠀⠀⠀⠀⣀⠀⠀\n");
    // printf("⢀⣿⠀⢀⣿⡇⠀⠀⠀⠻⢿⣿⣿⣿⣿⣿⠿⣿⡏⠀⠀⠀⠀⢴⣶⣶⣿⣿⣿⣆\n");
    // printf("⢸⣿⠀⢸⣿⡇⠀⠀⠀⠀⠀⠈⠉⠁⠀⠀⠀⣿⡇⣀⣠⣴⣾⣮⣝⠿⠿⠿⣻⡟\n");
    // printf("⢸⣿⠀⠘⣿⡇⠀⠀⠀⠀⠀⠀⠀⣠⣶⣾⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⡿⠁⠉⠀\n");
    // printf("⠸⣿⠀⠀⣿⡇⠀⠀⠀⠀⠀⣠⣾⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⡿⠟⠉⠀⠀⠀⠀\n");
    // printf("⠀⠻⣷⣶⣿⣇⠀⠀⠀⢠⣼⣿⣿⣿⣿⣿⣿⣿⣛⣛⣻⠉⠁⠀⠀⠀⠀⠀⠀⠀\n");
    // printf("⠀⠀⠀⠀⢸⣿⠀⠀⠀⢸⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⡇⠀⠀⠀⠀⠀⠀⠀⠀\n");
    // printf("⠀⠀⠀⠀⢸⣿⣀⣀⣀⣼⡿⢿⣿⣿⣿⣿⣿⡿⣿⣿⡿\n");
    // printf("         BIGUS DICKUS        \n");


    printf("⠀⠀⠀⠀⠀⢀⣴⡾⠿⠿⠿⠿⢶⣦⣄⠀⠀⠀⠀⠀⠀⠀⠀⠀⠀\n");
    printf("⠀⠀⠀⠀⢠⣿⠁⠀⠀⠀⣀⣀⣀⣈⣻⣷⡄⠀⠀⠀⠀⠀⠀⠀⠀\n");
    printf("⠀⠀⠀⠀⣾⡇⠀⠀⣾⣟⠛⠋⠉⠉⠙⠛⢷⣄⠀⠀⠀⠀⠀⠀⠀\n");
    printf("⢀⣤⣴⣶⣿⠀⠀⢸⣿⣿⣧⠀⠀⠀⠀⢀⣀⢹⡆⠀⠀⠀⠀⠀⠀\n");
    printf("⢸⡏⠀⢸⣿⠀⠀⠀⢿⣿⣿⣷⣶⣶⣿⣿⣿⣿⠃⠀⠀⠀⠀⠀⠀\n");
    printf("⣼⡇⠀⢸⣿⠀⠀⠀⠈⠻⠿⣿⣿⠿⠿⠛⢻⡇⠀⠀⠀⠀⠀⠀⠀\n");
    printf("⣿⡇⠀⢸⣿⠀⠀⠀⠀⠀⠀⠀⠀⠀⣀⣤⣼⣷⣶⣶⣶⣤⡀⠀⠀\n");
    printf("⣿⡇⠀⢸⣿⠀⠀⠀⠀⠀⠀⣀⣴⣾⣿⣿⣿⣿⣿⣿⣿⣿⣿⣦⡀\n");
    printf("⢻⡇⠀⢸⣿⠀⠀⠀⠀⢀⣾⣿⣿⣿⣿⣿⣿⣿⡿⠿⣿⣿⣿⣿⡇\n");
    printf("⠈⠻⠷⠾⣿⠀⠀⠀⠀⣾⣿⣿⣿⣿⣿⣿⣿⣿⡇⠀⢸⣿⣿⣿⣇\n");
    printf("⠀⠀⠀⠀⣿⠀⠀⠀⠀⣿⣿⣿⣿⣿⣿⣿⣿⣿⠃⠀⢸⣿⣿⣿⡿\n");
    printf("⠀⠀⠀⠀⢿⣧⣀⣠⣴⡿⠙⠛⠿⠿⠿⠿⠉⠀⠀⢠⣿⣿⣿⣿⠇\n");
    printf("⠀⠀⠀⠀⠀⢈⣩⣭⣥⣤⣤⣤⣤⣤⣤⣤⣤⣤⣶⣿⣿⣿⣿⠏⠀\n");
    printf("⠀⠀⠀⠀⣴⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⡿⠋⠀⠀\n");
    printf("⠀⠀⠀⢸⣿⣿⣿⡟⠛⠛⠛⠛⠛⠛⠛⠛⠛⠛⠛⠋⠁⠀⠀⠀⠀\n");
    printf("⠀⠀⠀⢸⣿⣿⣿⣷⣄⣀⣀⣀⣀⣀⣀⣀⣀⣀⡀⠀⠀⠀⠀⠀⠀\n");
    printf("⠀⠀⠀⠀⠻⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣷⣦⡀⠀⠀⠀\n");
    printf("⠀⠀⠀⠀⠀⠈⠛⠿⠿⣿⣿⣿⣿⣿⠿⠿⢿⣿⣿⣿⣿⣿⡄⠀⠀\n");
    printf("⠀⠀⠀⠀⠀⠀⢀⣀⣀⣀⡀⠀⠀⠀⠀⠀⠀⢀⣹⣿⣿⣿⡇⠀⠀\n");
    printf("⠀⠀⠀⠀⠀⢰⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⡿⠁⠀⠀\n");
    printf("⠀⠀⠀⠀⣼⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⠿⠛⠁⠀⠀⠀\n");
    printf("⠀⠀⠀⠀⣿⣿⣿⣿⠁⠀⠀⠀⠀⠀⠉⠉⠁⢤⣤⣤⣤⣤⣤⣤⡀\n");
    printf("⠀⠀⠀⠀⢿⣿⣿⣿⣷⣶⣶⣶⣶⣾⣿⣿⣿⣆⢻⣿⣿⣿⣿⣿⡇\n");
    printf("⠀⠀⠀⠀⠈⠻⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣦⠻⣿⣿⣿⡿⠁\n");
    printf("⠀⠀⠀⠀⠀⠀⠈⠙⠛⠛⠛⠛⠛⠛⠛⠛⠛⠛⠉⠀⠙⠛⠉⠀⠀\n");
    printf("          BIGUS DICKUS        \n");
}



