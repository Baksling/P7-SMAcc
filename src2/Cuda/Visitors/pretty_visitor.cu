﻿#include "pretty_visitor.h"

#include "../Domain/constraint_t.h"
#include "../Domain/edge_t.h"
#include "../Domain/node_t.h"
#include "../Domain/stochastic_model_t.h"

void pretty_visitor::indentation() const
{
    for (int i = 0; i < scope_; ++i)
    {
        printf("    ");
    }
}

void pretty_visitor::visit(constraint_t* constraint)
{
    if (constraint == nullptr)
    {
        return;
    }
    indentation();
    constraint->pretty_print();
    scope_++;
    constraint->accept(this);
    scope_--;
}

void pretty_visitor::visit(edge_t* edge)
{
    if (edge == nullptr)
    {
        return;
    }
    indentation();
    edge->pretty_print();
    scope_++;
    edge->accept(this);
    scope_--;
}

void pretty_visitor::visit(node_t* node)
{
    if (node == nullptr)
    {
        return;
    }
    if (checker_.find(node) != checker_.end())
    {
        return;
    }
    checker_.insert(node);
    indentation();
    node->pretty_print();
    scope_++;
    node->accept(this);
    scope_--;
}

void pretty_visitor::visit(stochastic_model_t* model)
{
    if (model == nullptr) return;
    model->pretty_print();
    model->accept(this);
    printf("Model end\n");
    pretty_helper();
}

void pretty_visitor::visit(clock_variable* timer)
{
    if (timer == nullptr)
    {
        return;
    }
    indentation();
    timer->pretty_print();
    scope_++;
    timer->accept(this);
    scope_--;
}

void pretty_visitor::visit(update_t* update)
{
    if (update == nullptr)
    {
        return;
    }
    indentation();
    update->pretty_print();
    scope_++;
    update->accept(this);
    scope_--;
}

void pretty_visitor::visit(expression* expression)
{
    if (expression == nullptr)
    {
        return;
    }
    indentation();
    std::cout << expression->to_string();
    scope_++;
    expression->accept(this);
    scope_--;
}

void pretty_visitor::pretty_helper()
{
    // printf("⠀⠀⠀⠀⠀⢰⡿⠋⠁⠀⠀⠈⠉⠙⠻⣷⣄⠀⠀⠀⠀⠀⠀⠀⠀⠀⠀⠀⠀⠀\n");
    // printf("⠀⠀⠀⠀⢀⣿⠇⠀⢀⣴⣶⡾⠿⠿⠿⢿⣿⣦⡀⠀⠀⠀⠀⠀⠀⠀⠀⠀⠀⠀\n");
    // printf("⠀⠀⣀⣀⣸⡿⠀⠀⢸⣿⣇⠀⠀⠀⠀⠀⠀⠙⣷⡀⠀⠀⠀⠀⠀⠀⠀⠀⠀⠀\n");
    // printf("⠀⣾⡟⠛⣿⡇⠀⠀⢸⣿⣿⣷⣤⣤⣤⣤⣶⣶⣿⠇⠀⠀⠀⠀⠀⠀⠀⣀⠀⠀\n");
    // printf("⢀⣿⠀⢀⣿⡇⠀⠀⠀⠻⢿⣿⣿⣿⣿⣿⠿⣿⡏⠀⠀⠀⠀⢴⣶⣶⣿⣿⣿⣆\n");
    // printf("⢸⣿⠀⢸⣿⡇⠀⠀⠀⠀⠀⠈⠉⠁⠀⠀⠀⣿⡇⣀⣠⣴⣾⣮⣝⠿⠿⠿⣻⡟\n");
    // printf("⢸⣿⠀⠘⣿⡇⠀⠀⠀⠀⠀⠀⠀⣠⣶⣾⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⡿⠁⠉⠀\n");
    // printf("⠸⣿⠀⠀⣿⡇⠀⠀⠀⠀⠀⣠⣾⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⡿⠟⠉⠀⠀⠀⠀\n");
    // printf("⠀⠻⣷⣶⣿⣇⠀⠀⠀⢠⣼⣿⣿⣿⣿⣿⣿⣿⣛⣛⣻⠉⠁⠀⠀⠀⠀⠀⠀⠀\n");
    // printf("⠀⠀⠀⠀⢸⣿⠀⠀⠀⢸⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⡇⠀⠀⠀⠀⠀⠀⠀⠀\n");
    // printf("⠀⠀⠀⠀⢸⣿⣀⣀⣀⣼⡿⢿⣿⣿⣿⣿⣿⡿⣿⣿⡿\n");
    // printf("         BIGUS DICKUS        \n");


    printf("⠀⠀⠀⠀⠀⢀⣴⡾⠿⠿⠿⠿⢶⣦⣄⠀⠀⠀⠀⠀⠀⠀⠀⠀⠀\n");
    printf("⠀⠀⠀⠀⢠⣿⠁⠀⠀⠀⣀⣀⣀⣈⣻⣷⡄⠀⠀⠀⠀⠀⠀⠀⠀\n");
    printf("⠀⠀⠀⠀⣾⡇⠀⠀⣾⣟⠛⠋⠉⠉⠙⠛⢷⣄⠀⠀⠀⠀⠀⠀⠀\n");
    printf("⢀⣤⣴⣶⣿⠀⠀⢸⣿⣿⣧⠀⠀⠀⠀⢀⣀⢹⡆⠀⠀⠀⠀⠀⠀\n");
    printf("⢸⡏⠀⢸⣿⠀⠀⠀⢿⣿⣿⣷⣶⣶⣿⣿⣿⣿⠃⠀⠀⠀⠀⠀⠀\n");
    printf("⣼⡇⠀⢸⣿⠀⠀⠀⠈⠻⠿⣿⣿⠿⠿⠛⢻⡇⠀⠀⠀⠀⠀⠀⠀\n");
    printf("⣿⡇⠀⢸⣿⠀⠀⠀⠀⠀⠀⠀⠀⠀⣀⣤⣼⣷⣶⣶⣶⣤⡀⠀⠀\n");
    printf("⣿⡇⠀⢸⣿⠀⠀⠀⠀⠀⠀⣀⣴⣾⣿⣿⣿⣿⣿⣿⣿⣿⣿⣦⡀\n");
    printf("⢻⡇⠀⢸⣿⠀⠀⠀⠀⢀⣾⣿⣿⣿⣿⣿⣿⣿⡿⠿⣿⣿⣿⣿⡇\n");
    printf("⠈⠻⠷⠾⣿⠀⠀⠀⠀⣾⣿⣿⣿⣿⣿⣿⣿⣿⡇⠀⢸⣿⣿⣿⣇\n");
    printf("⠀⠀⠀⠀⣿⠀⠀⠀⠀⣿⣿⣿⣿⣿⣿⣿⣿⣿⠃⠀⢸⣿⣿⣿⡿\n");
    printf("⠀⠀⠀⠀⢿⣧⣀⣠⣴⡿⠙⠛⠿⠿⠿⠿⠉⠀⠀⢠⣿⣿⣿⣿⠇\n");
    printf("⠀⠀⠀⠀⠀⢈⣩⣭⣥⣤⣤⣤⣤⣤⣤⣤⣤⣤⣶⣿⣿⣿⣿⠏⠀\n");
    printf("⠀⠀⠀⠀⣴⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⡿⠋⠀⠀\n");
    printf("⠀⠀⠀⢸⣿⣿⣿⡟⠛⠛⠛⠛⠛⠛⠛⠛⠛⠛⠛⠋⠁⠀⠀⠀⠀\n");
    printf("⠀⠀⠀⢸⣿⣿⣿⣷⣄⣀⣀⣀⣀⣀⣀⣀⣀⣀⡀⠀⠀⠀⠀⠀⠀\n");
    printf("⠀⠀⠀⠀⠻⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣷⣦⡀⠀⠀⠀\n");
    printf("⠀⠀⠀⠀⠀⠈⠛⠿⠿⣿⣿⣿⣿⣿⠿⠿⢿⣿⣿⣿⣿⣿⡄⠀⠀\n");
    printf("⠀⠀⠀⠀⠀⠀⢀⣀⣀⣀⡀⠀⠀⠀⠀⠀⠀⢀⣹⣿⣿⣿⡇⠀⠀\n");
    printf("⠀⠀⠀⠀⠀⢰⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⡿⠁⠀⠀\n");
    printf("⠀⠀⠀⠀⣼⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⠿⠛⠁⠀⠀⠀\n");
    printf("⠀⠀⠀⠀⣿⣿⣿⣿⠁⠀⠀⠀⠀⠀⠉⠉⠁⢤⣤⣤⣤⣤⣤⣤⡀\n");
    printf("⠀⠀⠀⠀⢿⣿⣿⣿⣷⣶⣶⣶⣶⣾⣿⣿⣿⣆⢻⣿⣿⣿⣿⣿⡇\n");
    printf("⠀⠀⠀⠀⠈⠻⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣦⠻⣿⣿⣿⡿⠁\n");
    printf("⠀⠀⠀⠀⠀⠀⠈⠙⠛⠛⠛⠛⠛⠛⠛⠛⠛⠛⠉⠀⠙⠛⠉⠀⠀\n");
    printf("          BIGUS DICKUS        \n");
}



